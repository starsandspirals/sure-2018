#include "hip/hip_runtime.h"

/*
 * FLAME GPU v 1.5.X for CUDA 9
 * Copyright University of Sheffield.
 * Original Author: Dr Paul Richmond (user contributions tracked on https://github.com/FLAMEGPU/FLAMEGPU)
 * Contact: p.richmond@sheffield.ac.uk (http://www.paulrichmond.staff.shef.ac.uk)
 *
 * University of Sheffield retain all intellectual property and
 * proprietary rights in and to this software and related documentation.
 * Any use, reproduction, disclosure, or distribution of this software
 * and related documentation without an express license agreement from
 * University of Sheffield is strictly prohibited.
 *
 * For terms of licence agreement please attached licence or view licence
 * on www.flamegpu.com website.
 *
 */


#ifndef _FLAMEGPU_KERNELS_H_
#define _FLAMEGPU_KERNELS_H_

#include "header.h"


/* Agent count constants */

__constant__ int d_xmachine_memory_Person_count;

__constant__ int d_xmachine_memory_TBAssignment_count;

__constant__ int d_xmachine_memory_Household_count;

__constant__ int d_xmachine_memory_HouseholdMembership_count;

__constant__ int d_xmachine_memory_Church_count;

__constant__ int d_xmachine_memory_ChurchMembership_count;

__constant__ int d_xmachine_memory_Transport_count;

__constant__ int d_xmachine_memory_TransportMembership_count;

__constant__ int d_xmachine_memory_Clinic_count;

__constant__ int d_xmachine_memory_Workplace_count;

__constant__ int d_xmachine_memory_WorkplaceMembership_count;

__constant__ int d_xmachine_memory_Bar_count;

/* Agent state count constants */

__constant__ int d_xmachine_memory_Person_default_count;

__constant__ int d_xmachine_memory_Person_s2_count;

__constant__ int d_xmachine_memory_TBAssignment_tbdefault_count;

__constant__ int d_xmachine_memory_Household_hhdefault_count;

__constant__ int d_xmachine_memory_HouseholdMembership_hhmembershipdefault_count;

__constant__ int d_xmachine_memory_Church_chudefault_count;

__constant__ int d_xmachine_memory_ChurchMembership_chumembershipdefault_count;

__constant__ int d_xmachine_memory_Transport_trdefault_count;

__constant__ int d_xmachine_memory_TransportMembership_trmembershipdefault_count;

__constant__ int d_xmachine_memory_Clinic_cldefault_count;

__constant__ int d_xmachine_memory_Workplace_wpdefault_count;

__constant__ int d_xmachine_memory_WorkplaceMembership_wpmembershipdefault_count;

__constant__ int d_xmachine_memory_Bar_bdefault_count;


/* Message constants */

/* tb_assignment Message variables */
/* Non partitioned and spatial partitioned message variables  */
__constant__ int d_message_tb_assignment_count;         /**< message list counter*/
__constant__ int d_message_tb_assignment_output_type;   /**< message output type (single or optional)*/

/* household_membership Message variables */
/* Non partitioned and spatial partitioned message variables  */
__constant__ int d_message_household_membership_count;         /**< message list counter*/
__constant__ int d_message_household_membership_output_type;   /**< message output type (single or optional)*/

/* church_membership Message variables */
/* Non partitioned and spatial partitioned message variables  */
__constant__ int d_message_church_membership_count;         /**< message list counter*/
__constant__ int d_message_church_membership_output_type;   /**< message output type (single or optional)*/

/* transport_membership Message variables */
/* Non partitioned and spatial partitioned message variables  */
__constant__ int d_message_transport_membership_count;         /**< message list counter*/
__constant__ int d_message_transport_membership_output_type;   /**< message output type (single or optional)*/

/* workplace_membership Message variables */
/* Non partitioned and spatial partitioned message variables  */
__constant__ int d_message_workplace_membership_count;         /**< message list counter*/
__constant__ int d_message_workplace_membership_output_type;   /**< message output type (single or optional)*/

/* location Message variables */
/* Non partitioned and spatial partitioned message variables  */
__constant__ int d_message_location_count;         /**< message list counter*/
__constant__ int d_message_location_output_type;   /**< message output type (single or optional)*/

/* infection Message variables */
/* Non partitioned and spatial partitioned message variables  */
__constant__ int d_message_infection_count;         /**< message list counter*/
__constant__ int d_message_infection_output_type;   /**< message output type (single or optional)*/

	
    
//include each function file

#include "functions.c"
    
/* Texture bindings */







    
#define WRAP(x,m) (((x)<m)?(x):(x%m)) /**< Simple wrap */
#define sWRAP(x,m) (((x)<m)?(((x)<0)?(m+(x)):(x)):(m-(x))) /**<signed integer wrap (no modulus) for negatives where 2m > |x| > m */

//PADDING WILL ONLY AVOID SM CONFLICTS FOR 32BIT
//SM_OFFSET REQUIRED AS FERMI STARTS INDEXING MEMORY FROM LOCATION 0 (i.e. NULL)??
__constant__ int d_SM_START;
__constant__ int d_PADDING;

//SM addressing macro to avoid conflicts (32 bit only)
#define SHARE_INDEX(i, s) ((((s) + d_PADDING)* (i))+d_SM_START) /**<offset struct size by padding to avoid bank conflicts */

//if doubel support is needed then define the following function which requires sm_13 or later
#ifdef _DOUBLE_SUPPORT_REQUIRED_
__inline__ __device__ double tex1DfetchDouble(texture<int2, 1, hipReadModeElementType> tex, int i)
{
	int2 v = tex1Dfetch(tex, i);
  //IF YOU HAVE AN ERROR HERE THEN YOU ARE USING DOUBLE VALUES IN AGENT MEMORY AND NOT COMPILING FOR DOUBLE SUPPORTED HARDWARE
  //To compile for double supported hardware change the CUDA Build rule property "Use sm_13 Architecture (double support)" on the CUDA-Specific Propert Page of the CUDA Build Rule for simulation.cu
	return __hiloint2double(v.y, v.x);
}
#endif

/* Helper functions */
/** next_cell
 * Function used for finding the next cell when using spatial partitioning
 * Upddates the relative cell variable which can have value of -1, 0 or +1
 * @param relative_cell pointer to the relative cell position
 * @return boolean if there is a next cell. True unless relative_Cell value was 1,1,1
 */
__device__ bool next_cell3D(glm::ivec3* relative_cell)
{
	if (relative_cell->x < 1)
	{
		relative_cell->x++;
		return true;
	}
	relative_cell->x = -1;

	if (relative_cell->y < 1)
	{
		relative_cell->y++;
		return true;
	}
	relative_cell->y = -1;
	
	if (relative_cell->z < 1)
	{
		relative_cell->z++;
		return true;
	}
	relative_cell->z = -1;
	
	return false;
}

/** next_cell2D
 * Function used for finding the next cell when using spatial partitioning. Z component is ignored
 * Upddates the relative cell variable which can have value of -1, 0 or +1
 * @param relative_cell pointer to the relative cell position
 * @return boolean if there is a next cell. True unless relative_Cell value was 1,1
 */
__device__ bool next_cell2D(glm::ivec3* relative_cell)
{
	if (relative_cell->x < 1)
	{
		relative_cell->x++;
		return true;
	}
	relative_cell->x = -1;

	if (relative_cell->y < 1)
	{
		relative_cell->y++;
		return true;
	}
	relative_cell->y = -1;
	
	return false;
}


/** infect_function_filter
 *	Standard agent condition function. Filters agents from one state list to the next depending on the condition
 * @param currentState xmachine_memory_Person_list representing agent i the current state
 * @param nextState xmachine_memory_Person_list representing agent i the next state
 */
 __global__ void infect_function_filter(xmachine_memory_Person_list* currentState, xmachine_memory_Person_list* nextState)
 {
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;
	
	//check thread max
	if (index < d_xmachine_memory_Person_count){
	
		//apply the filter
		if ((((currentState->lambda[index]==0.0)||(currentState->location[index]==7))||(currentState->activetb[index]==1))==0)
		{	//copy agent data to newstate list
			nextState->id[index] = currentState->id[index];
			nextState->step[index] = currentState->step[index];
			nextState->householdtime[index] = currentState->householdtime[index];
			nextState->churchtime[index] = currentState->churchtime[index];
			nextState->transporttime[index] = currentState->transporttime[index];
			nextState->clinictime[index] = currentState->clinictime[index];
			nextState->workplacetime[index] = currentState->workplacetime[index];
			nextState->outsidetime[index] = currentState->outsidetime[index];
			nextState->age[index] = currentState->age[index];
			nextState->gender[index] = currentState->gender[index];
			nextState->householdsize[index] = currentState->householdsize[index];
			nextState->churchfreq[index] = currentState->churchfreq[index];
			nextState->churchdur[index] = currentState->churchdur[index];
			nextState->transportdur[index] = currentState->transportdur[index];
			nextState->transportday1[index] = currentState->transportday1[index];
			nextState->transportday2[index] = currentState->transportday2[index];
			nextState->household[index] = currentState->household[index];
			nextState->church[index] = currentState->church[index];
			nextState->transport[index] = currentState->transport[index];
			nextState->workplace[index] = currentState->workplace[index];
			nextState->busy[index] = currentState->busy[index];
			nextState->startstep[index] = currentState->startstep[index];
			nextState->location[index] = currentState->location[index];
			nextState->locationid[index] = currentState->locationid[index];
			nextState->hiv[index] = currentState->hiv[index];
			nextState->art[index] = currentState->art[index];
			nextState->activetb[index] = currentState->activetb[index];
			nextState->artday[index] = currentState->artday[index];
			nextState->p[index] = currentState->p[index];
			nextState->q[index] = currentState->q[index];
			nextState->infections[index] = currentState->infections[index];
			nextState->lastinfected[index] = currentState->lastinfected[index];
			nextState->lastinfectedid[index] = currentState->lastinfectedid[index];
			nextState->time_step[index] = currentState->time_step[index];
			nextState->lambda[index] = currentState->lambda[index];
			nextState->timevisiting[index] = currentState->timevisiting[index];
			nextState->bargoing[index] = currentState->bargoing[index];
			nextState->barday[index] = currentState->barday[index];
			//set scan input flag to 1
			nextState->_scan_input[index] = 1;
		}
		else
		{
			//set scan input flag of current state to 1 (keep agent)
			currentState->_scan_input[index] = 1;
		}
	
	}
 }

/** hhupdate_function_filter
 *	Standard agent condition function. Filters agents from one state list to the next depending on the condition
 * @param currentState xmachine_memory_Household_list representing agent i the current state
 * @param nextState xmachine_memory_Household_list representing agent i the next state
 */
 __global__ void hhupdate_function_filter(xmachine_memory_Household_list* currentState, xmachine_memory_Household_list* nextState)
 {
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;
	
	//check thread max
	if (index < d_xmachine_memory_Household_count){
	
		//apply the filter
		if (currentState->active[index]==1)
		{	//copy agent data to newstate list
			nextState->id[index] = currentState->id[index];
			nextState->lambda[index] = currentState->lambda[index];
			nextState->active[index] = currentState->active[index];
			//set scan input flag to 1
			nextState->_scan_input[index] = 1;
		}
		else
		{
			//set scan input flag of current state to 1 (keep agent)
			currentState->_scan_input[index] = 1;
		}
	
	}
 }

/** chuupdate_function_filter
 *	Standard agent condition function. Filters agents from one state list to the next depending on the condition
 * @param currentState xmachine_memory_Church_list representing agent i the current state
 * @param nextState xmachine_memory_Church_list representing agent i the next state
 */
 __global__ void chuupdate_function_filter(xmachine_memory_Church_list* currentState, xmachine_memory_Church_list* nextState)
 {
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;
	
	//check thread max
	if (index < d_xmachine_memory_Church_count){
	
		//apply the filter
		if (currentState->active[index]==1)
		{	//copy agent data to newstate list
			nextState->id[index] = currentState->id[index];
			nextState->size[index] = currentState->size[index];
			nextState->lambda[index] = currentState->lambda[index];
			nextState->active[index] = currentState->active[index];
			//set scan input flag to 1
			nextState->_scan_input[index] = 1;
		}
		else
		{
			//set scan input flag of current state to 1 (keep agent)
			currentState->_scan_input[index] = 1;
		}
	
	}
 }

/** trupdate_function_filter
 *	Standard agent condition function. Filters agents from one state list to the next depending on the condition
 * @param currentState xmachine_memory_Transport_list representing agent i the current state
 * @param nextState xmachine_memory_Transport_list representing agent i the next state
 */
 __global__ void trupdate_function_filter(xmachine_memory_Transport_list* currentState, xmachine_memory_Transport_list* nextState)
 {
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;
	
	//check thread max
	if (index < d_xmachine_memory_Transport_count){
	
		//apply the filter
		if (currentState->active[index]==1)
		{	//copy agent data to newstate list
			nextState->id[index] = currentState->id[index];
			nextState->lambda[index] = currentState->lambda[index];
			nextState->active[index] = currentState->active[index];
			//set scan input flag to 1
			nextState->_scan_input[index] = 1;
		}
		else
		{
			//set scan input flag of current state to 1 (keep agent)
			currentState->_scan_input[index] = 1;
		}
	
	}
 }

////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dyanamically created Person agent functions */

/** reset_Person_scan_input
 * Person agent reset scan input function
 * @param agents The xmachine_memory_Person_list agent list
 */
__global__ void reset_Person_scan_input(xmachine_memory_Person_list* agents){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	agents->_position[index] = 0;
	agents->_scan_input[index] = 0;
}



/** scatter_Person_Agents
 * Person scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_Person_list agent list destination
 * @param agents_src xmachine_memory_Person_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void scatter_Person_Agents(xmachine_memory_Person_list* agents_dst, xmachine_memory_Person_list* agents_src, int dst_agent_count, int number_to_scatter){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = agents_src->_scan_input[index];

	//if optional message is to be written. 
	//must check agent is within number to scatter as unused threads may have scan input = 1
	if ((_scan_input == 1)&&(index < number_to_scatter)){
		int output_index = agents_src->_position[index] + dst_agent_count;

		//AoS - xmachine_message_location Un-Coalesced scattered memory write     
        agents_dst->_position[output_index] = output_index;        
		agents_dst->id[output_index] = agents_src->id[index];        
		agents_dst->step[output_index] = agents_src->step[index];        
		agents_dst->householdtime[output_index] = agents_src->householdtime[index];        
		agents_dst->churchtime[output_index] = agents_src->churchtime[index];        
		agents_dst->transporttime[output_index] = agents_src->transporttime[index];        
		agents_dst->clinictime[output_index] = agents_src->clinictime[index];        
		agents_dst->workplacetime[output_index] = agents_src->workplacetime[index];        
		agents_dst->outsidetime[output_index] = agents_src->outsidetime[index];        
		agents_dst->age[output_index] = agents_src->age[index];        
		agents_dst->gender[output_index] = agents_src->gender[index];        
		agents_dst->householdsize[output_index] = agents_src->householdsize[index];        
		agents_dst->churchfreq[output_index] = agents_src->churchfreq[index];        
		agents_dst->churchdur[output_index] = agents_src->churchdur[index];        
		agents_dst->transportdur[output_index] = agents_src->transportdur[index];        
		agents_dst->transportday1[output_index] = agents_src->transportday1[index];        
		agents_dst->transportday2[output_index] = agents_src->transportday2[index];        
		agents_dst->household[output_index] = agents_src->household[index];        
		agents_dst->church[output_index] = agents_src->church[index];        
		agents_dst->transport[output_index] = agents_src->transport[index];        
		agents_dst->workplace[output_index] = agents_src->workplace[index];        
		agents_dst->busy[output_index] = agents_src->busy[index];        
		agents_dst->startstep[output_index] = agents_src->startstep[index];        
		agents_dst->location[output_index] = agents_src->location[index];        
		agents_dst->locationid[output_index] = agents_src->locationid[index];        
		agents_dst->hiv[output_index] = agents_src->hiv[index];        
		agents_dst->art[output_index] = agents_src->art[index];        
		agents_dst->activetb[output_index] = agents_src->activetb[index];        
		agents_dst->artday[output_index] = agents_src->artday[index];        
		agents_dst->p[output_index] = agents_src->p[index];        
		agents_dst->q[output_index] = agents_src->q[index];        
		agents_dst->infections[output_index] = agents_src->infections[index];        
		agents_dst->lastinfected[output_index] = agents_src->lastinfected[index];        
		agents_dst->lastinfectedid[output_index] = agents_src->lastinfectedid[index];        
		agents_dst->time_step[output_index] = agents_src->time_step[index];        
		agents_dst->lambda[output_index] = agents_src->lambda[index];        
		agents_dst->timevisiting[output_index] = agents_src->timevisiting[index];        
		agents_dst->bargoing[output_index] = agents_src->bargoing[index];        
		agents_dst->barday[output_index] = agents_src->barday[index];
	}
}

/** append_Person_Agents
 * Person scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_Person_list agent list destination
 * @param agents_src xmachine_memory_Person_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void append_Person_Agents(xmachine_memory_Person_list* agents_dst, xmachine_memory_Person_list* agents_src, int dst_agent_count, int number_to_append){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	//must check agent is within number to append as unused threads may have scan input = 1
    if (index < number_to_append){
	    int output_index = index + dst_agent_count;

	    //AoS - xmachine_message_location Un-Coalesced scattered memory write
	    agents_dst->_position[output_index] = output_index;
	    agents_dst->id[output_index] = agents_src->id[index];
	    agents_dst->step[output_index] = agents_src->step[index];
	    agents_dst->householdtime[output_index] = agents_src->householdtime[index];
	    agents_dst->churchtime[output_index] = agents_src->churchtime[index];
	    agents_dst->transporttime[output_index] = agents_src->transporttime[index];
	    agents_dst->clinictime[output_index] = agents_src->clinictime[index];
	    agents_dst->workplacetime[output_index] = agents_src->workplacetime[index];
	    agents_dst->outsidetime[output_index] = agents_src->outsidetime[index];
	    agents_dst->age[output_index] = agents_src->age[index];
	    agents_dst->gender[output_index] = agents_src->gender[index];
	    agents_dst->householdsize[output_index] = agents_src->householdsize[index];
	    agents_dst->churchfreq[output_index] = agents_src->churchfreq[index];
	    agents_dst->churchdur[output_index] = agents_src->churchdur[index];
	    agents_dst->transportdur[output_index] = agents_src->transportdur[index];
	    agents_dst->transportday1[output_index] = agents_src->transportday1[index];
	    agents_dst->transportday2[output_index] = agents_src->transportday2[index];
	    agents_dst->household[output_index] = agents_src->household[index];
	    agents_dst->church[output_index] = agents_src->church[index];
	    agents_dst->transport[output_index] = agents_src->transport[index];
	    agents_dst->workplace[output_index] = agents_src->workplace[index];
	    agents_dst->busy[output_index] = agents_src->busy[index];
	    agents_dst->startstep[output_index] = agents_src->startstep[index];
	    agents_dst->location[output_index] = agents_src->location[index];
	    agents_dst->locationid[output_index] = agents_src->locationid[index];
	    agents_dst->hiv[output_index] = agents_src->hiv[index];
	    agents_dst->art[output_index] = agents_src->art[index];
	    agents_dst->activetb[output_index] = agents_src->activetb[index];
	    agents_dst->artday[output_index] = agents_src->artday[index];
	    agents_dst->p[output_index] = agents_src->p[index];
	    agents_dst->q[output_index] = agents_src->q[index];
	    agents_dst->infections[output_index] = agents_src->infections[index];
	    agents_dst->lastinfected[output_index] = agents_src->lastinfected[index];
	    agents_dst->lastinfectedid[output_index] = agents_src->lastinfectedid[index];
	    agents_dst->time_step[output_index] = agents_src->time_step[index];
	    agents_dst->lambda[output_index] = agents_src->lambda[index];
	    agents_dst->timevisiting[output_index] = agents_src->timevisiting[index];
	    agents_dst->bargoing[output_index] = agents_src->bargoing[index];
	    agents_dst->barday[output_index] = agents_src->barday[index];
    }
}

/** add_Person_agent
 * Continuous Person agent add agent function writes agent data to agent swap
 * @param agents xmachine_memory_Person_list to add agents to 
 * @param id agent variable of type unsigned int
 * @param step agent variable of type unsigned int
 * @param householdtime agent variable of type unsigned int
 * @param churchtime agent variable of type unsigned int
 * @param transporttime agent variable of type unsigned int
 * @param clinictime agent variable of type unsigned int
 * @param workplacetime agent variable of type unsigned int
 * @param outsidetime agent variable of type unsigned int
 * @param age agent variable of type unsigned int
 * @param gender agent variable of type unsigned int
 * @param householdsize agent variable of type unsigned int
 * @param churchfreq agent variable of type unsigned int
 * @param churchdur agent variable of type float
 * @param transportdur agent variable of type unsigned int
 * @param transportday1 agent variable of type int
 * @param transportday2 agent variable of type int
 * @param household agent variable of type unsigned int
 * @param church agent variable of type int
 * @param transport agent variable of type int
 * @param workplace agent variable of type int
 * @param busy agent variable of type unsigned int
 * @param startstep agent variable of type unsigned int
 * @param location agent variable of type unsigned int
 * @param locationid agent variable of type unsigned int
 * @param hiv agent variable of type unsigned int
 * @param art agent variable of type unsigned int
 * @param activetb agent variable of type unsigned int
 * @param artday agent variable of type unsigned int
 * @param p agent variable of type float
 * @param q agent variable of type float
 * @param infections agent variable of type unsigned int
 * @param lastinfected agent variable of type int
 * @param lastinfectedid agent variable of type int
 * @param time_step agent variable of type float
 * @param lambda agent variable of type float
 * @param timevisiting agent variable of type unsigned int
 * @param bargoing agent variable of type unsigned int
 * @param barday agent variable of type unsigned int
 */
template <int AGENT_TYPE>
__device__ void add_Person_agent(xmachine_memory_Person_list* agents, unsigned int id, unsigned int step, unsigned int householdtime, unsigned int churchtime, unsigned int transporttime, unsigned int clinictime, unsigned int workplacetime, unsigned int outsidetime, unsigned int age, unsigned int gender, unsigned int householdsize, unsigned int churchfreq, float churchdur, unsigned int transportdur, int transportday1, int transportday2, unsigned int household, int church, int transport, int workplace, unsigned int busy, unsigned int startstep, unsigned int location, unsigned int locationid, unsigned int hiv, unsigned int art, unsigned int activetb, unsigned int artday, float p, float q, unsigned int infections, int lastinfected, int lastinfectedid, float time_step, float lambda, unsigned int timevisiting, unsigned int bargoing, unsigned int barday){
	
	int index;
    
    //calculate the agents index in global agent list (depends on agent type)
	if (AGENT_TYPE == DISCRETE_2D){
		int width = (blockDim.x* gridDim.x);
		glm::ivec2 global_position;
		global_position.x = (blockIdx.x*blockDim.x) + threadIdx.x;
		global_position.y = (blockIdx.y*blockDim.y) + threadIdx.y;
		index = global_position.x + (global_position.y* width);
	}else//AGENT_TYPE == CONTINOUS
		index = threadIdx.x + blockIdx.x*blockDim.x;

	//for prefix sum
	agents->_position[index] = 0;
	agents->_scan_input[index] = 1;

	//write data to new buffer
	agents->id[index] = id;
	agents->step[index] = step;
	agents->householdtime[index] = householdtime;
	agents->churchtime[index] = churchtime;
	agents->transporttime[index] = transporttime;
	agents->clinictime[index] = clinictime;
	agents->workplacetime[index] = workplacetime;
	agents->outsidetime[index] = outsidetime;
	agents->age[index] = age;
	agents->gender[index] = gender;
	agents->householdsize[index] = householdsize;
	agents->churchfreq[index] = churchfreq;
	agents->churchdur[index] = churchdur;
	agents->transportdur[index] = transportdur;
	agents->transportday1[index] = transportday1;
	agents->transportday2[index] = transportday2;
	agents->household[index] = household;
	agents->church[index] = church;
	agents->transport[index] = transport;
	agents->workplace[index] = workplace;
	agents->busy[index] = busy;
	agents->startstep[index] = startstep;
	agents->location[index] = location;
	agents->locationid[index] = locationid;
	agents->hiv[index] = hiv;
	agents->art[index] = art;
	agents->activetb[index] = activetb;
	agents->artday[index] = artday;
	agents->p[index] = p;
	agents->q[index] = q;
	agents->infections[index] = infections;
	agents->lastinfected[index] = lastinfected;
	agents->lastinfectedid[index] = lastinfectedid;
	agents->time_step[index] = time_step;
	agents->lambda[index] = lambda;
	agents->timevisiting[index] = timevisiting;
	agents->bargoing[index] = bargoing;
	agents->barday[index] = barday;

}

//non templated version assumes DISCRETE_2D but works also for CONTINUOUS
__device__ void add_Person_agent(xmachine_memory_Person_list* agents, unsigned int id, unsigned int step, unsigned int householdtime, unsigned int churchtime, unsigned int transporttime, unsigned int clinictime, unsigned int workplacetime, unsigned int outsidetime, unsigned int age, unsigned int gender, unsigned int householdsize, unsigned int churchfreq, float churchdur, unsigned int transportdur, int transportday1, int transportday2, unsigned int household, int church, int transport, int workplace, unsigned int busy, unsigned int startstep, unsigned int location, unsigned int locationid, unsigned int hiv, unsigned int art, unsigned int activetb, unsigned int artday, float p, float q, unsigned int infections, int lastinfected, int lastinfectedid, float time_step, float lambda, unsigned int timevisiting, unsigned int bargoing, unsigned int barday){
    add_Person_agent<DISCRETE_2D>(agents, id, step, householdtime, churchtime, transporttime, clinictime, workplacetime, outsidetime, age, gender, householdsize, churchfreq, churchdur, transportdur, transportday1, transportday2, household, church, transport, workplace, busy, startstep, location, locationid, hiv, art, activetb, artday, p, q, infections, lastinfected, lastinfectedid, time_step, lambda, timevisiting, bargoing, barday);
}

/** reorder_Person_agents
 * Continuous Person agent areorder function used after key value pairs have been sorted
 * @param values sorted index values
 * @param unordered_agents list of unordered agents
 * @ param ordered_agents list used to output ordered agents
 */
__global__ void reorder_Person_agents(unsigned int* values, xmachine_memory_Person_list* unordered_agents, xmachine_memory_Person_list* ordered_agents)
{
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	uint old_pos = values[index];

	//reorder agent data
	ordered_agents->id[index] = unordered_agents->id[old_pos];
	ordered_agents->step[index] = unordered_agents->step[old_pos];
	ordered_agents->householdtime[index] = unordered_agents->householdtime[old_pos];
	ordered_agents->churchtime[index] = unordered_agents->churchtime[old_pos];
	ordered_agents->transporttime[index] = unordered_agents->transporttime[old_pos];
	ordered_agents->clinictime[index] = unordered_agents->clinictime[old_pos];
	ordered_agents->workplacetime[index] = unordered_agents->workplacetime[old_pos];
	ordered_agents->outsidetime[index] = unordered_agents->outsidetime[old_pos];
	ordered_agents->age[index] = unordered_agents->age[old_pos];
	ordered_agents->gender[index] = unordered_agents->gender[old_pos];
	ordered_agents->householdsize[index] = unordered_agents->householdsize[old_pos];
	ordered_agents->churchfreq[index] = unordered_agents->churchfreq[old_pos];
	ordered_agents->churchdur[index] = unordered_agents->churchdur[old_pos];
	ordered_agents->transportdur[index] = unordered_agents->transportdur[old_pos];
	ordered_agents->transportday1[index] = unordered_agents->transportday1[old_pos];
	ordered_agents->transportday2[index] = unordered_agents->transportday2[old_pos];
	ordered_agents->household[index] = unordered_agents->household[old_pos];
	ordered_agents->church[index] = unordered_agents->church[old_pos];
	ordered_agents->transport[index] = unordered_agents->transport[old_pos];
	ordered_agents->workplace[index] = unordered_agents->workplace[old_pos];
	ordered_agents->busy[index] = unordered_agents->busy[old_pos];
	ordered_agents->startstep[index] = unordered_agents->startstep[old_pos];
	ordered_agents->location[index] = unordered_agents->location[old_pos];
	ordered_agents->locationid[index] = unordered_agents->locationid[old_pos];
	ordered_agents->hiv[index] = unordered_agents->hiv[old_pos];
	ordered_agents->art[index] = unordered_agents->art[old_pos];
	ordered_agents->activetb[index] = unordered_agents->activetb[old_pos];
	ordered_agents->artday[index] = unordered_agents->artday[old_pos];
	ordered_agents->p[index] = unordered_agents->p[old_pos];
	ordered_agents->q[index] = unordered_agents->q[old_pos];
	ordered_agents->infections[index] = unordered_agents->infections[old_pos];
	ordered_agents->lastinfected[index] = unordered_agents->lastinfected[old_pos];
	ordered_agents->lastinfectedid[index] = unordered_agents->lastinfectedid[old_pos];
	ordered_agents->time_step[index] = unordered_agents->time_step[old_pos];
	ordered_agents->lambda[index] = unordered_agents->lambda[old_pos];
	ordered_agents->timevisiting[index] = unordered_agents->timevisiting[old_pos];
	ordered_agents->bargoing[index] = unordered_agents->bargoing[old_pos];
	ordered_agents->barday[index] = unordered_agents->barday[old_pos];
}

////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dyanamically created TBAssignment agent functions */

/** reset_TBAssignment_scan_input
 * TBAssignment agent reset scan input function
 * @param agents The xmachine_memory_TBAssignment_list agent list
 */
__global__ void reset_TBAssignment_scan_input(xmachine_memory_TBAssignment_list* agents){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	agents->_position[index] = 0;
	agents->_scan_input[index] = 0;
}



/** scatter_TBAssignment_Agents
 * TBAssignment scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_TBAssignment_list agent list destination
 * @param agents_src xmachine_memory_TBAssignment_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void scatter_TBAssignment_Agents(xmachine_memory_TBAssignment_list* agents_dst, xmachine_memory_TBAssignment_list* agents_src, int dst_agent_count, int number_to_scatter){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = agents_src->_scan_input[index];

	//if optional message is to be written. 
	//must check agent is within number to scatter as unused threads may have scan input = 1
	if ((_scan_input == 1)&&(index < number_to_scatter)){
		int output_index = agents_src->_position[index] + dst_agent_count;

		//AoS - xmachine_message_location Un-Coalesced scattered memory write     
        agents_dst->_position[output_index] = output_index;        
		agents_dst->id[output_index] = agents_src->id[index];
	}
}

/** append_TBAssignment_Agents
 * TBAssignment scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_TBAssignment_list agent list destination
 * @param agents_src xmachine_memory_TBAssignment_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void append_TBAssignment_Agents(xmachine_memory_TBAssignment_list* agents_dst, xmachine_memory_TBAssignment_list* agents_src, int dst_agent_count, int number_to_append){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	//must check agent is within number to append as unused threads may have scan input = 1
    if (index < number_to_append){
	    int output_index = index + dst_agent_count;

	    //AoS - xmachine_message_location Un-Coalesced scattered memory write
	    agents_dst->_position[output_index] = output_index;
	    agents_dst->id[output_index] = agents_src->id[index];
    }
}

/** add_TBAssignment_agent
 * Continuous TBAssignment agent add agent function writes agent data to agent swap
 * @param agents xmachine_memory_TBAssignment_list to add agents to 
 * @param id agent variable of type unsigned int
 */
template <int AGENT_TYPE>
__device__ void add_TBAssignment_agent(xmachine_memory_TBAssignment_list* agents, unsigned int id){
	
	int index;
    
    //calculate the agents index in global agent list (depends on agent type)
	if (AGENT_TYPE == DISCRETE_2D){
		int width = (blockDim.x* gridDim.x);
		glm::ivec2 global_position;
		global_position.x = (blockIdx.x*blockDim.x) + threadIdx.x;
		global_position.y = (blockIdx.y*blockDim.y) + threadIdx.y;
		index = global_position.x + (global_position.y* width);
	}else//AGENT_TYPE == CONTINOUS
		index = threadIdx.x + blockIdx.x*blockDim.x;

	//for prefix sum
	agents->_position[index] = 0;
	agents->_scan_input[index] = 1;

	//write data to new buffer
	agents->id[index] = id;

}

//non templated version assumes DISCRETE_2D but works also for CONTINUOUS
__device__ void add_TBAssignment_agent(xmachine_memory_TBAssignment_list* agents, unsigned int id){
    add_TBAssignment_agent<DISCRETE_2D>(agents, id);
}

/** reorder_TBAssignment_agents
 * Continuous TBAssignment agent areorder function used after key value pairs have been sorted
 * @param values sorted index values
 * @param unordered_agents list of unordered agents
 * @ param ordered_agents list used to output ordered agents
 */
__global__ void reorder_TBAssignment_agents(unsigned int* values, xmachine_memory_TBAssignment_list* unordered_agents, xmachine_memory_TBAssignment_list* ordered_agents)
{
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	uint old_pos = values[index];

	//reorder agent data
	ordered_agents->id[index] = unordered_agents->id[old_pos];
}

////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dyanamically created Household agent functions */

/** reset_Household_scan_input
 * Household agent reset scan input function
 * @param agents The xmachine_memory_Household_list agent list
 */
__global__ void reset_Household_scan_input(xmachine_memory_Household_list* agents){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	agents->_position[index] = 0;
	agents->_scan_input[index] = 0;
}



/** scatter_Household_Agents
 * Household scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_Household_list agent list destination
 * @param agents_src xmachine_memory_Household_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void scatter_Household_Agents(xmachine_memory_Household_list* agents_dst, xmachine_memory_Household_list* agents_src, int dst_agent_count, int number_to_scatter){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = agents_src->_scan_input[index];

	//if optional message is to be written. 
	//must check agent is within number to scatter as unused threads may have scan input = 1
	if ((_scan_input == 1)&&(index < number_to_scatter)){
		int output_index = agents_src->_position[index] + dst_agent_count;

		//AoS - xmachine_message_location Un-Coalesced scattered memory write     
        agents_dst->_position[output_index] = output_index;        
		agents_dst->id[output_index] = agents_src->id[index];        
		agents_dst->lambda[output_index] = agents_src->lambda[index];        
		agents_dst->active[output_index] = agents_src->active[index];
	}
}

/** append_Household_Agents
 * Household scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_Household_list agent list destination
 * @param agents_src xmachine_memory_Household_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void append_Household_Agents(xmachine_memory_Household_list* agents_dst, xmachine_memory_Household_list* agents_src, int dst_agent_count, int number_to_append){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	//must check agent is within number to append as unused threads may have scan input = 1
    if (index < number_to_append){
	    int output_index = index + dst_agent_count;

	    //AoS - xmachine_message_location Un-Coalesced scattered memory write
	    agents_dst->_position[output_index] = output_index;
	    agents_dst->id[output_index] = agents_src->id[index];
	    agents_dst->lambda[output_index] = agents_src->lambda[index];
	    agents_dst->active[output_index] = agents_src->active[index];
    }
}

/** add_Household_agent
 * Continuous Household agent add agent function writes agent data to agent swap
 * @param agents xmachine_memory_Household_list to add agents to 
 * @param id agent variable of type unsigned int
 * @param lambda agent variable of type float
 * @param active agent variable of type unsigned int
 */
template <int AGENT_TYPE>
__device__ void add_Household_agent(xmachine_memory_Household_list* agents, unsigned int id, float lambda, unsigned int active){
	
	int index;
    
    //calculate the agents index in global agent list (depends on agent type)
	if (AGENT_TYPE == DISCRETE_2D){
		int width = (blockDim.x* gridDim.x);
		glm::ivec2 global_position;
		global_position.x = (blockIdx.x*blockDim.x) + threadIdx.x;
		global_position.y = (blockIdx.y*blockDim.y) + threadIdx.y;
		index = global_position.x + (global_position.y* width);
	}else//AGENT_TYPE == CONTINOUS
		index = threadIdx.x + blockIdx.x*blockDim.x;

	//for prefix sum
	agents->_position[index] = 0;
	agents->_scan_input[index] = 1;

	//write data to new buffer
	agents->id[index] = id;
	agents->lambda[index] = lambda;
	agents->active[index] = active;

}

//non templated version assumes DISCRETE_2D but works also for CONTINUOUS
__device__ void add_Household_agent(xmachine_memory_Household_list* agents, unsigned int id, float lambda, unsigned int active){
    add_Household_agent<DISCRETE_2D>(agents, id, lambda, active);
}

/** reorder_Household_agents
 * Continuous Household agent areorder function used after key value pairs have been sorted
 * @param values sorted index values
 * @param unordered_agents list of unordered agents
 * @ param ordered_agents list used to output ordered agents
 */
__global__ void reorder_Household_agents(unsigned int* values, xmachine_memory_Household_list* unordered_agents, xmachine_memory_Household_list* ordered_agents)
{
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	uint old_pos = values[index];

	//reorder agent data
	ordered_agents->id[index] = unordered_agents->id[old_pos];
	ordered_agents->lambda[index] = unordered_agents->lambda[old_pos];
	ordered_agents->active[index] = unordered_agents->active[old_pos];
}

////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dyanamically created HouseholdMembership agent functions */

/** reset_HouseholdMembership_scan_input
 * HouseholdMembership agent reset scan input function
 * @param agents The xmachine_memory_HouseholdMembership_list agent list
 */
__global__ void reset_HouseholdMembership_scan_input(xmachine_memory_HouseholdMembership_list* agents){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	agents->_position[index] = 0;
	agents->_scan_input[index] = 0;
}



/** scatter_HouseholdMembership_Agents
 * HouseholdMembership scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_HouseholdMembership_list agent list destination
 * @param agents_src xmachine_memory_HouseholdMembership_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void scatter_HouseholdMembership_Agents(xmachine_memory_HouseholdMembership_list* agents_dst, xmachine_memory_HouseholdMembership_list* agents_src, int dst_agent_count, int number_to_scatter){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = agents_src->_scan_input[index];

	//if optional message is to be written. 
	//must check agent is within number to scatter as unused threads may have scan input = 1
	if ((_scan_input == 1)&&(index < number_to_scatter)){
		int output_index = agents_src->_position[index] + dst_agent_count;

		//AoS - xmachine_message_location Un-Coalesced scattered memory write     
        agents_dst->_position[output_index] = output_index;        
		agents_dst->household_id[output_index] = agents_src->household_id[index];        
		agents_dst->person_id[output_index] = agents_src->person_id[index];        
		agents_dst->household_size[output_index] = agents_src->household_size[index];        
		agents_dst->churchgoing[output_index] = agents_src->churchgoing[index];        
		agents_dst->churchfreq[output_index] = agents_src->churchfreq[index];
	}
}

/** append_HouseholdMembership_Agents
 * HouseholdMembership scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_HouseholdMembership_list agent list destination
 * @param agents_src xmachine_memory_HouseholdMembership_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void append_HouseholdMembership_Agents(xmachine_memory_HouseholdMembership_list* agents_dst, xmachine_memory_HouseholdMembership_list* agents_src, int dst_agent_count, int number_to_append){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	//must check agent is within number to append as unused threads may have scan input = 1
    if (index < number_to_append){
	    int output_index = index + dst_agent_count;

	    //AoS - xmachine_message_location Un-Coalesced scattered memory write
	    agents_dst->_position[output_index] = output_index;
	    agents_dst->household_id[output_index] = agents_src->household_id[index];
	    agents_dst->person_id[output_index] = agents_src->person_id[index];
	    agents_dst->household_size[output_index] = agents_src->household_size[index];
	    agents_dst->churchgoing[output_index] = agents_src->churchgoing[index];
	    agents_dst->churchfreq[output_index] = agents_src->churchfreq[index];
    }
}

/** add_HouseholdMembership_agent
 * Continuous HouseholdMembership agent add agent function writes agent data to agent swap
 * @param agents xmachine_memory_HouseholdMembership_list to add agents to 
 * @param household_id agent variable of type unsigned int
 * @param person_id agent variable of type unsigned int
 * @param household_size agent variable of type unsigned int
 * @param churchgoing agent variable of type unsigned int
 * @param churchfreq agent variable of type unsigned int
 */
template <int AGENT_TYPE>
__device__ void add_HouseholdMembership_agent(xmachine_memory_HouseholdMembership_list* agents, unsigned int household_id, unsigned int person_id, unsigned int household_size, unsigned int churchgoing, unsigned int churchfreq){
	
	int index;
    
    //calculate the agents index in global agent list (depends on agent type)
	if (AGENT_TYPE == DISCRETE_2D){
		int width = (blockDim.x* gridDim.x);
		glm::ivec2 global_position;
		global_position.x = (blockIdx.x*blockDim.x) + threadIdx.x;
		global_position.y = (blockIdx.y*blockDim.y) + threadIdx.y;
		index = global_position.x + (global_position.y* width);
	}else//AGENT_TYPE == CONTINOUS
		index = threadIdx.x + blockIdx.x*blockDim.x;

	//for prefix sum
	agents->_position[index] = 0;
	agents->_scan_input[index] = 1;

	//write data to new buffer
	agents->household_id[index] = household_id;
	agents->person_id[index] = person_id;
	agents->household_size[index] = household_size;
	agents->churchgoing[index] = churchgoing;
	agents->churchfreq[index] = churchfreq;

}

//non templated version assumes DISCRETE_2D but works also for CONTINUOUS
__device__ void add_HouseholdMembership_agent(xmachine_memory_HouseholdMembership_list* agents, unsigned int household_id, unsigned int person_id, unsigned int household_size, unsigned int churchgoing, unsigned int churchfreq){
    add_HouseholdMembership_agent<DISCRETE_2D>(agents, household_id, person_id, household_size, churchgoing, churchfreq);
}

/** reorder_HouseholdMembership_agents
 * Continuous HouseholdMembership agent areorder function used after key value pairs have been sorted
 * @param values sorted index values
 * @param unordered_agents list of unordered agents
 * @ param ordered_agents list used to output ordered agents
 */
__global__ void reorder_HouseholdMembership_agents(unsigned int* values, xmachine_memory_HouseholdMembership_list* unordered_agents, xmachine_memory_HouseholdMembership_list* ordered_agents)
{
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	uint old_pos = values[index];

	//reorder agent data
	ordered_agents->household_id[index] = unordered_agents->household_id[old_pos];
	ordered_agents->person_id[index] = unordered_agents->person_id[old_pos];
	ordered_agents->household_size[index] = unordered_agents->household_size[old_pos];
	ordered_agents->churchgoing[index] = unordered_agents->churchgoing[old_pos];
	ordered_agents->churchfreq[index] = unordered_agents->churchfreq[old_pos];
}

////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dyanamically created Church agent functions */

/** reset_Church_scan_input
 * Church agent reset scan input function
 * @param agents The xmachine_memory_Church_list agent list
 */
__global__ void reset_Church_scan_input(xmachine_memory_Church_list* agents){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	agents->_position[index] = 0;
	agents->_scan_input[index] = 0;
}



/** scatter_Church_Agents
 * Church scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_Church_list agent list destination
 * @param agents_src xmachine_memory_Church_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void scatter_Church_Agents(xmachine_memory_Church_list* agents_dst, xmachine_memory_Church_list* agents_src, int dst_agent_count, int number_to_scatter){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = agents_src->_scan_input[index];

	//if optional message is to be written. 
	//must check agent is within number to scatter as unused threads may have scan input = 1
	if ((_scan_input == 1)&&(index < number_to_scatter)){
		int output_index = agents_src->_position[index] + dst_agent_count;

		//AoS - xmachine_message_location Un-Coalesced scattered memory write     
        agents_dst->_position[output_index] = output_index;        
		agents_dst->id[output_index] = agents_src->id[index];        
		agents_dst->size[output_index] = agents_src->size[index];        
		agents_dst->lambda[output_index] = agents_src->lambda[index];        
		agents_dst->active[output_index] = agents_src->active[index];
	}
}

/** append_Church_Agents
 * Church scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_Church_list agent list destination
 * @param agents_src xmachine_memory_Church_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void append_Church_Agents(xmachine_memory_Church_list* agents_dst, xmachine_memory_Church_list* agents_src, int dst_agent_count, int number_to_append){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	//must check agent is within number to append as unused threads may have scan input = 1
    if (index < number_to_append){
	    int output_index = index + dst_agent_count;

	    //AoS - xmachine_message_location Un-Coalesced scattered memory write
	    agents_dst->_position[output_index] = output_index;
	    agents_dst->id[output_index] = agents_src->id[index];
	    agents_dst->size[output_index] = agents_src->size[index];
	    agents_dst->lambda[output_index] = agents_src->lambda[index];
	    agents_dst->active[output_index] = agents_src->active[index];
    }
}

/** add_Church_agent
 * Continuous Church agent add agent function writes agent data to agent swap
 * @param agents xmachine_memory_Church_list to add agents to 
 * @param id agent variable of type unsigned int
 * @param size agent variable of type unsigned int
 * @param lambda agent variable of type float
 * @param active agent variable of type unsigned int
 */
template <int AGENT_TYPE>
__device__ void add_Church_agent(xmachine_memory_Church_list* agents, unsigned int id, unsigned int size, float lambda, unsigned int active){
	
	int index;
    
    //calculate the agents index in global agent list (depends on agent type)
	if (AGENT_TYPE == DISCRETE_2D){
		int width = (blockDim.x* gridDim.x);
		glm::ivec2 global_position;
		global_position.x = (blockIdx.x*blockDim.x) + threadIdx.x;
		global_position.y = (blockIdx.y*blockDim.y) + threadIdx.y;
		index = global_position.x + (global_position.y* width);
	}else//AGENT_TYPE == CONTINOUS
		index = threadIdx.x + blockIdx.x*blockDim.x;

	//for prefix sum
	agents->_position[index] = 0;
	agents->_scan_input[index] = 1;

	//write data to new buffer
	agents->id[index] = id;
	agents->size[index] = size;
	agents->lambda[index] = lambda;
	agents->active[index] = active;

}

//non templated version assumes DISCRETE_2D but works also for CONTINUOUS
__device__ void add_Church_agent(xmachine_memory_Church_list* agents, unsigned int id, unsigned int size, float lambda, unsigned int active){
    add_Church_agent<DISCRETE_2D>(agents, id, size, lambda, active);
}

/** reorder_Church_agents
 * Continuous Church agent areorder function used after key value pairs have been sorted
 * @param values sorted index values
 * @param unordered_agents list of unordered agents
 * @ param ordered_agents list used to output ordered agents
 */
__global__ void reorder_Church_agents(unsigned int* values, xmachine_memory_Church_list* unordered_agents, xmachine_memory_Church_list* ordered_agents)
{
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	uint old_pos = values[index];

	//reorder agent data
	ordered_agents->id[index] = unordered_agents->id[old_pos];
	ordered_agents->size[index] = unordered_agents->size[old_pos];
	ordered_agents->lambda[index] = unordered_agents->lambda[old_pos];
	ordered_agents->active[index] = unordered_agents->active[old_pos];
}

////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dyanamically created ChurchMembership agent functions */

/** reset_ChurchMembership_scan_input
 * ChurchMembership agent reset scan input function
 * @param agents The xmachine_memory_ChurchMembership_list agent list
 */
__global__ void reset_ChurchMembership_scan_input(xmachine_memory_ChurchMembership_list* agents){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	agents->_position[index] = 0;
	agents->_scan_input[index] = 0;
}



/** scatter_ChurchMembership_Agents
 * ChurchMembership scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_ChurchMembership_list agent list destination
 * @param agents_src xmachine_memory_ChurchMembership_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void scatter_ChurchMembership_Agents(xmachine_memory_ChurchMembership_list* agents_dst, xmachine_memory_ChurchMembership_list* agents_src, int dst_agent_count, int number_to_scatter){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = agents_src->_scan_input[index];

	//if optional message is to be written. 
	//must check agent is within number to scatter as unused threads may have scan input = 1
	if ((_scan_input == 1)&&(index < number_to_scatter)){
		int output_index = agents_src->_position[index] + dst_agent_count;

		//AoS - xmachine_message_location Un-Coalesced scattered memory write     
        agents_dst->_position[output_index] = output_index;        
		agents_dst->church_id[output_index] = agents_src->church_id[index];        
		agents_dst->household_id[output_index] = agents_src->household_id[index];        
		agents_dst->churchdur[output_index] = agents_src->churchdur[index];
	}
}

/** append_ChurchMembership_Agents
 * ChurchMembership scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_ChurchMembership_list agent list destination
 * @param agents_src xmachine_memory_ChurchMembership_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void append_ChurchMembership_Agents(xmachine_memory_ChurchMembership_list* agents_dst, xmachine_memory_ChurchMembership_list* agents_src, int dst_agent_count, int number_to_append){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	//must check agent is within number to append as unused threads may have scan input = 1
    if (index < number_to_append){
	    int output_index = index + dst_agent_count;

	    //AoS - xmachine_message_location Un-Coalesced scattered memory write
	    agents_dst->_position[output_index] = output_index;
	    agents_dst->church_id[output_index] = agents_src->church_id[index];
	    agents_dst->household_id[output_index] = agents_src->household_id[index];
	    agents_dst->churchdur[output_index] = agents_src->churchdur[index];
    }
}

/** add_ChurchMembership_agent
 * Continuous ChurchMembership agent add agent function writes agent data to agent swap
 * @param agents xmachine_memory_ChurchMembership_list to add agents to 
 * @param church_id agent variable of type unsigned int
 * @param household_id agent variable of type unsigned int
 * @param churchdur agent variable of type float
 */
template <int AGENT_TYPE>
__device__ void add_ChurchMembership_agent(xmachine_memory_ChurchMembership_list* agents, unsigned int church_id, unsigned int household_id, float churchdur){
	
	int index;
    
    //calculate the agents index in global agent list (depends on agent type)
	if (AGENT_TYPE == DISCRETE_2D){
		int width = (blockDim.x* gridDim.x);
		glm::ivec2 global_position;
		global_position.x = (blockIdx.x*blockDim.x) + threadIdx.x;
		global_position.y = (blockIdx.y*blockDim.y) + threadIdx.y;
		index = global_position.x + (global_position.y* width);
	}else//AGENT_TYPE == CONTINOUS
		index = threadIdx.x + blockIdx.x*blockDim.x;

	//for prefix sum
	agents->_position[index] = 0;
	agents->_scan_input[index] = 1;

	//write data to new buffer
	agents->church_id[index] = church_id;
	agents->household_id[index] = household_id;
	agents->churchdur[index] = churchdur;

}

//non templated version assumes DISCRETE_2D but works also for CONTINUOUS
__device__ void add_ChurchMembership_agent(xmachine_memory_ChurchMembership_list* agents, unsigned int church_id, unsigned int household_id, float churchdur){
    add_ChurchMembership_agent<DISCRETE_2D>(agents, church_id, household_id, churchdur);
}

/** reorder_ChurchMembership_agents
 * Continuous ChurchMembership agent areorder function used after key value pairs have been sorted
 * @param values sorted index values
 * @param unordered_agents list of unordered agents
 * @ param ordered_agents list used to output ordered agents
 */
__global__ void reorder_ChurchMembership_agents(unsigned int* values, xmachine_memory_ChurchMembership_list* unordered_agents, xmachine_memory_ChurchMembership_list* ordered_agents)
{
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	uint old_pos = values[index];

	//reorder agent data
	ordered_agents->church_id[index] = unordered_agents->church_id[old_pos];
	ordered_agents->household_id[index] = unordered_agents->household_id[old_pos];
	ordered_agents->churchdur[index] = unordered_agents->churchdur[old_pos];
}

////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dyanamically created Transport agent functions */

/** reset_Transport_scan_input
 * Transport agent reset scan input function
 * @param agents The xmachine_memory_Transport_list agent list
 */
__global__ void reset_Transport_scan_input(xmachine_memory_Transport_list* agents){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	agents->_position[index] = 0;
	agents->_scan_input[index] = 0;
}



/** scatter_Transport_Agents
 * Transport scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_Transport_list agent list destination
 * @param agents_src xmachine_memory_Transport_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void scatter_Transport_Agents(xmachine_memory_Transport_list* agents_dst, xmachine_memory_Transport_list* agents_src, int dst_agent_count, int number_to_scatter){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = agents_src->_scan_input[index];

	//if optional message is to be written. 
	//must check agent is within number to scatter as unused threads may have scan input = 1
	if ((_scan_input == 1)&&(index < number_to_scatter)){
		int output_index = agents_src->_position[index] + dst_agent_count;

		//AoS - xmachine_message_location Un-Coalesced scattered memory write     
        agents_dst->_position[output_index] = output_index;        
		agents_dst->id[output_index] = agents_src->id[index];        
		agents_dst->lambda[output_index] = agents_src->lambda[index];        
		agents_dst->active[output_index] = agents_src->active[index];
	}
}

/** append_Transport_Agents
 * Transport scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_Transport_list agent list destination
 * @param agents_src xmachine_memory_Transport_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void append_Transport_Agents(xmachine_memory_Transport_list* agents_dst, xmachine_memory_Transport_list* agents_src, int dst_agent_count, int number_to_append){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	//must check agent is within number to append as unused threads may have scan input = 1
    if (index < number_to_append){
	    int output_index = index + dst_agent_count;

	    //AoS - xmachine_message_location Un-Coalesced scattered memory write
	    agents_dst->_position[output_index] = output_index;
	    agents_dst->id[output_index] = agents_src->id[index];
	    agents_dst->lambda[output_index] = agents_src->lambda[index];
	    agents_dst->active[output_index] = agents_src->active[index];
    }
}

/** add_Transport_agent
 * Continuous Transport agent add agent function writes agent data to agent swap
 * @param agents xmachine_memory_Transport_list to add agents to 
 * @param id agent variable of type unsigned int
 * @param lambda agent variable of type float
 * @param active agent variable of type unsigned int
 */
template <int AGENT_TYPE>
__device__ void add_Transport_agent(xmachine_memory_Transport_list* agents, unsigned int id, float lambda, unsigned int active){
	
	int index;
    
    //calculate the agents index in global agent list (depends on agent type)
	if (AGENT_TYPE == DISCRETE_2D){
		int width = (blockDim.x* gridDim.x);
		glm::ivec2 global_position;
		global_position.x = (blockIdx.x*blockDim.x) + threadIdx.x;
		global_position.y = (blockIdx.y*blockDim.y) + threadIdx.y;
		index = global_position.x + (global_position.y* width);
	}else//AGENT_TYPE == CONTINOUS
		index = threadIdx.x + blockIdx.x*blockDim.x;

	//for prefix sum
	agents->_position[index] = 0;
	agents->_scan_input[index] = 1;

	//write data to new buffer
	agents->id[index] = id;
	agents->lambda[index] = lambda;
	agents->active[index] = active;

}

//non templated version assumes DISCRETE_2D but works also for CONTINUOUS
__device__ void add_Transport_agent(xmachine_memory_Transport_list* agents, unsigned int id, float lambda, unsigned int active){
    add_Transport_agent<DISCRETE_2D>(agents, id, lambda, active);
}

/** reorder_Transport_agents
 * Continuous Transport agent areorder function used after key value pairs have been sorted
 * @param values sorted index values
 * @param unordered_agents list of unordered agents
 * @ param ordered_agents list used to output ordered agents
 */
__global__ void reorder_Transport_agents(unsigned int* values, xmachine_memory_Transport_list* unordered_agents, xmachine_memory_Transport_list* ordered_agents)
{
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	uint old_pos = values[index];

	//reorder agent data
	ordered_agents->id[index] = unordered_agents->id[old_pos];
	ordered_agents->lambda[index] = unordered_agents->lambda[old_pos];
	ordered_agents->active[index] = unordered_agents->active[old_pos];
}

////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dyanamically created TransportMembership agent functions */

/** reset_TransportMembership_scan_input
 * TransportMembership agent reset scan input function
 * @param agents The xmachine_memory_TransportMembership_list agent list
 */
__global__ void reset_TransportMembership_scan_input(xmachine_memory_TransportMembership_list* agents){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	agents->_position[index] = 0;
	agents->_scan_input[index] = 0;
}



/** scatter_TransportMembership_Agents
 * TransportMembership scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_TransportMembership_list agent list destination
 * @param agents_src xmachine_memory_TransportMembership_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void scatter_TransportMembership_Agents(xmachine_memory_TransportMembership_list* agents_dst, xmachine_memory_TransportMembership_list* agents_src, int dst_agent_count, int number_to_scatter){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = agents_src->_scan_input[index];

	//if optional message is to be written. 
	//must check agent is within number to scatter as unused threads may have scan input = 1
	if ((_scan_input == 1)&&(index < number_to_scatter)){
		int output_index = agents_src->_position[index] + dst_agent_count;

		//AoS - xmachine_message_location Un-Coalesced scattered memory write     
        agents_dst->_position[output_index] = output_index;        
		agents_dst->person_id[output_index] = agents_src->person_id[index];        
		agents_dst->transport_id[output_index] = agents_src->transport_id[index];        
		agents_dst->duration[output_index] = agents_src->duration[index];
	}
}

/** append_TransportMembership_Agents
 * TransportMembership scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_TransportMembership_list agent list destination
 * @param agents_src xmachine_memory_TransportMembership_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void append_TransportMembership_Agents(xmachine_memory_TransportMembership_list* agents_dst, xmachine_memory_TransportMembership_list* agents_src, int dst_agent_count, int number_to_append){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	//must check agent is within number to append as unused threads may have scan input = 1
    if (index < number_to_append){
	    int output_index = index + dst_agent_count;

	    //AoS - xmachine_message_location Un-Coalesced scattered memory write
	    agents_dst->_position[output_index] = output_index;
	    agents_dst->person_id[output_index] = agents_src->person_id[index];
	    agents_dst->transport_id[output_index] = agents_src->transport_id[index];
	    agents_dst->duration[output_index] = agents_src->duration[index];
    }
}

/** add_TransportMembership_agent
 * Continuous TransportMembership agent add agent function writes agent data to agent swap
 * @param agents xmachine_memory_TransportMembership_list to add agents to 
 * @param person_id agent variable of type int
 * @param transport_id agent variable of type unsigned int
 * @param duration agent variable of type unsigned int
 */
template <int AGENT_TYPE>
__device__ void add_TransportMembership_agent(xmachine_memory_TransportMembership_list* agents, int person_id, unsigned int transport_id, unsigned int duration){
	
	int index;
    
    //calculate the agents index in global agent list (depends on agent type)
	if (AGENT_TYPE == DISCRETE_2D){
		int width = (blockDim.x* gridDim.x);
		glm::ivec2 global_position;
		global_position.x = (blockIdx.x*blockDim.x) + threadIdx.x;
		global_position.y = (blockIdx.y*blockDim.y) + threadIdx.y;
		index = global_position.x + (global_position.y* width);
	}else//AGENT_TYPE == CONTINOUS
		index = threadIdx.x + blockIdx.x*blockDim.x;

	//for prefix sum
	agents->_position[index] = 0;
	agents->_scan_input[index] = 1;

	//write data to new buffer
	agents->person_id[index] = person_id;
	agents->transport_id[index] = transport_id;
	agents->duration[index] = duration;

}

//non templated version assumes DISCRETE_2D but works also for CONTINUOUS
__device__ void add_TransportMembership_agent(xmachine_memory_TransportMembership_list* agents, int person_id, unsigned int transport_id, unsigned int duration){
    add_TransportMembership_agent<DISCRETE_2D>(agents, person_id, transport_id, duration);
}

/** reorder_TransportMembership_agents
 * Continuous TransportMembership agent areorder function used after key value pairs have been sorted
 * @param values sorted index values
 * @param unordered_agents list of unordered agents
 * @ param ordered_agents list used to output ordered agents
 */
__global__ void reorder_TransportMembership_agents(unsigned int* values, xmachine_memory_TransportMembership_list* unordered_agents, xmachine_memory_TransportMembership_list* ordered_agents)
{
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	uint old_pos = values[index];

	//reorder agent data
	ordered_agents->person_id[index] = unordered_agents->person_id[old_pos];
	ordered_agents->transport_id[index] = unordered_agents->transport_id[old_pos];
	ordered_agents->duration[index] = unordered_agents->duration[old_pos];
}

////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dyanamically created Clinic agent functions */

/** reset_Clinic_scan_input
 * Clinic agent reset scan input function
 * @param agents The xmachine_memory_Clinic_list agent list
 */
__global__ void reset_Clinic_scan_input(xmachine_memory_Clinic_list* agents){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	agents->_position[index] = 0;
	agents->_scan_input[index] = 0;
}



/** scatter_Clinic_Agents
 * Clinic scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_Clinic_list agent list destination
 * @param agents_src xmachine_memory_Clinic_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void scatter_Clinic_Agents(xmachine_memory_Clinic_list* agents_dst, xmachine_memory_Clinic_list* agents_src, int dst_agent_count, int number_to_scatter){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = agents_src->_scan_input[index];

	//if optional message is to be written. 
	//must check agent is within number to scatter as unused threads may have scan input = 1
	if ((_scan_input == 1)&&(index < number_to_scatter)){
		int output_index = agents_src->_position[index] + dst_agent_count;

		//AoS - xmachine_message_location Un-Coalesced scattered memory write     
        agents_dst->_position[output_index] = output_index;        
		agents_dst->id[output_index] = agents_src->id[index];        
		agents_dst->lambda[output_index] = agents_src->lambda[index];
	}
}

/** append_Clinic_Agents
 * Clinic scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_Clinic_list agent list destination
 * @param agents_src xmachine_memory_Clinic_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void append_Clinic_Agents(xmachine_memory_Clinic_list* agents_dst, xmachine_memory_Clinic_list* agents_src, int dst_agent_count, int number_to_append){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	//must check agent is within number to append as unused threads may have scan input = 1
    if (index < number_to_append){
	    int output_index = index + dst_agent_count;

	    //AoS - xmachine_message_location Un-Coalesced scattered memory write
	    agents_dst->_position[output_index] = output_index;
	    agents_dst->id[output_index] = agents_src->id[index];
	    agents_dst->lambda[output_index] = agents_src->lambda[index];
    }
}

/** add_Clinic_agent
 * Continuous Clinic agent add agent function writes agent data to agent swap
 * @param agents xmachine_memory_Clinic_list to add agents to 
 * @param id agent variable of type unsigned int
 * @param lambda agent variable of type float
 */
template <int AGENT_TYPE>
__device__ void add_Clinic_agent(xmachine_memory_Clinic_list* agents, unsigned int id, float lambda){
	
	int index;
    
    //calculate the agents index in global agent list (depends on agent type)
	if (AGENT_TYPE == DISCRETE_2D){
		int width = (blockDim.x* gridDim.x);
		glm::ivec2 global_position;
		global_position.x = (blockIdx.x*blockDim.x) + threadIdx.x;
		global_position.y = (blockIdx.y*blockDim.y) + threadIdx.y;
		index = global_position.x + (global_position.y* width);
	}else//AGENT_TYPE == CONTINOUS
		index = threadIdx.x + blockIdx.x*blockDim.x;

	//for prefix sum
	agents->_position[index] = 0;
	agents->_scan_input[index] = 1;

	//write data to new buffer
	agents->id[index] = id;
	agents->lambda[index] = lambda;

}

//non templated version assumes DISCRETE_2D but works also for CONTINUOUS
__device__ void add_Clinic_agent(xmachine_memory_Clinic_list* agents, unsigned int id, float lambda){
    add_Clinic_agent<DISCRETE_2D>(agents, id, lambda);
}

/** reorder_Clinic_agents
 * Continuous Clinic agent areorder function used after key value pairs have been sorted
 * @param values sorted index values
 * @param unordered_agents list of unordered agents
 * @ param ordered_agents list used to output ordered agents
 */
__global__ void reorder_Clinic_agents(unsigned int* values, xmachine_memory_Clinic_list* unordered_agents, xmachine_memory_Clinic_list* ordered_agents)
{
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	uint old_pos = values[index];

	//reorder agent data
	ordered_agents->id[index] = unordered_agents->id[old_pos];
	ordered_agents->lambda[index] = unordered_agents->lambda[old_pos];
}

////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dyanamically created Workplace agent functions */

/** reset_Workplace_scan_input
 * Workplace agent reset scan input function
 * @param agents The xmachine_memory_Workplace_list agent list
 */
__global__ void reset_Workplace_scan_input(xmachine_memory_Workplace_list* agents){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	agents->_position[index] = 0;
	agents->_scan_input[index] = 0;
}



/** scatter_Workplace_Agents
 * Workplace scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_Workplace_list agent list destination
 * @param agents_src xmachine_memory_Workplace_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void scatter_Workplace_Agents(xmachine_memory_Workplace_list* agents_dst, xmachine_memory_Workplace_list* agents_src, int dst_agent_count, int number_to_scatter){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = agents_src->_scan_input[index];

	//if optional message is to be written. 
	//must check agent is within number to scatter as unused threads may have scan input = 1
	if ((_scan_input == 1)&&(index < number_to_scatter)){
		int output_index = agents_src->_position[index] + dst_agent_count;

		//AoS - xmachine_message_location Un-Coalesced scattered memory write     
        agents_dst->_position[output_index] = output_index;        
		agents_dst->id[output_index] = agents_src->id[index];        
		agents_dst->lambda[output_index] = agents_src->lambda[index];
	}
}

/** append_Workplace_Agents
 * Workplace scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_Workplace_list agent list destination
 * @param agents_src xmachine_memory_Workplace_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void append_Workplace_Agents(xmachine_memory_Workplace_list* agents_dst, xmachine_memory_Workplace_list* agents_src, int dst_agent_count, int number_to_append){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	//must check agent is within number to append as unused threads may have scan input = 1
    if (index < number_to_append){
	    int output_index = index + dst_agent_count;

	    //AoS - xmachine_message_location Un-Coalesced scattered memory write
	    agents_dst->_position[output_index] = output_index;
	    agents_dst->id[output_index] = agents_src->id[index];
	    agents_dst->lambda[output_index] = agents_src->lambda[index];
    }
}

/** add_Workplace_agent
 * Continuous Workplace agent add agent function writes agent data to agent swap
 * @param agents xmachine_memory_Workplace_list to add agents to 
 * @param id agent variable of type unsigned int
 * @param lambda agent variable of type float
 */
template <int AGENT_TYPE>
__device__ void add_Workplace_agent(xmachine_memory_Workplace_list* agents, unsigned int id, float lambda){
	
	int index;
    
    //calculate the agents index in global agent list (depends on agent type)
	if (AGENT_TYPE == DISCRETE_2D){
		int width = (blockDim.x* gridDim.x);
		glm::ivec2 global_position;
		global_position.x = (blockIdx.x*blockDim.x) + threadIdx.x;
		global_position.y = (blockIdx.y*blockDim.y) + threadIdx.y;
		index = global_position.x + (global_position.y* width);
	}else//AGENT_TYPE == CONTINOUS
		index = threadIdx.x + blockIdx.x*blockDim.x;

	//for prefix sum
	agents->_position[index] = 0;
	agents->_scan_input[index] = 1;

	//write data to new buffer
	agents->id[index] = id;
	agents->lambda[index] = lambda;

}

//non templated version assumes DISCRETE_2D but works also for CONTINUOUS
__device__ void add_Workplace_agent(xmachine_memory_Workplace_list* agents, unsigned int id, float lambda){
    add_Workplace_agent<DISCRETE_2D>(agents, id, lambda);
}

/** reorder_Workplace_agents
 * Continuous Workplace agent areorder function used after key value pairs have been sorted
 * @param values sorted index values
 * @param unordered_agents list of unordered agents
 * @ param ordered_agents list used to output ordered agents
 */
__global__ void reorder_Workplace_agents(unsigned int* values, xmachine_memory_Workplace_list* unordered_agents, xmachine_memory_Workplace_list* ordered_agents)
{
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	uint old_pos = values[index];

	//reorder agent data
	ordered_agents->id[index] = unordered_agents->id[old_pos];
	ordered_agents->lambda[index] = unordered_agents->lambda[old_pos];
}

////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dyanamically created WorkplaceMembership agent functions */

/** reset_WorkplaceMembership_scan_input
 * WorkplaceMembership agent reset scan input function
 * @param agents The xmachine_memory_WorkplaceMembership_list agent list
 */
__global__ void reset_WorkplaceMembership_scan_input(xmachine_memory_WorkplaceMembership_list* agents){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	agents->_position[index] = 0;
	agents->_scan_input[index] = 0;
}



/** scatter_WorkplaceMembership_Agents
 * WorkplaceMembership scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_WorkplaceMembership_list agent list destination
 * @param agents_src xmachine_memory_WorkplaceMembership_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void scatter_WorkplaceMembership_Agents(xmachine_memory_WorkplaceMembership_list* agents_dst, xmachine_memory_WorkplaceMembership_list* agents_src, int dst_agent_count, int number_to_scatter){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = agents_src->_scan_input[index];

	//if optional message is to be written. 
	//must check agent is within number to scatter as unused threads may have scan input = 1
	if ((_scan_input == 1)&&(index < number_to_scatter)){
		int output_index = agents_src->_position[index] + dst_agent_count;

		//AoS - xmachine_message_location Un-Coalesced scattered memory write     
        agents_dst->_position[output_index] = output_index;        
		agents_dst->person_id[output_index] = agents_src->person_id[index];        
		agents_dst->workplace_id[output_index] = agents_src->workplace_id[index];
	}
}

/** append_WorkplaceMembership_Agents
 * WorkplaceMembership scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_WorkplaceMembership_list agent list destination
 * @param agents_src xmachine_memory_WorkplaceMembership_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void append_WorkplaceMembership_Agents(xmachine_memory_WorkplaceMembership_list* agents_dst, xmachine_memory_WorkplaceMembership_list* agents_src, int dst_agent_count, int number_to_append){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	//must check agent is within number to append as unused threads may have scan input = 1
    if (index < number_to_append){
	    int output_index = index + dst_agent_count;

	    //AoS - xmachine_message_location Un-Coalesced scattered memory write
	    agents_dst->_position[output_index] = output_index;
	    agents_dst->person_id[output_index] = agents_src->person_id[index];
	    agents_dst->workplace_id[output_index] = agents_src->workplace_id[index];
    }
}

/** add_WorkplaceMembership_agent
 * Continuous WorkplaceMembership agent add agent function writes agent data to agent swap
 * @param agents xmachine_memory_WorkplaceMembership_list to add agents to 
 * @param person_id agent variable of type unsigned int
 * @param workplace_id agent variable of type unsigned int
 */
template <int AGENT_TYPE>
__device__ void add_WorkplaceMembership_agent(xmachine_memory_WorkplaceMembership_list* agents, unsigned int person_id, unsigned int workplace_id){
	
	int index;
    
    //calculate the agents index in global agent list (depends on agent type)
	if (AGENT_TYPE == DISCRETE_2D){
		int width = (blockDim.x* gridDim.x);
		glm::ivec2 global_position;
		global_position.x = (blockIdx.x*blockDim.x) + threadIdx.x;
		global_position.y = (blockIdx.y*blockDim.y) + threadIdx.y;
		index = global_position.x + (global_position.y* width);
	}else//AGENT_TYPE == CONTINOUS
		index = threadIdx.x + blockIdx.x*blockDim.x;

	//for prefix sum
	agents->_position[index] = 0;
	agents->_scan_input[index] = 1;

	//write data to new buffer
	agents->person_id[index] = person_id;
	agents->workplace_id[index] = workplace_id;

}

//non templated version assumes DISCRETE_2D but works also for CONTINUOUS
__device__ void add_WorkplaceMembership_agent(xmachine_memory_WorkplaceMembership_list* agents, unsigned int person_id, unsigned int workplace_id){
    add_WorkplaceMembership_agent<DISCRETE_2D>(agents, person_id, workplace_id);
}

/** reorder_WorkplaceMembership_agents
 * Continuous WorkplaceMembership agent areorder function used after key value pairs have been sorted
 * @param values sorted index values
 * @param unordered_agents list of unordered agents
 * @ param ordered_agents list used to output ordered agents
 */
__global__ void reorder_WorkplaceMembership_agents(unsigned int* values, xmachine_memory_WorkplaceMembership_list* unordered_agents, xmachine_memory_WorkplaceMembership_list* ordered_agents)
{
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	uint old_pos = values[index];

	//reorder agent data
	ordered_agents->person_id[index] = unordered_agents->person_id[old_pos];
	ordered_agents->workplace_id[index] = unordered_agents->workplace_id[old_pos];
}

////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dyanamically created Bar agent functions */

/** reset_Bar_scan_input
 * Bar agent reset scan input function
 * @param agents The xmachine_memory_Bar_list agent list
 */
__global__ void reset_Bar_scan_input(xmachine_memory_Bar_list* agents){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	agents->_position[index] = 0;
	agents->_scan_input[index] = 0;
}



/** scatter_Bar_Agents
 * Bar scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_Bar_list agent list destination
 * @param agents_src xmachine_memory_Bar_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void scatter_Bar_Agents(xmachine_memory_Bar_list* agents_dst, xmachine_memory_Bar_list* agents_src, int dst_agent_count, int number_to_scatter){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = agents_src->_scan_input[index];

	//if optional message is to be written. 
	//must check agent is within number to scatter as unused threads may have scan input = 1
	if ((_scan_input == 1)&&(index < number_to_scatter)){
		int output_index = agents_src->_position[index] + dst_agent_count;

		//AoS - xmachine_message_location Un-Coalesced scattered memory write     
        agents_dst->_position[output_index] = output_index;        
		agents_dst->id[output_index] = agents_src->id[index];        
		agents_dst->lambda[output_index] = agents_src->lambda[index];
	}
}

/** append_Bar_Agents
 * Bar scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_Bar_list agent list destination
 * @param agents_src xmachine_memory_Bar_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void append_Bar_Agents(xmachine_memory_Bar_list* agents_dst, xmachine_memory_Bar_list* agents_src, int dst_agent_count, int number_to_append){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	//must check agent is within number to append as unused threads may have scan input = 1
    if (index < number_to_append){
	    int output_index = index + dst_agent_count;

	    //AoS - xmachine_message_location Un-Coalesced scattered memory write
	    agents_dst->_position[output_index] = output_index;
	    agents_dst->id[output_index] = agents_src->id[index];
	    agents_dst->lambda[output_index] = agents_src->lambda[index];
    }
}

/** add_Bar_agent
 * Continuous Bar agent add agent function writes agent data to agent swap
 * @param agents xmachine_memory_Bar_list to add agents to 
 * @param id agent variable of type unsigned int
 * @param lambda agent variable of type float
 */
template <int AGENT_TYPE>
__device__ void add_Bar_agent(xmachine_memory_Bar_list* agents, unsigned int id, float lambda){
	
	int index;
    
    //calculate the agents index in global agent list (depends on agent type)
	if (AGENT_TYPE == DISCRETE_2D){
		int width = (blockDim.x* gridDim.x);
		glm::ivec2 global_position;
		global_position.x = (blockIdx.x*blockDim.x) + threadIdx.x;
		global_position.y = (blockIdx.y*blockDim.y) + threadIdx.y;
		index = global_position.x + (global_position.y* width);
	}else//AGENT_TYPE == CONTINOUS
		index = threadIdx.x + blockIdx.x*blockDim.x;

	//for prefix sum
	agents->_position[index] = 0;
	agents->_scan_input[index] = 1;

	//write data to new buffer
	agents->id[index] = id;
	agents->lambda[index] = lambda;

}

//non templated version assumes DISCRETE_2D but works also for CONTINUOUS
__device__ void add_Bar_agent(xmachine_memory_Bar_list* agents, unsigned int id, float lambda){
    add_Bar_agent<DISCRETE_2D>(agents, id, lambda);
}

/** reorder_Bar_agents
 * Continuous Bar agent areorder function used after key value pairs have been sorted
 * @param values sorted index values
 * @param unordered_agents list of unordered agents
 * @ param ordered_agents list used to output ordered agents
 */
__global__ void reorder_Bar_agents(unsigned int* values, xmachine_memory_Bar_list* unordered_agents, xmachine_memory_Bar_list* ordered_agents)
{
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	uint old_pos = values[index];

	//reorder agent data
	ordered_agents->id[index] = unordered_agents->id[old_pos];
	ordered_agents->lambda[index] = unordered_agents->lambda[old_pos];
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dyanamically created tb_assignment message functions */


/** add_tb_assignment_message
 * Add non partitioned or spatially partitioned tb_assignment message
 * @param messages xmachine_message_tb_assignment_list message list to add too
 * @param id agent variable of type unsigned int
 */
__device__ void add_tb_assignment_message(xmachine_message_tb_assignment_list* messages, unsigned int id){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x + d_message_tb_assignment_count;

	int _position;
	int _scan_input;

	//decide output position
	if(d_message_tb_assignment_output_type == single_message){
		_position = index; //same as agent position
		_scan_input = 0;
	}else if (d_message_tb_assignment_output_type == optional_message){
		_position = 0;	   //to be calculated using Prefix sum
		_scan_input = 1;
	}

	//AoS - xmachine_message_tb_assignment Coalesced memory write
	messages->_scan_input[index] = _scan_input;	
	messages->_position[index] = _position;
	messages->id[index] = id;

}

/**
 * Scatter non partitioned or spatially partitioned tb_assignment message (for optional messages)
 * @param messages scatter_optional_tb_assignment_messages Sparse xmachine_message_tb_assignment_list message list
 * @param message_swap temp xmachine_message_tb_assignment_list message list to scatter sparse messages to
 */
__global__ void scatter_optional_tb_assignment_messages(xmachine_message_tb_assignment_list* messages, xmachine_message_tb_assignment_list* messages_swap){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = messages_swap->_scan_input[index];

	//if optional message is to be written
	if (_scan_input == 1){
		int output_index = messages_swap->_position[index] + d_message_tb_assignment_count;

		//AoS - xmachine_message_tb_assignment Un-Coalesced scattered memory write
		messages->_position[output_index] = output_index;
		messages->id[output_index] = messages_swap->id[index];				
	}
}

/** reset_tb_assignment_swaps
 * Reset non partitioned or spatially partitioned tb_assignment message swaps (for scattering optional messages)
 * @param message_swap message list to reset _position and _scan_input values back to 0
 */
__global__ void reset_tb_assignment_swaps(xmachine_message_tb_assignment_list* messages_swap){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	messages_swap->_position[index] = 0;
	messages_swap->_scan_input[index] = 0;
}

/* Message functions */

__device__ xmachine_message_tb_assignment* get_first_tb_assignment_message(xmachine_message_tb_assignment_list* messages){

	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];
	
	//wrap size is the number of tiles required to load all messages
	int wrap_size = (ceil((float)d_message_tb_assignment_count/ blockDim.x)* blockDim.x);

	//if no messages then return a null pointer (false)
	if (wrap_size == 0)
		return nullptr;

	//global thread index
	int global_index = (blockIdx.x*blockDim.x) + threadIdx.x;

	//global thread index
	int index = WRAP(global_index, wrap_size);

	//SoA to AoS - xmachine_message_tb_assignment Coalesced memory read
	xmachine_message_tb_assignment temp_message;
	temp_message._position = messages->_position[index];
	temp_message.id = messages->id[index];

	//AoS to shared memory
	int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_tb_assignment));
	xmachine_message_tb_assignment* sm_message = ((xmachine_message_tb_assignment*)&message_share[message_index]);
	sm_message[0] = temp_message;

	__syncthreads();

  //HACK FOR 64 bit addressing issue in sm
	return ((xmachine_message_tb_assignment*)&message_share[d_SM_START]);
}

__device__ xmachine_message_tb_assignment* get_next_tb_assignment_message(xmachine_message_tb_assignment* message, xmachine_message_tb_assignment_list* messages){

	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];
	
	//wrap size is the number of tiles required to load all messages
	int wrap_size = ceil((float)d_message_tb_assignment_count/ blockDim.x)*blockDim.x;

	int i = WRAP((message->_position + 1),wrap_size);

	//If end of messages (last message not multiple of gridsize) go to 0 index
	if (i >= d_message_tb_assignment_count)
		i = 0;

	//Check if back to start position of first message
	if (i == WRAP((blockDim.x* blockIdx.x), wrap_size))
		return nullptr;

	int tile = floor((float)i/(blockDim.x)); //tile is round down position over blockDim
	i = i % blockDim.x;						 //mod i for shared memory index

	//if count == Block Size load next tile int shared memory values
	if (i == 0){
		__syncthreads();					//make sure we don't change shared memory until all threads are here (important for emu-debug mode)
		
		//SoA to AoS - xmachine_message_tb_assignment Coalesced memory read
		int index = (tile* blockDim.x) + threadIdx.x;
		xmachine_message_tb_assignment temp_message;
		temp_message._position = messages->_position[index];
		temp_message.id = messages->id[index];

		//AoS to shared memory
		int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_tb_assignment));
		xmachine_message_tb_assignment* sm_message = ((xmachine_message_tb_assignment*)&message_share[message_index]);
		sm_message[0] = temp_message;

		__syncthreads();					//make sure we don't start returning messages until all threads have updated shared memory
	}

	int message_index = SHARE_INDEX(i, sizeof(xmachine_message_tb_assignment));
	return ((xmachine_message_tb_assignment*)&message_share[message_index]);
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dyanamically created household_membership message functions */


/** add_household_membership_message
 * Add non partitioned or spatially partitioned household_membership message
 * @param messages xmachine_message_household_membership_list message list to add too
 * @param household_id agent variable of type unsigned int
 * @param person_id agent variable of type unsigned int
 * @param household_size agent variable of type unsigned int
 * @param church_id agent variable of type unsigned int
 * @param churchfreq agent variable of type unsigned int
 * @param churchdur agent variable of type float
 */
__device__ void add_household_membership_message(xmachine_message_household_membership_list* messages, unsigned int household_id, unsigned int person_id, unsigned int household_size, unsigned int church_id, unsigned int churchfreq, float churchdur){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x + d_message_household_membership_count;

	int _position;
	int _scan_input;

	//decide output position
	if(d_message_household_membership_output_type == single_message){
		_position = index; //same as agent position
		_scan_input = 0;
	}else if (d_message_household_membership_output_type == optional_message){
		_position = 0;	   //to be calculated using Prefix sum
		_scan_input = 1;
	}

	//AoS - xmachine_message_household_membership Coalesced memory write
	messages->_scan_input[index] = _scan_input;	
	messages->_position[index] = _position;
	messages->household_id[index] = household_id;
	messages->person_id[index] = person_id;
	messages->household_size[index] = household_size;
	messages->church_id[index] = church_id;
	messages->churchfreq[index] = churchfreq;
	messages->churchdur[index] = churchdur;

}

/**
 * Scatter non partitioned or spatially partitioned household_membership message (for optional messages)
 * @param messages scatter_optional_household_membership_messages Sparse xmachine_message_household_membership_list message list
 * @param message_swap temp xmachine_message_household_membership_list message list to scatter sparse messages to
 */
__global__ void scatter_optional_household_membership_messages(xmachine_message_household_membership_list* messages, xmachine_message_household_membership_list* messages_swap){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = messages_swap->_scan_input[index];

	//if optional message is to be written
	if (_scan_input == 1){
		int output_index = messages_swap->_position[index] + d_message_household_membership_count;

		//AoS - xmachine_message_household_membership Un-Coalesced scattered memory write
		messages->_position[output_index] = output_index;
		messages->household_id[output_index] = messages_swap->household_id[index];
		messages->person_id[output_index] = messages_swap->person_id[index];
		messages->household_size[output_index] = messages_swap->household_size[index];
		messages->church_id[output_index] = messages_swap->church_id[index];
		messages->churchfreq[output_index] = messages_swap->churchfreq[index];
		messages->churchdur[output_index] = messages_swap->churchdur[index];				
	}
}

/** reset_household_membership_swaps
 * Reset non partitioned or spatially partitioned household_membership message swaps (for scattering optional messages)
 * @param message_swap message list to reset _position and _scan_input values back to 0
 */
__global__ void reset_household_membership_swaps(xmachine_message_household_membership_list* messages_swap){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	messages_swap->_position[index] = 0;
	messages_swap->_scan_input[index] = 0;
}

/* Message functions */

__device__ xmachine_message_household_membership* get_first_household_membership_message(xmachine_message_household_membership_list* messages){

	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];
	
	//wrap size is the number of tiles required to load all messages
	int wrap_size = (ceil((float)d_message_household_membership_count/ blockDim.x)* blockDim.x);

	//if no messages then return a null pointer (false)
	if (wrap_size == 0)
		return nullptr;

	//global thread index
	int global_index = (blockIdx.x*blockDim.x) + threadIdx.x;

	//global thread index
	int index = WRAP(global_index, wrap_size);

	//SoA to AoS - xmachine_message_household_membership Coalesced memory read
	xmachine_message_household_membership temp_message;
	temp_message._position = messages->_position[index];
	temp_message.household_id = messages->household_id[index];
	temp_message.person_id = messages->person_id[index];
	temp_message.household_size = messages->household_size[index];
	temp_message.church_id = messages->church_id[index];
	temp_message.churchfreq = messages->churchfreq[index];
	temp_message.churchdur = messages->churchdur[index];

	//AoS to shared memory
	int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_household_membership));
	xmachine_message_household_membership* sm_message = ((xmachine_message_household_membership*)&message_share[message_index]);
	sm_message[0] = temp_message;

	__syncthreads();

  //HACK FOR 64 bit addressing issue in sm
	return ((xmachine_message_household_membership*)&message_share[d_SM_START]);
}

__device__ xmachine_message_household_membership* get_next_household_membership_message(xmachine_message_household_membership* message, xmachine_message_household_membership_list* messages){

	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];
	
	//wrap size is the number of tiles required to load all messages
	int wrap_size = ceil((float)d_message_household_membership_count/ blockDim.x)*blockDim.x;

	int i = WRAP((message->_position + 1),wrap_size);

	//If end of messages (last message not multiple of gridsize) go to 0 index
	if (i >= d_message_household_membership_count)
		i = 0;

	//Check if back to start position of first message
	if (i == WRAP((blockDim.x* blockIdx.x), wrap_size))
		return nullptr;

	int tile = floor((float)i/(blockDim.x)); //tile is round down position over blockDim
	i = i % blockDim.x;						 //mod i for shared memory index

	//if count == Block Size load next tile int shared memory values
	if (i == 0){
		__syncthreads();					//make sure we don't change shared memory until all threads are here (important for emu-debug mode)
		
		//SoA to AoS - xmachine_message_household_membership Coalesced memory read
		int index = (tile* blockDim.x) + threadIdx.x;
		xmachine_message_household_membership temp_message;
		temp_message._position = messages->_position[index];
		temp_message.household_id = messages->household_id[index];
		temp_message.person_id = messages->person_id[index];
		temp_message.household_size = messages->household_size[index];
		temp_message.church_id = messages->church_id[index];
		temp_message.churchfreq = messages->churchfreq[index];
		temp_message.churchdur = messages->churchdur[index];

		//AoS to shared memory
		int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_household_membership));
		xmachine_message_household_membership* sm_message = ((xmachine_message_household_membership*)&message_share[message_index]);
		sm_message[0] = temp_message;

		__syncthreads();					//make sure we don't start returning messages until all threads have updated shared memory
	}

	int message_index = SHARE_INDEX(i, sizeof(xmachine_message_household_membership));
	return ((xmachine_message_household_membership*)&message_share[message_index]);
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dyanamically created church_membership message functions */


/** add_church_membership_message
 * Add non partitioned or spatially partitioned church_membership message
 * @param messages xmachine_message_church_membership_list message list to add too
 * @param church_id agent variable of type unsigned int
 * @param household_id agent variable of type unsigned int
 * @param churchdur agent variable of type float
 */
__device__ void add_church_membership_message(xmachine_message_church_membership_list* messages, unsigned int church_id, unsigned int household_id, float churchdur){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x + d_message_church_membership_count;

	int _position;
	int _scan_input;

	//decide output position
	if(d_message_church_membership_output_type == single_message){
		_position = index; //same as agent position
		_scan_input = 0;
	}else if (d_message_church_membership_output_type == optional_message){
		_position = 0;	   //to be calculated using Prefix sum
		_scan_input = 1;
	}

	//AoS - xmachine_message_church_membership Coalesced memory write
	messages->_scan_input[index] = _scan_input;	
	messages->_position[index] = _position;
	messages->church_id[index] = church_id;
	messages->household_id[index] = household_id;
	messages->churchdur[index] = churchdur;

}

/**
 * Scatter non partitioned or spatially partitioned church_membership message (for optional messages)
 * @param messages scatter_optional_church_membership_messages Sparse xmachine_message_church_membership_list message list
 * @param message_swap temp xmachine_message_church_membership_list message list to scatter sparse messages to
 */
__global__ void scatter_optional_church_membership_messages(xmachine_message_church_membership_list* messages, xmachine_message_church_membership_list* messages_swap){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = messages_swap->_scan_input[index];

	//if optional message is to be written
	if (_scan_input == 1){
		int output_index = messages_swap->_position[index] + d_message_church_membership_count;

		//AoS - xmachine_message_church_membership Un-Coalesced scattered memory write
		messages->_position[output_index] = output_index;
		messages->church_id[output_index] = messages_swap->church_id[index];
		messages->household_id[output_index] = messages_swap->household_id[index];
		messages->churchdur[output_index] = messages_swap->churchdur[index];				
	}
}

/** reset_church_membership_swaps
 * Reset non partitioned or spatially partitioned church_membership message swaps (for scattering optional messages)
 * @param message_swap message list to reset _position and _scan_input values back to 0
 */
__global__ void reset_church_membership_swaps(xmachine_message_church_membership_list* messages_swap){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	messages_swap->_position[index] = 0;
	messages_swap->_scan_input[index] = 0;
}

/* Message functions */

__device__ xmachine_message_church_membership* get_first_church_membership_message(xmachine_message_church_membership_list* messages){

	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];
	
	//wrap size is the number of tiles required to load all messages
	int wrap_size = (ceil((float)d_message_church_membership_count/ blockDim.x)* blockDim.x);

	//if no messages then return a null pointer (false)
	if (wrap_size == 0)
		return nullptr;

	//global thread index
	int global_index = (blockIdx.x*blockDim.x) + threadIdx.x;

	//global thread index
	int index = WRAP(global_index, wrap_size);

	//SoA to AoS - xmachine_message_church_membership Coalesced memory read
	xmachine_message_church_membership temp_message;
	temp_message._position = messages->_position[index];
	temp_message.church_id = messages->church_id[index];
	temp_message.household_id = messages->household_id[index];
	temp_message.churchdur = messages->churchdur[index];

	//AoS to shared memory
	int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_church_membership));
	xmachine_message_church_membership* sm_message = ((xmachine_message_church_membership*)&message_share[message_index]);
	sm_message[0] = temp_message;

	__syncthreads();

  //HACK FOR 64 bit addressing issue in sm
	return ((xmachine_message_church_membership*)&message_share[d_SM_START]);
}

__device__ xmachine_message_church_membership* get_next_church_membership_message(xmachine_message_church_membership* message, xmachine_message_church_membership_list* messages){

	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];
	
	//wrap size is the number of tiles required to load all messages
	int wrap_size = ceil((float)d_message_church_membership_count/ blockDim.x)*blockDim.x;

	int i = WRAP((message->_position + 1),wrap_size);

	//If end of messages (last message not multiple of gridsize) go to 0 index
	if (i >= d_message_church_membership_count)
		i = 0;

	//Check if back to start position of first message
	if (i == WRAP((blockDim.x* blockIdx.x), wrap_size))
		return nullptr;

	int tile = floor((float)i/(blockDim.x)); //tile is round down position over blockDim
	i = i % blockDim.x;						 //mod i for shared memory index

	//if count == Block Size load next tile int shared memory values
	if (i == 0){
		__syncthreads();					//make sure we don't change shared memory until all threads are here (important for emu-debug mode)
		
		//SoA to AoS - xmachine_message_church_membership Coalesced memory read
		int index = (tile* blockDim.x) + threadIdx.x;
		xmachine_message_church_membership temp_message;
		temp_message._position = messages->_position[index];
		temp_message.church_id = messages->church_id[index];
		temp_message.household_id = messages->household_id[index];
		temp_message.churchdur = messages->churchdur[index];

		//AoS to shared memory
		int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_church_membership));
		xmachine_message_church_membership* sm_message = ((xmachine_message_church_membership*)&message_share[message_index]);
		sm_message[0] = temp_message;

		__syncthreads();					//make sure we don't start returning messages until all threads have updated shared memory
	}

	int message_index = SHARE_INDEX(i, sizeof(xmachine_message_church_membership));
	return ((xmachine_message_church_membership*)&message_share[message_index]);
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dyanamically created transport_membership message functions */


/** add_transport_membership_message
 * Add non partitioned or spatially partitioned transport_membership message
 * @param messages xmachine_message_transport_membership_list message list to add too
 * @param person_id agent variable of type unsigned int
 * @param transport_id agent variable of type unsigned int
 * @param duration agent variable of type unsigned int
 */
__device__ void add_transport_membership_message(xmachine_message_transport_membership_list* messages, unsigned int person_id, unsigned int transport_id, unsigned int duration){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x + d_message_transport_membership_count;

	int _position;
	int _scan_input;

	//decide output position
	if(d_message_transport_membership_output_type == single_message){
		_position = index; //same as agent position
		_scan_input = 0;
	}else if (d_message_transport_membership_output_type == optional_message){
		_position = 0;	   //to be calculated using Prefix sum
		_scan_input = 1;
	}

	//AoS - xmachine_message_transport_membership Coalesced memory write
	messages->_scan_input[index] = _scan_input;	
	messages->_position[index] = _position;
	messages->person_id[index] = person_id;
	messages->transport_id[index] = transport_id;
	messages->duration[index] = duration;

}

/**
 * Scatter non partitioned or spatially partitioned transport_membership message (for optional messages)
 * @param messages scatter_optional_transport_membership_messages Sparse xmachine_message_transport_membership_list message list
 * @param message_swap temp xmachine_message_transport_membership_list message list to scatter sparse messages to
 */
__global__ void scatter_optional_transport_membership_messages(xmachine_message_transport_membership_list* messages, xmachine_message_transport_membership_list* messages_swap){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = messages_swap->_scan_input[index];

	//if optional message is to be written
	if (_scan_input == 1){
		int output_index = messages_swap->_position[index] + d_message_transport_membership_count;

		//AoS - xmachine_message_transport_membership Un-Coalesced scattered memory write
		messages->_position[output_index] = output_index;
		messages->person_id[output_index] = messages_swap->person_id[index];
		messages->transport_id[output_index] = messages_swap->transport_id[index];
		messages->duration[output_index] = messages_swap->duration[index];				
	}
}

/** reset_transport_membership_swaps
 * Reset non partitioned or spatially partitioned transport_membership message swaps (for scattering optional messages)
 * @param message_swap message list to reset _position and _scan_input values back to 0
 */
__global__ void reset_transport_membership_swaps(xmachine_message_transport_membership_list* messages_swap){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	messages_swap->_position[index] = 0;
	messages_swap->_scan_input[index] = 0;
}

/* Message functions */

__device__ xmachine_message_transport_membership* get_first_transport_membership_message(xmachine_message_transport_membership_list* messages){

	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];
	
	//wrap size is the number of tiles required to load all messages
	int wrap_size = (ceil((float)d_message_transport_membership_count/ blockDim.x)* blockDim.x);

	//if no messages then return a null pointer (false)
	if (wrap_size == 0)
		return nullptr;

	//global thread index
	int global_index = (blockIdx.x*blockDim.x) + threadIdx.x;

	//global thread index
	int index = WRAP(global_index, wrap_size);

	//SoA to AoS - xmachine_message_transport_membership Coalesced memory read
	xmachine_message_transport_membership temp_message;
	temp_message._position = messages->_position[index];
	temp_message.person_id = messages->person_id[index];
	temp_message.transport_id = messages->transport_id[index];
	temp_message.duration = messages->duration[index];

	//AoS to shared memory
	int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_transport_membership));
	xmachine_message_transport_membership* sm_message = ((xmachine_message_transport_membership*)&message_share[message_index]);
	sm_message[0] = temp_message;

	__syncthreads();

  //HACK FOR 64 bit addressing issue in sm
	return ((xmachine_message_transport_membership*)&message_share[d_SM_START]);
}

__device__ xmachine_message_transport_membership* get_next_transport_membership_message(xmachine_message_transport_membership* message, xmachine_message_transport_membership_list* messages){

	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];
	
	//wrap size is the number of tiles required to load all messages
	int wrap_size = ceil((float)d_message_transport_membership_count/ blockDim.x)*blockDim.x;

	int i = WRAP((message->_position + 1),wrap_size);

	//If end of messages (last message not multiple of gridsize) go to 0 index
	if (i >= d_message_transport_membership_count)
		i = 0;

	//Check if back to start position of first message
	if (i == WRAP((blockDim.x* blockIdx.x), wrap_size))
		return nullptr;

	int tile = floor((float)i/(blockDim.x)); //tile is round down position over blockDim
	i = i % blockDim.x;						 //mod i for shared memory index

	//if count == Block Size load next tile int shared memory values
	if (i == 0){
		__syncthreads();					//make sure we don't change shared memory until all threads are here (important for emu-debug mode)
		
		//SoA to AoS - xmachine_message_transport_membership Coalesced memory read
		int index = (tile* blockDim.x) + threadIdx.x;
		xmachine_message_transport_membership temp_message;
		temp_message._position = messages->_position[index];
		temp_message.person_id = messages->person_id[index];
		temp_message.transport_id = messages->transport_id[index];
		temp_message.duration = messages->duration[index];

		//AoS to shared memory
		int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_transport_membership));
		xmachine_message_transport_membership* sm_message = ((xmachine_message_transport_membership*)&message_share[message_index]);
		sm_message[0] = temp_message;

		__syncthreads();					//make sure we don't start returning messages until all threads have updated shared memory
	}

	int message_index = SHARE_INDEX(i, sizeof(xmachine_message_transport_membership));
	return ((xmachine_message_transport_membership*)&message_share[message_index]);
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dyanamically created workplace_membership message functions */


/** add_workplace_membership_message
 * Add non partitioned or spatially partitioned workplace_membership message
 * @param messages xmachine_message_workplace_membership_list message list to add too
 * @param person_id agent variable of type unsigned int
 * @param workplace_id agent variable of type unsigned int
 */
__device__ void add_workplace_membership_message(xmachine_message_workplace_membership_list* messages, unsigned int person_id, unsigned int workplace_id){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x + d_message_workplace_membership_count;

	int _position;
	int _scan_input;

	//decide output position
	if(d_message_workplace_membership_output_type == single_message){
		_position = index; //same as agent position
		_scan_input = 0;
	}else if (d_message_workplace_membership_output_type == optional_message){
		_position = 0;	   //to be calculated using Prefix sum
		_scan_input = 1;
	}

	//AoS - xmachine_message_workplace_membership Coalesced memory write
	messages->_scan_input[index] = _scan_input;	
	messages->_position[index] = _position;
	messages->person_id[index] = person_id;
	messages->workplace_id[index] = workplace_id;

}

/**
 * Scatter non partitioned or spatially partitioned workplace_membership message (for optional messages)
 * @param messages scatter_optional_workplace_membership_messages Sparse xmachine_message_workplace_membership_list message list
 * @param message_swap temp xmachine_message_workplace_membership_list message list to scatter sparse messages to
 */
__global__ void scatter_optional_workplace_membership_messages(xmachine_message_workplace_membership_list* messages, xmachine_message_workplace_membership_list* messages_swap){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = messages_swap->_scan_input[index];

	//if optional message is to be written
	if (_scan_input == 1){
		int output_index = messages_swap->_position[index] + d_message_workplace_membership_count;

		//AoS - xmachine_message_workplace_membership Un-Coalesced scattered memory write
		messages->_position[output_index] = output_index;
		messages->person_id[output_index] = messages_swap->person_id[index];
		messages->workplace_id[output_index] = messages_swap->workplace_id[index];				
	}
}

/** reset_workplace_membership_swaps
 * Reset non partitioned or spatially partitioned workplace_membership message swaps (for scattering optional messages)
 * @param message_swap message list to reset _position and _scan_input values back to 0
 */
__global__ void reset_workplace_membership_swaps(xmachine_message_workplace_membership_list* messages_swap){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	messages_swap->_position[index] = 0;
	messages_swap->_scan_input[index] = 0;
}

/* Message functions */

__device__ xmachine_message_workplace_membership* get_first_workplace_membership_message(xmachine_message_workplace_membership_list* messages){

	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];
	
	//wrap size is the number of tiles required to load all messages
	int wrap_size = (ceil((float)d_message_workplace_membership_count/ blockDim.x)* blockDim.x);

	//if no messages then return a null pointer (false)
	if (wrap_size == 0)
		return nullptr;

	//global thread index
	int global_index = (blockIdx.x*blockDim.x) + threadIdx.x;

	//global thread index
	int index = WRAP(global_index, wrap_size);

	//SoA to AoS - xmachine_message_workplace_membership Coalesced memory read
	xmachine_message_workplace_membership temp_message;
	temp_message._position = messages->_position[index];
	temp_message.person_id = messages->person_id[index];
	temp_message.workplace_id = messages->workplace_id[index];

	//AoS to shared memory
	int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_workplace_membership));
	xmachine_message_workplace_membership* sm_message = ((xmachine_message_workplace_membership*)&message_share[message_index]);
	sm_message[0] = temp_message;

	__syncthreads();

  //HACK FOR 64 bit addressing issue in sm
	return ((xmachine_message_workplace_membership*)&message_share[d_SM_START]);
}

__device__ xmachine_message_workplace_membership* get_next_workplace_membership_message(xmachine_message_workplace_membership* message, xmachine_message_workplace_membership_list* messages){

	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];
	
	//wrap size is the number of tiles required to load all messages
	int wrap_size = ceil((float)d_message_workplace_membership_count/ blockDim.x)*blockDim.x;

	int i = WRAP((message->_position + 1),wrap_size);

	//If end of messages (last message not multiple of gridsize) go to 0 index
	if (i >= d_message_workplace_membership_count)
		i = 0;

	//Check if back to start position of first message
	if (i == WRAP((blockDim.x* blockIdx.x), wrap_size))
		return nullptr;

	int tile = floor((float)i/(blockDim.x)); //tile is round down position over blockDim
	i = i % blockDim.x;						 //mod i for shared memory index

	//if count == Block Size load next tile int shared memory values
	if (i == 0){
		__syncthreads();					//make sure we don't change shared memory until all threads are here (important for emu-debug mode)
		
		//SoA to AoS - xmachine_message_workplace_membership Coalesced memory read
		int index = (tile* blockDim.x) + threadIdx.x;
		xmachine_message_workplace_membership temp_message;
		temp_message._position = messages->_position[index];
		temp_message.person_id = messages->person_id[index];
		temp_message.workplace_id = messages->workplace_id[index];

		//AoS to shared memory
		int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_workplace_membership));
		xmachine_message_workplace_membership* sm_message = ((xmachine_message_workplace_membership*)&message_share[message_index]);
		sm_message[0] = temp_message;

		__syncthreads();					//make sure we don't start returning messages until all threads have updated shared memory
	}

	int message_index = SHARE_INDEX(i, sizeof(xmachine_message_workplace_membership));
	return ((xmachine_message_workplace_membership*)&message_share[message_index]);
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dyanamically created location message functions */


/** add_location_message
 * Add non partitioned or spatially partitioned location message
 * @param messages xmachine_message_location_list message list to add too
 * @param person_id agent variable of type unsigned int
 * @param location agent variable of type unsigned int
 * @param locationid agent variable of type unsigned int
 * @param p agent variable of type float
 * @param q agent variable of type float
 */
__device__ void add_location_message(xmachine_message_location_list* messages, unsigned int person_id, unsigned int location, unsigned int locationid, float p, float q){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x + d_message_location_count;

	int _position;
	int _scan_input;

	//decide output position
	if(d_message_location_output_type == single_message){
		_position = index; //same as agent position
		_scan_input = 0;
	}else if (d_message_location_output_type == optional_message){
		_position = 0;	   //to be calculated using Prefix sum
		_scan_input = 1;
	}

	//AoS - xmachine_message_location Coalesced memory write
	messages->_scan_input[index] = _scan_input;	
	messages->_position[index] = _position;
	messages->person_id[index] = person_id;
	messages->location[index] = location;
	messages->locationid[index] = locationid;
	messages->p[index] = p;
	messages->q[index] = q;

}

/**
 * Scatter non partitioned or spatially partitioned location message (for optional messages)
 * @param messages scatter_optional_location_messages Sparse xmachine_message_location_list message list
 * @param message_swap temp xmachine_message_location_list message list to scatter sparse messages to
 */
__global__ void scatter_optional_location_messages(xmachine_message_location_list* messages, xmachine_message_location_list* messages_swap){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = messages_swap->_scan_input[index];

	//if optional message is to be written
	if (_scan_input == 1){
		int output_index = messages_swap->_position[index] + d_message_location_count;

		//AoS - xmachine_message_location Un-Coalesced scattered memory write
		messages->_position[output_index] = output_index;
		messages->person_id[output_index] = messages_swap->person_id[index];
		messages->location[output_index] = messages_swap->location[index];
		messages->locationid[output_index] = messages_swap->locationid[index];
		messages->p[output_index] = messages_swap->p[index];
		messages->q[output_index] = messages_swap->q[index];				
	}
}

/** reset_location_swaps
 * Reset non partitioned or spatially partitioned location message swaps (for scattering optional messages)
 * @param message_swap message list to reset _position and _scan_input values back to 0
 */
__global__ void reset_location_swaps(xmachine_message_location_list* messages_swap){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	messages_swap->_position[index] = 0;
	messages_swap->_scan_input[index] = 0;
}

/* Message functions */

__device__ xmachine_message_location* get_first_location_message(xmachine_message_location_list* messages){

	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];
	
	//wrap size is the number of tiles required to load all messages
	int wrap_size = (ceil((float)d_message_location_count/ blockDim.x)* blockDim.x);

	//if no messages then return a null pointer (false)
	if (wrap_size == 0)
		return nullptr;

	//global thread index
	int global_index = (blockIdx.x*blockDim.x) + threadIdx.x;

	//global thread index
	int index = WRAP(global_index, wrap_size);

	//SoA to AoS - xmachine_message_location Coalesced memory read
	xmachine_message_location temp_message;
	temp_message._position = messages->_position[index];
	temp_message.person_id = messages->person_id[index];
	temp_message.location = messages->location[index];
	temp_message.locationid = messages->locationid[index];
	temp_message.p = messages->p[index];
	temp_message.q = messages->q[index];

	//AoS to shared memory
	int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_location));
	xmachine_message_location* sm_message = ((xmachine_message_location*)&message_share[message_index]);
	sm_message[0] = temp_message;

	__syncthreads();

  //HACK FOR 64 bit addressing issue in sm
	return ((xmachine_message_location*)&message_share[d_SM_START]);
}

__device__ xmachine_message_location* get_next_location_message(xmachine_message_location* message, xmachine_message_location_list* messages){

	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];
	
	//wrap size is the number of tiles required to load all messages
	int wrap_size = ceil((float)d_message_location_count/ blockDim.x)*blockDim.x;

	int i = WRAP((message->_position + 1),wrap_size);

	//If end of messages (last message not multiple of gridsize) go to 0 index
	if (i >= d_message_location_count)
		i = 0;

	//Check if back to start position of first message
	if (i == WRAP((blockDim.x* blockIdx.x), wrap_size))
		return nullptr;

	int tile = floor((float)i/(blockDim.x)); //tile is round down position over blockDim
	i = i % blockDim.x;						 //mod i for shared memory index

	//if count == Block Size load next tile int shared memory values
	if (i == 0){
		__syncthreads();					//make sure we don't change shared memory until all threads are here (important for emu-debug mode)
		
		//SoA to AoS - xmachine_message_location Coalesced memory read
		int index = (tile* blockDim.x) + threadIdx.x;
		xmachine_message_location temp_message;
		temp_message._position = messages->_position[index];
		temp_message.person_id = messages->person_id[index];
		temp_message.location = messages->location[index];
		temp_message.locationid = messages->locationid[index];
		temp_message.p = messages->p[index];
		temp_message.q = messages->q[index];

		//AoS to shared memory
		int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_location));
		xmachine_message_location* sm_message = ((xmachine_message_location*)&message_share[message_index]);
		sm_message[0] = temp_message;

		__syncthreads();					//make sure we don't start returning messages until all threads have updated shared memory
	}

	int message_index = SHARE_INDEX(i, sizeof(xmachine_message_location));
	return ((xmachine_message_location*)&message_share[message_index]);
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dyanamically created infection message functions */


/** add_infection_message
 * Add non partitioned or spatially partitioned infection message
 * @param messages xmachine_message_infection_list message list to add too
 * @param location agent variable of type unsigned int
 * @param locationid agent variable of type unsigned int
 * @param lambda agent variable of type float
 */
__device__ void add_infection_message(xmachine_message_infection_list* messages, unsigned int location, unsigned int locationid, float lambda){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x + d_message_infection_count;

	int _position;
	int _scan_input;

	//decide output position
	if(d_message_infection_output_type == single_message){
		_position = index; //same as agent position
		_scan_input = 0;
	}else if (d_message_infection_output_type == optional_message){
		_position = 0;	   //to be calculated using Prefix sum
		_scan_input = 1;
	}

	//AoS - xmachine_message_infection Coalesced memory write
	messages->_scan_input[index] = _scan_input;	
	messages->_position[index] = _position;
	messages->location[index] = location;
	messages->locationid[index] = locationid;
	messages->lambda[index] = lambda;

}

/**
 * Scatter non partitioned or spatially partitioned infection message (for optional messages)
 * @param messages scatter_optional_infection_messages Sparse xmachine_message_infection_list message list
 * @param message_swap temp xmachine_message_infection_list message list to scatter sparse messages to
 */
__global__ void scatter_optional_infection_messages(xmachine_message_infection_list* messages, xmachine_message_infection_list* messages_swap){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = messages_swap->_scan_input[index];

	//if optional message is to be written
	if (_scan_input == 1){
		int output_index = messages_swap->_position[index] + d_message_infection_count;

		//AoS - xmachine_message_infection Un-Coalesced scattered memory write
		messages->_position[output_index] = output_index;
		messages->location[output_index] = messages_swap->location[index];
		messages->locationid[output_index] = messages_swap->locationid[index];
		messages->lambda[output_index] = messages_swap->lambda[index];				
	}
}

/** reset_infection_swaps
 * Reset non partitioned or spatially partitioned infection message swaps (for scattering optional messages)
 * @param message_swap message list to reset _position and _scan_input values back to 0
 */
__global__ void reset_infection_swaps(xmachine_message_infection_list* messages_swap){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	messages_swap->_position[index] = 0;
	messages_swap->_scan_input[index] = 0;
}

/* Message functions */

__device__ xmachine_message_infection* get_first_infection_message(xmachine_message_infection_list* messages){

	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];
	
	//wrap size is the number of tiles required to load all messages
	int wrap_size = (ceil((float)d_message_infection_count/ blockDim.x)* blockDim.x);

	//if no messages then return a null pointer (false)
	if (wrap_size == 0)
		return nullptr;

	//global thread index
	int global_index = (blockIdx.x*blockDim.x) + threadIdx.x;

	//global thread index
	int index = WRAP(global_index, wrap_size);

	//SoA to AoS - xmachine_message_infection Coalesced memory read
	xmachine_message_infection temp_message;
	temp_message._position = messages->_position[index];
	temp_message.location = messages->location[index];
	temp_message.locationid = messages->locationid[index];
	temp_message.lambda = messages->lambda[index];

	//AoS to shared memory
	int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_infection));
	xmachine_message_infection* sm_message = ((xmachine_message_infection*)&message_share[message_index]);
	sm_message[0] = temp_message;

	__syncthreads();

  //HACK FOR 64 bit addressing issue in sm
	return ((xmachine_message_infection*)&message_share[d_SM_START]);
}

__device__ xmachine_message_infection* get_next_infection_message(xmachine_message_infection* message, xmachine_message_infection_list* messages){

	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];
	
	//wrap size is the number of tiles required to load all messages
	int wrap_size = ceil((float)d_message_infection_count/ blockDim.x)*blockDim.x;

	int i = WRAP((message->_position + 1),wrap_size);

	//If end of messages (last message not multiple of gridsize) go to 0 index
	if (i >= d_message_infection_count)
		i = 0;

	//Check if back to start position of first message
	if (i == WRAP((blockDim.x* blockIdx.x), wrap_size))
		return nullptr;

	int tile = floor((float)i/(blockDim.x)); //tile is round down position over blockDim
	i = i % blockDim.x;						 //mod i for shared memory index

	//if count == Block Size load next tile int shared memory values
	if (i == 0){
		__syncthreads();					//make sure we don't change shared memory until all threads are here (important for emu-debug mode)
		
		//SoA to AoS - xmachine_message_infection Coalesced memory read
		int index = (tile* blockDim.x) + threadIdx.x;
		xmachine_message_infection temp_message;
		temp_message._position = messages->_position[index];
		temp_message.location = messages->location[index];
		temp_message.locationid = messages->locationid[index];
		temp_message.lambda = messages->lambda[index];

		//AoS to shared memory
		int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_infection));
		xmachine_message_infection* sm_message = ((xmachine_message_infection*)&message_share[message_index]);
		sm_message[0] = temp_message;

		__syncthreads();					//make sure we don't start returning messages until all threads have updated shared memory
	}

	int message_index = SHARE_INDEX(i, sizeof(xmachine_message_infection));
	return ((xmachine_message_infection*)&message_share[message_index]);
}


	
/////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dynamically created GPU kernels  */



/**
 *
 */
__global__ void GPUFLAME_update(xmachine_memory_Person_list* agents, xmachine_message_location_list* location_messages, RNG_rand48* rand48){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    //For agents not using non partitioned message input check the agent bounds
    if (index >= d_xmachine_memory_Person_count)
        return;
    

	//SoA to AoS - xmachine_memory_update Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_Person agent;
    
    // Thread bounds already checked, but the agent function will still execute. load default values?
	
	agent.id = agents->id[index];
	agent.step = agents->step[index];
	agent.householdtime = agents->householdtime[index];
	agent.churchtime = agents->churchtime[index];
	agent.transporttime = agents->transporttime[index];
	agent.clinictime = agents->clinictime[index];
	agent.workplacetime = agents->workplacetime[index];
	agent.outsidetime = agents->outsidetime[index];
	agent.age = agents->age[index];
	agent.gender = agents->gender[index];
	agent.householdsize = agents->householdsize[index];
	agent.churchfreq = agents->churchfreq[index];
	agent.churchdur = agents->churchdur[index];
	agent.transportdur = agents->transportdur[index];
	agent.transportday1 = agents->transportday1[index];
	agent.transportday2 = agents->transportday2[index];
	agent.household = agents->household[index];
	agent.church = agents->church[index];
	agent.transport = agents->transport[index];
	agent.workplace = agents->workplace[index];
	agent.busy = agents->busy[index];
	agent.startstep = agents->startstep[index];
	agent.location = agents->location[index];
	agent.locationid = agents->locationid[index];
	agent.hiv = agents->hiv[index];
	agent.art = agents->art[index];
	agent.activetb = agents->activetb[index];
	agent.artday = agents->artday[index];
	agent.p = agents->p[index];
	agent.q = agents->q[index];
	agent.infections = agents->infections[index];
	agent.lastinfected = agents->lastinfected[index];
	agent.lastinfectedid = agents->lastinfectedid[index];
	agent.time_step = agents->time_step[index];
	agent.lambda = agents->lambda[index];
	agent.timevisiting = agents->timevisiting[index];
	agent.bargoing = agents->bargoing[index];
	agent.barday = agents->barday[index];

	//FLAME function call
	int dead = !update(&agent, location_messages	, rand48);
	

	//continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_update Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
	agents->step[index] = agent.step;
	agents->householdtime[index] = agent.householdtime;
	agents->churchtime[index] = agent.churchtime;
	agents->transporttime[index] = agent.transporttime;
	agents->clinictime[index] = agent.clinictime;
	agents->workplacetime[index] = agent.workplacetime;
	agents->outsidetime[index] = agent.outsidetime;
	agents->age[index] = agent.age;
	agents->gender[index] = agent.gender;
	agents->householdsize[index] = agent.householdsize;
	agents->churchfreq[index] = agent.churchfreq;
	agents->churchdur[index] = agent.churchdur;
	agents->transportdur[index] = agent.transportdur;
	agents->transportday1[index] = agent.transportday1;
	agents->transportday2[index] = agent.transportday2;
	agents->household[index] = agent.household;
	agents->church[index] = agent.church;
	agents->transport[index] = agent.transport;
	agents->workplace[index] = agent.workplace;
	agents->busy[index] = agent.busy;
	agents->startstep[index] = agent.startstep;
	agents->location[index] = agent.location;
	agents->locationid[index] = agent.locationid;
	agents->hiv[index] = agent.hiv;
	agents->art[index] = agent.art;
	agents->activetb[index] = agent.activetb;
	agents->artday[index] = agent.artday;
	agents->p[index] = agent.p;
	agents->q[index] = agent.q;
	agents->infections[index] = agent.infections;
	agents->lastinfected[index] = agent.lastinfected;
	agents->lastinfectedid[index] = agent.lastinfectedid;
	agents->time_step[index] = agent.time_step;
	agents->lambda[index] = agent.lambda;
	agents->timevisiting[index] = agent.timevisiting;
	agents->bargoing[index] = agent.bargoing;
	agents->barday[index] = agent.barday;
}

/**
 *
 */
__global__ void GPUFLAME_updatelambda(xmachine_memory_Person_list* agents, xmachine_message_infection_list* infection_messages){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    
    //No partitioned input requires threads to be launched beyond the agent count to ensure full block sizes
    

	//SoA to AoS - xmachine_memory_updatelambda Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_Person agent;
    //No partitioned input may launch more threads than required - only load agent data within bounds. 
    if (index < d_xmachine_memory_Person_count){
    
	agent.id = agents->id[index];
	agent.step = agents->step[index];
	agent.householdtime = agents->householdtime[index];
	agent.churchtime = agents->churchtime[index];
	agent.transporttime = agents->transporttime[index];
	agent.clinictime = agents->clinictime[index];
	agent.workplacetime = agents->workplacetime[index];
	agent.outsidetime = agents->outsidetime[index];
	agent.age = agents->age[index];
	agent.gender = agents->gender[index];
	agent.householdsize = agents->householdsize[index];
	agent.churchfreq = agents->churchfreq[index];
	agent.churchdur = agents->churchdur[index];
	agent.transportdur = agents->transportdur[index];
	agent.transportday1 = agents->transportday1[index];
	agent.transportday2 = agents->transportday2[index];
	agent.household = agents->household[index];
	agent.church = agents->church[index];
	agent.transport = agents->transport[index];
	agent.workplace = agents->workplace[index];
	agent.busy = agents->busy[index];
	agent.startstep = agents->startstep[index];
	agent.location = agents->location[index];
	agent.locationid = agents->locationid[index];
	agent.hiv = agents->hiv[index];
	agent.art = agents->art[index];
	agent.activetb = agents->activetb[index];
	agent.artday = agents->artday[index];
	agent.p = agents->p[index];
	agent.q = agents->q[index];
	agent.infections = agents->infections[index];
	agent.lastinfected = agents->lastinfected[index];
	agent.lastinfectedid = agents->lastinfectedid[index];
	agent.time_step = agents->time_step[index];
	agent.lambda = agents->lambda[index];
	agent.timevisiting = agents->timevisiting[index];
	agent.bargoing = agents->bargoing[index];
	agent.barday = agents->barday[index];
	} else {
	
	agent.id = 0;
	agent.step = 0;
	agent.householdtime = 0;
	agent.churchtime = 0;
	agent.transporttime = 0;
	agent.clinictime = 0;
	agent.workplacetime = 0;
	agent.outsidetime = 0;
	agent.age = 0;
	agent.gender = 0;
	agent.householdsize = 0;
	agent.churchfreq = 0;
	agent.churchdur = 0;
	agent.transportdur = 0;
	agent.transportday1 = 0;
	agent.transportday2 = 0;
	agent.household = 0;
	agent.church = 0;
	agent.transport = 0;
	agent.workplace = 0;
	agent.busy = 0;
	agent.startstep = 0;
	agent.location = 0;
	agent.locationid = 0;
	agent.hiv = 0;
	agent.art = 0;
	agent.activetb = 0;
	agent.artday = 0;
	agent.p = 0;
	agent.q = 0;
	agent.infections = 0;
	agent.lastinfected = 0;
	agent.lastinfectedid = 0;
	agent.time_step = 0;
	agent.lambda = 0;
	agent.timevisiting = 0;
	agent.bargoing = 0;
	agent.barday = 0;
	}

	//FLAME function call
	int dead = !updatelambda(&agent, infection_messages);
	

	
    //No partitioned input may launch more threads than required - only write agent data within bounds. 
    if (index < d_xmachine_memory_Person_count){
    //continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_updatelambda Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
	agents->step[index] = agent.step;
	agents->householdtime[index] = agent.householdtime;
	agents->churchtime[index] = agent.churchtime;
	agents->transporttime[index] = agent.transporttime;
	agents->clinictime[index] = agent.clinictime;
	agents->workplacetime[index] = agent.workplacetime;
	agents->outsidetime[index] = agent.outsidetime;
	agents->age[index] = agent.age;
	agents->gender[index] = agent.gender;
	agents->householdsize[index] = agent.householdsize;
	agents->churchfreq[index] = agent.churchfreq;
	agents->churchdur[index] = agent.churchdur;
	agents->transportdur[index] = agent.transportdur;
	agents->transportday1[index] = agent.transportday1;
	agents->transportday2[index] = agent.transportday2;
	agents->household[index] = agent.household;
	agents->church[index] = agent.church;
	agents->transport[index] = agent.transport;
	agents->workplace[index] = agent.workplace;
	agents->busy[index] = agent.busy;
	agents->startstep[index] = agent.startstep;
	agents->location[index] = agent.location;
	agents->locationid[index] = agent.locationid;
	agents->hiv[index] = agent.hiv;
	agents->art[index] = agent.art;
	agents->activetb[index] = agent.activetb;
	agents->artday[index] = agent.artday;
	agents->p[index] = agent.p;
	agents->q[index] = agent.q;
	agents->infections[index] = agent.infections;
	agents->lastinfected[index] = agent.lastinfected;
	agents->lastinfectedid[index] = agent.lastinfectedid;
	agents->time_step[index] = agent.time_step;
	agents->lambda[index] = agent.lambda;
	agents->timevisiting[index] = agent.timevisiting;
	agents->bargoing[index] = agent.bargoing;
	agents->barday[index] = agent.barday;
	}
}

/**
 *
 */
__global__ void GPUFLAME_infect(xmachine_memory_Person_list* agents, RNG_rand48* rand48){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    //For agents not using non partitioned message input check the agent bounds
    if (index >= d_xmachine_memory_Person_count)
        return;
    

	//SoA to AoS - xmachine_memory_infect Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_Person agent;
    
    // Thread bounds already checked, but the agent function will still execute. load default values?
	
	agent.id = agents->id[index];
	agent.step = agents->step[index];
	agent.householdtime = agents->householdtime[index];
	agent.churchtime = agents->churchtime[index];
	agent.transporttime = agents->transporttime[index];
	agent.clinictime = agents->clinictime[index];
	agent.workplacetime = agents->workplacetime[index];
	agent.outsidetime = agents->outsidetime[index];
	agent.age = agents->age[index];
	agent.gender = agents->gender[index];
	agent.householdsize = agents->householdsize[index];
	agent.churchfreq = agents->churchfreq[index];
	agent.churchdur = agents->churchdur[index];
	agent.transportdur = agents->transportdur[index];
	agent.transportday1 = agents->transportday1[index];
	agent.transportday2 = agents->transportday2[index];
	agent.household = agents->household[index];
	agent.church = agents->church[index];
	agent.transport = agents->transport[index];
	agent.workplace = agents->workplace[index];
	agent.busy = agents->busy[index];
	agent.startstep = agents->startstep[index];
	agent.location = agents->location[index];
	agent.locationid = agents->locationid[index];
	agent.hiv = agents->hiv[index];
	agent.art = agents->art[index];
	agent.activetb = agents->activetb[index];
	agent.artday = agents->artday[index];
	agent.p = agents->p[index];
	agent.q = agents->q[index];
	agent.infections = agents->infections[index];
	agent.lastinfected = agents->lastinfected[index];
	agent.lastinfectedid = agents->lastinfectedid[index];
	agent.time_step = agents->time_step[index];
	agent.lambda = agents->lambda[index];
	agent.timevisiting = agents->timevisiting[index];
	agent.bargoing = agents->bargoing[index];
	agent.barday = agents->barday[index];

	//FLAME function call
	int dead = !infect(&agent, rand48);
	

	//continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_infect Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
	agents->step[index] = agent.step;
	agents->householdtime[index] = agent.householdtime;
	agents->churchtime[index] = agent.churchtime;
	agents->transporttime[index] = agent.transporttime;
	agents->clinictime[index] = agent.clinictime;
	agents->workplacetime[index] = agent.workplacetime;
	agents->outsidetime[index] = agent.outsidetime;
	agents->age[index] = agent.age;
	agents->gender[index] = agent.gender;
	agents->householdsize[index] = agent.householdsize;
	agents->churchfreq[index] = agent.churchfreq;
	agents->churchdur[index] = agent.churchdur;
	agents->transportdur[index] = agent.transportdur;
	agents->transportday1[index] = agent.transportday1;
	agents->transportday2[index] = agent.transportday2;
	agents->household[index] = agent.household;
	agents->church[index] = agent.church;
	agents->transport[index] = agent.transport;
	agents->workplace[index] = agent.workplace;
	agents->busy[index] = agent.busy;
	agents->startstep[index] = agent.startstep;
	agents->location[index] = agent.location;
	agents->locationid[index] = agent.locationid;
	agents->hiv[index] = agent.hiv;
	agents->art[index] = agent.art;
	agents->activetb[index] = agent.activetb;
	agents->artday[index] = agent.artday;
	agents->p[index] = agent.p;
	agents->q[index] = agent.q;
	agents->infections[index] = agent.infections;
	agents->lastinfected[index] = agent.lastinfected;
	agents->lastinfectedid[index] = agent.lastinfectedid;
	agents->time_step[index] = agent.time_step;
	agents->lambda[index] = agent.lambda;
	agents->timevisiting[index] = agent.timevisiting;
	agents->bargoing[index] = agent.bargoing;
	agents->barday[index] = agent.barday;
}

/**
 *
 */
__global__ void GPUFLAME_personhhinit(xmachine_memory_Person_list* agents, xmachine_message_household_membership_list* household_membership_messages){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    
    //No partitioned input requires threads to be launched beyond the agent count to ensure full block sizes
    

	//SoA to AoS - xmachine_memory_personhhinit Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_Person agent;
    //No partitioned input may launch more threads than required - only load agent data within bounds. 
    if (index < d_xmachine_memory_Person_count){
    
	agent.id = agents->id[index];
	agent.step = agents->step[index];
	agent.householdtime = agents->householdtime[index];
	agent.churchtime = agents->churchtime[index];
	agent.transporttime = agents->transporttime[index];
	agent.clinictime = agents->clinictime[index];
	agent.workplacetime = agents->workplacetime[index];
	agent.outsidetime = agents->outsidetime[index];
	agent.age = agents->age[index];
	agent.gender = agents->gender[index];
	agent.householdsize = agents->householdsize[index];
	agent.churchfreq = agents->churchfreq[index];
	agent.churchdur = agents->churchdur[index];
	agent.transportdur = agents->transportdur[index];
	agent.transportday1 = agents->transportday1[index];
	agent.transportday2 = agents->transportday2[index];
	agent.household = agents->household[index];
	agent.church = agents->church[index];
	agent.transport = agents->transport[index];
	agent.workplace = agents->workplace[index];
	agent.busy = agents->busy[index];
	agent.startstep = agents->startstep[index];
	agent.location = agents->location[index];
	agent.locationid = agents->locationid[index];
	agent.hiv = agents->hiv[index];
	agent.art = agents->art[index];
	agent.activetb = agents->activetb[index];
	agent.artday = agents->artday[index];
	agent.p = agents->p[index];
	agent.q = agents->q[index];
	agent.infections = agents->infections[index];
	agent.lastinfected = agents->lastinfected[index];
	agent.lastinfectedid = agents->lastinfectedid[index];
	agent.time_step = agents->time_step[index];
	agent.lambda = agents->lambda[index];
	agent.timevisiting = agents->timevisiting[index];
	agent.bargoing = agents->bargoing[index];
	agent.barday = agents->barday[index];
	} else {
	
	agent.id = 0;
	agent.step = 0;
	agent.householdtime = 0;
	agent.churchtime = 0;
	agent.transporttime = 0;
	agent.clinictime = 0;
	agent.workplacetime = 0;
	agent.outsidetime = 0;
	agent.age = 0;
	agent.gender = 0;
	agent.householdsize = 0;
	agent.churchfreq = 0;
	agent.churchdur = 0;
	agent.transportdur = 0;
	agent.transportday1 = 0;
	agent.transportday2 = 0;
	agent.household = 0;
	agent.church = 0;
	agent.transport = 0;
	agent.workplace = 0;
	agent.busy = 0;
	agent.startstep = 0;
	agent.location = 0;
	agent.locationid = 0;
	agent.hiv = 0;
	agent.art = 0;
	agent.activetb = 0;
	agent.artday = 0;
	agent.p = 0;
	agent.q = 0;
	agent.infections = 0;
	agent.lastinfected = 0;
	agent.lastinfectedid = 0;
	agent.time_step = 0;
	agent.lambda = 0;
	agent.timevisiting = 0;
	agent.bargoing = 0;
	agent.barday = 0;
	}

	//FLAME function call
	int dead = !personhhinit(&agent, household_membership_messages);
	

	
    //No partitioned input may launch more threads than required - only write agent data within bounds. 
    if (index < d_xmachine_memory_Person_count){
    //continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_personhhinit Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
	agents->step[index] = agent.step;
	agents->householdtime[index] = agent.householdtime;
	agents->churchtime[index] = agent.churchtime;
	agents->transporttime[index] = agent.transporttime;
	agents->clinictime[index] = agent.clinictime;
	agents->workplacetime[index] = agent.workplacetime;
	agents->outsidetime[index] = agent.outsidetime;
	agents->age[index] = agent.age;
	agents->gender[index] = agent.gender;
	agents->householdsize[index] = agent.householdsize;
	agents->churchfreq[index] = agent.churchfreq;
	agents->churchdur[index] = agent.churchdur;
	agents->transportdur[index] = agent.transportdur;
	agents->transportday1[index] = agent.transportday1;
	agents->transportday2[index] = agent.transportday2;
	agents->household[index] = agent.household;
	agents->church[index] = agent.church;
	agents->transport[index] = agent.transport;
	agents->workplace[index] = agent.workplace;
	agents->busy[index] = agent.busy;
	agents->startstep[index] = agent.startstep;
	agents->location[index] = agent.location;
	agents->locationid[index] = agent.locationid;
	agents->hiv[index] = agent.hiv;
	agents->art[index] = agent.art;
	agents->activetb[index] = agent.activetb;
	agents->artday[index] = agent.artday;
	agents->p[index] = agent.p;
	agents->q[index] = agent.q;
	agents->infections[index] = agent.infections;
	agents->lastinfected[index] = agent.lastinfected;
	agents->lastinfectedid[index] = agent.lastinfectedid;
	agents->time_step[index] = agent.time_step;
	agents->lambda[index] = agent.lambda;
	agents->timevisiting[index] = agent.timevisiting;
	agents->bargoing[index] = agent.bargoing;
	agents->barday[index] = agent.barday;
	}
}

/**
 *
 */
__global__ void GPUFLAME_persontbinit(xmachine_memory_Person_list* agents, xmachine_message_tb_assignment_list* tb_assignment_messages){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    
    //No partitioned input requires threads to be launched beyond the agent count to ensure full block sizes
    

	//SoA to AoS - xmachine_memory_persontbinit Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_Person agent;
    //No partitioned input may launch more threads than required - only load agent data within bounds. 
    if (index < d_xmachine_memory_Person_count){
    
	agent.id = agents->id[index];
	agent.step = agents->step[index];
	agent.householdtime = agents->householdtime[index];
	agent.churchtime = agents->churchtime[index];
	agent.transporttime = agents->transporttime[index];
	agent.clinictime = agents->clinictime[index];
	agent.workplacetime = agents->workplacetime[index];
	agent.outsidetime = agents->outsidetime[index];
	agent.age = agents->age[index];
	agent.gender = agents->gender[index];
	agent.householdsize = agents->householdsize[index];
	agent.churchfreq = agents->churchfreq[index];
	agent.churchdur = agents->churchdur[index];
	agent.transportdur = agents->transportdur[index];
	agent.transportday1 = agents->transportday1[index];
	agent.transportday2 = agents->transportday2[index];
	agent.household = agents->household[index];
	agent.church = agents->church[index];
	agent.transport = agents->transport[index];
	agent.workplace = agents->workplace[index];
	agent.busy = agents->busy[index];
	agent.startstep = agents->startstep[index];
	agent.location = agents->location[index];
	agent.locationid = agents->locationid[index];
	agent.hiv = agents->hiv[index];
	agent.art = agents->art[index];
	agent.activetb = agents->activetb[index];
	agent.artday = agents->artday[index];
	agent.p = agents->p[index];
	agent.q = agents->q[index];
	agent.infections = agents->infections[index];
	agent.lastinfected = agents->lastinfected[index];
	agent.lastinfectedid = agents->lastinfectedid[index];
	agent.time_step = agents->time_step[index];
	agent.lambda = agents->lambda[index];
	agent.timevisiting = agents->timevisiting[index];
	agent.bargoing = agents->bargoing[index];
	agent.barday = agents->barday[index];
	} else {
	
	agent.id = 0;
	agent.step = 0;
	agent.householdtime = 0;
	agent.churchtime = 0;
	agent.transporttime = 0;
	agent.clinictime = 0;
	agent.workplacetime = 0;
	agent.outsidetime = 0;
	agent.age = 0;
	agent.gender = 0;
	agent.householdsize = 0;
	agent.churchfreq = 0;
	agent.churchdur = 0;
	agent.transportdur = 0;
	agent.transportday1 = 0;
	agent.transportday2 = 0;
	agent.household = 0;
	agent.church = 0;
	agent.transport = 0;
	agent.workplace = 0;
	agent.busy = 0;
	agent.startstep = 0;
	agent.location = 0;
	agent.locationid = 0;
	agent.hiv = 0;
	agent.art = 0;
	agent.activetb = 0;
	agent.artday = 0;
	agent.p = 0;
	agent.q = 0;
	agent.infections = 0;
	agent.lastinfected = 0;
	agent.lastinfectedid = 0;
	agent.time_step = 0;
	agent.lambda = 0;
	agent.timevisiting = 0;
	agent.bargoing = 0;
	agent.barday = 0;
	}

	//FLAME function call
	int dead = !persontbinit(&agent, tb_assignment_messages);
	

	
    //No partitioned input may launch more threads than required - only write agent data within bounds. 
    if (index < d_xmachine_memory_Person_count){
    //continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_persontbinit Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
	agents->step[index] = agent.step;
	agents->householdtime[index] = agent.householdtime;
	agents->churchtime[index] = agent.churchtime;
	agents->transporttime[index] = agent.transporttime;
	agents->clinictime[index] = agent.clinictime;
	agents->workplacetime[index] = agent.workplacetime;
	agents->outsidetime[index] = agent.outsidetime;
	agents->age[index] = agent.age;
	agents->gender[index] = agent.gender;
	agents->householdsize[index] = agent.householdsize;
	agents->churchfreq[index] = agent.churchfreq;
	agents->churchdur[index] = agent.churchdur;
	agents->transportdur[index] = agent.transportdur;
	agents->transportday1[index] = agent.transportday1;
	agents->transportday2[index] = agent.transportday2;
	agents->household[index] = agent.household;
	agents->church[index] = agent.church;
	agents->transport[index] = agent.transport;
	agents->workplace[index] = agent.workplace;
	agents->busy[index] = agent.busy;
	agents->startstep[index] = agent.startstep;
	agents->location[index] = agent.location;
	agents->locationid[index] = agent.locationid;
	agents->hiv[index] = agent.hiv;
	agents->art[index] = agent.art;
	agents->activetb[index] = agent.activetb;
	agents->artday[index] = agent.artday;
	agents->p[index] = agent.p;
	agents->q[index] = agent.q;
	agents->infections[index] = agent.infections;
	agents->lastinfected[index] = agent.lastinfected;
	agents->lastinfectedid[index] = agent.lastinfectedid;
	agents->time_step[index] = agent.time_step;
	agents->lambda[index] = agent.lambda;
	agents->timevisiting[index] = agent.timevisiting;
	agents->bargoing[index] = agent.bargoing;
	agents->barday[index] = agent.barday;
	}
}

/**
 *
 */
__global__ void GPUFLAME_persontrinit(xmachine_memory_Person_list* agents, xmachine_message_transport_membership_list* transport_membership_messages){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    
    //No partitioned input requires threads to be launched beyond the agent count to ensure full block sizes
    

	//SoA to AoS - xmachine_memory_persontrinit Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_Person agent;
    //No partitioned input may launch more threads than required - only load agent data within bounds. 
    if (index < d_xmachine_memory_Person_count){
    
	agent.id = agents->id[index];
	agent.step = agents->step[index];
	agent.householdtime = agents->householdtime[index];
	agent.churchtime = agents->churchtime[index];
	agent.transporttime = agents->transporttime[index];
	agent.clinictime = agents->clinictime[index];
	agent.workplacetime = agents->workplacetime[index];
	agent.outsidetime = agents->outsidetime[index];
	agent.age = agents->age[index];
	agent.gender = agents->gender[index];
	agent.householdsize = agents->householdsize[index];
	agent.churchfreq = agents->churchfreq[index];
	agent.churchdur = agents->churchdur[index];
	agent.transportdur = agents->transportdur[index];
	agent.transportday1 = agents->transportday1[index];
	agent.transportday2 = agents->transportday2[index];
	agent.household = agents->household[index];
	agent.church = agents->church[index];
	agent.transport = agents->transport[index];
	agent.workplace = agents->workplace[index];
	agent.busy = agents->busy[index];
	agent.startstep = agents->startstep[index];
	agent.location = agents->location[index];
	agent.locationid = agents->locationid[index];
	agent.hiv = agents->hiv[index];
	agent.art = agents->art[index];
	agent.activetb = agents->activetb[index];
	agent.artday = agents->artday[index];
	agent.p = agents->p[index];
	agent.q = agents->q[index];
	agent.infections = agents->infections[index];
	agent.lastinfected = agents->lastinfected[index];
	agent.lastinfectedid = agents->lastinfectedid[index];
	agent.time_step = agents->time_step[index];
	agent.lambda = agents->lambda[index];
	agent.timevisiting = agents->timevisiting[index];
	agent.bargoing = agents->bargoing[index];
	agent.barday = agents->barday[index];
	} else {
	
	agent.id = 0;
	agent.step = 0;
	agent.householdtime = 0;
	agent.churchtime = 0;
	agent.transporttime = 0;
	agent.clinictime = 0;
	agent.workplacetime = 0;
	agent.outsidetime = 0;
	agent.age = 0;
	agent.gender = 0;
	agent.householdsize = 0;
	agent.churchfreq = 0;
	agent.churchdur = 0;
	agent.transportdur = 0;
	agent.transportday1 = 0;
	agent.transportday2 = 0;
	agent.household = 0;
	agent.church = 0;
	agent.transport = 0;
	agent.workplace = 0;
	agent.busy = 0;
	agent.startstep = 0;
	agent.location = 0;
	agent.locationid = 0;
	agent.hiv = 0;
	agent.art = 0;
	agent.activetb = 0;
	agent.artday = 0;
	agent.p = 0;
	agent.q = 0;
	agent.infections = 0;
	agent.lastinfected = 0;
	agent.lastinfectedid = 0;
	agent.time_step = 0;
	agent.lambda = 0;
	agent.timevisiting = 0;
	agent.bargoing = 0;
	agent.barday = 0;
	}

	//FLAME function call
	int dead = !persontrinit(&agent, transport_membership_messages);
	

	
    //No partitioned input may launch more threads than required - only write agent data within bounds. 
    if (index < d_xmachine_memory_Person_count){
    //continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_persontrinit Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
	agents->step[index] = agent.step;
	agents->householdtime[index] = agent.householdtime;
	agents->churchtime[index] = agent.churchtime;
	agents->transporttime[index] = agent.transporttime;
	agents->clinictime[index] = agent.clinictime;
	agents->workplacetime[index] = agent.workplacetime;
	agents->outsidetime[index] = agent.outsidetime;
	agents->age[index] = agent.age;
	agents->gender[index] = agent.gender;
	agents->householdsize[index] = agent.householdsize;
	agents->churchfreq[index] = agent.churchfreq;
	agents->churchdur[index] = agent.churchdur;
	agents->transportdur[index] = agent.transportdur;
	agents->transportday1[index] = agent.transportday1;
	agents->transportday2[index] = agent.transportday2;
	agents->household[index] = agent.household;
	agents->church[index] = agent.church;
	agents->transport[index] = agent.transport;
	agents->workplace[index] = agent.workplace;
	agents->busy[index] = agent.busy;
	agents->startstep[index] = agent.startstep;
	agents->location[index] = agent.location;
	agents->locationid[index] = agent.locationid;
	agents->hiv[index] = agent.hiv;
	agents->art[index] = agent.art;
	agents->activetb[index] = agent.activetb;
	agents->artday[index] = agent.artday;
	agents->p[index] = agent.p;
	agents->q[index] = agent.q;
	agents->infections[index] = agent.infections;
	agents->lastinfected[index] = agent.lastinfected;
	agents->lastinfectedid[index] = agent.lastinfectedid;
	agents->time_step[index] = agent.time_step;
	agents->lambda[index] = agent.lambda;
	agents->timevisiting[index] = agent.timevisiting;
	agents->bargoing[index] = agent.bargoing;
	agents->barday[index] = agent.barday;
	}
}

/**
 *
 */
__global__ void GPUFLAME_personwpinit(xmachine_memory_Person_list* agents, xmachine_message_workplace_membership_list* workplace_membership_messages){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    
    //No partitioned input requires threads to be launched beyond the agent count to ensure full block sizes
    

	//SoA to AoS - xmachine_memory_personwpinit Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_Person agent;
    //No partitioned input may launch more threads than required - only load agent data within bounds. 
    if (index < d_xmachine_memory_Person_count){
    
	agent.id = agents->id[index];
	agent.step = agents->step[index];
	agent.householdtime = agents->householdtime[index];
	agent.churchtime = agents->churchtime[index];
	agent.transporttime = agents->transporttime[index];
	agent.clinictime = agents->clinictime[index];
	agent.workplacetime = agents->workplacetime[index];
	agent.outsidetime = agents->outsidetime[index];
	agent.age = agents->age[index];
	agent.gender = agents->gender[index];
	agent.householdsize = agents->householdsize[index];
	agent.churchfreq = agents->churchfreq[index];
	agent.churchdur = agents->churchdur[index];
	agent.transportdur = agents->transportdur[index];
	agent.transportday1 = agents->transportday1[index];
	agent.transportday2 = agents->transportday2[index];
	agent.household = agents->household[index];
	agent.church = agents->church[index];
	agent.transport = agents->transport[index];
	agent.workplace = agents->workplace[index];
	agent.busy = agents->busy[index];
	agent.startstep = agents->startstep[index];
	agent.location = agents->location[index];
	agent.locationid = agents->locationid[index];
	agent.hiv = agents->hiv[index];
	agent.art = agents->art[index];
	agent.activetb = agents->activetb[index];
	agent.artday = agents->artday[index];
	agent.p = agents->p[index];
	agent.q = agents->q[index];
	agent.infections = agents->infections[index];
	agent.lastinfected = agents->lastinfected[index];
	agent.lastinfectedid = agents->lastinfectedid[index];
	agent.time_step = agents->time_step[index];
	agent.lambda = agents->lambda[index];
	agent.timevisiting = agents->timevisiting[index];
	agent.bargoing = agents->bargoing[index];
	agent.barday = agents->barday[index];
	} else {
	
	agent.id = 0;
	agent.step = 0;
	agent.householdtime = 0;
	agent.churchtime = 0;
	agent.transporttime = 0;
	agent.clinictime = 0;
	agent.workplacetime = 0;
	agent.outsidetime = 0;
	agent.age = 0;
	agent.gender = 0;
	agent.householdsize = 0;
	agent.churchfreq = 0;
	agent.churchdur = 0;
	agent.transportdur = 0;
	agent.transportday1 = 0;
	agent.transportday2 = 0;
	agent.household = 0;
	agent.church = 0;
	agent.transport = 0;
	agent.workplace = 0;
	agent.busy = 0;
	agent.startstep = 0;
	agent.location = 0;
	agent.locationid = 0;
	agent.hiv = 0;
	agent.art = 0;
	agent.activetb = 0;
	agent.artday = 0;
	agent.p = 0;
	agent.q = 0;
	agent.infections = 0;
	agent.lastinfected = 0;
	agent.lastinfectedid = 0;
	agent.time_step = 0;
	agent.lambda = 0;
	agent.timevisiting = 0;
	agent.bargoing = 0;
	agent.barday = 0;
	}

	//FLAME function call
	int dead = !personwpinit(&agent, workplace_membership_messages);
	

	
    //No partitioned input may launch more threads than required - only write agent data within bounds. 
    if (index < d_xmachine_memory_Person_count){
    //continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_personwpinit Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
	agents->step[index] = agent.step;
	agents->householdtime[index] = agent.householdtime;
	agents->churchtime[index] = agent.churchtime;
	agents->transporttime[index] = agent.transporttime;
	agents->clinictime[index] = agent.clinictime;
	agents->workplacetime[index] = agent.workplacetime;
	agents->outsidetime[index] = agent.outsidetime;
	agents->age[index] = agent.age;
	agents->gender[index] = agent.gender;
	agents->householdsize[index] = agent.householdsize;
	agents->churchfreq[index] = agent.churchfreq;
	agents->churchdur[index] = agent.churchdur;
	agents->transportdur[index] = agent.transportdur;
	agents->transportday1[index] = agent.transportday1;
	agents->transportday2[index] = agent.transportday2;
	agents->household[index] = agent.household;
	agents->church[index] = agent.church;
	agents->transport[index] = agent.transport;
	agents->workplace[index] = agent.workplace;
	agents->busy[index] = agent.busy;
	agents->startstep[index] = agent.startstep;
	agents->location[index] = agent.location;
	agents->locationid[index] = agent.locationid;
	agents->hiv[index] = agent.hiv;
	agents->art[index] = agent.art;
	agents->activetb[index] = agent.activetb;
	agents->artday[index] = agent.artday;
	agents->p[index] = agent.p;
	agents->q[index] = agent.q;
	agents->infections[index] = agent.infections;
	agents->lastinfected[index] = agent.lastinfected;
	agents->lastinfectedid[index] = agent.lastinfectedid;
	agents->time_step[index] = agent.time_step;
	agents->lambda[index] = agent.lambda;
	agents->timevisiting[index] = agent.timevisiting;
	agents->bargoing[index] = agent.bargoing;
	agents->barday[index] = agent.barday;
	}
}

/**
 *
 */
__global__ void GPUFLAME_tbinit(xmachine_memory_TBAssignment_list* agents, xmachine_message_tb_assignment_list* tb_assignment_messages){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    //For agents not using non partitioned message input check the agent bounds
    if (index >= d_xmachine_memory_TBAssignment_count)
        return;
    

	//SoA to AoS - xmachine_memory_tbinit Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_TBAssignment agent;
    
    // Thread bounds already checked, but the agent function will still execute. load default values?
	
	agent.id = agents->id[index];

	//FLAME function call
	int dead = !tbinit(&agent, tb_assignment_messages	);
	

	//continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_tbinit Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
}

/**
 *
 */
__global__ void GPUFLAME_hhupdate(xmachine_memory_Household_list* agents, xmachine_message_location_list* location_messages, xmachine_message_infection_list* infection_messages){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    
    //No partitioned input requires threads to be launched beyond the agent count to ensure full block sizes
    

	//SoA to AoS - xmachine_memory_hhupdate Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_Household agent;
    //No partitioned input may launch more threads than required - only load agent data within bounds. 
    if (index < d_xmachine_memory_Household_count){
    
	agent.id = agents->id[index];
	agent.lambda = agents->lambda[index];
	agent.active = agents->active[index];
	} else {
	
	agent.id = 0;
	agent.lambda = 0;
	agent.active = 0;
	}

	//FLAME function call
	int dead = !hhupdate(&agent, location_messages, infection_messages	);
	

	
    //No partitioned input may launch more threads than required - only write agent data within bounds. 
    if (index < d_xmachine_memory_Household_count){
    //continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_hhupdate Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
	agents->lambda[index] = agent.lambda;
	agents->active[index] = agent.active;
	}
}

/**
 *
 */
__global__ void GPUFLAME_hhinit(xmachine_memory_HouseholdMembership_list* agents, xmachine_message_church_membership_list* church_membership_messages, xmachine_message_household_membership_list* household_membership_messages){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    
    //No partitioned input requires threads to be launched beyond the agent count to ensure full block sizes
    

	//SoA to AoS - xmachine_memory_hhinit Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_HouseholdMembership agent;
    //No partitioned input may launch more threads than required - only load agent data within bounds. 
    if (index < d_xmachine_memory_HouseholdMembership_count){
    
	agent.household_id = agents->household_id[index];
	agent.person_id = agents->person_id[index];
	agent.household_size = agents->household_size[index];
	agent.churchgoing = agents->churchgoing[index];
	agent.churchfreq = agents->churchfreq[index];
	} else {
	
	agent.household_id = 0;
	agent.person_id = 0;
	agent.household_size = 0;
	agent.churchgoing = 0;
	agent.churchfreq = 0;
	}

	//FLAME function call
	int dead = !hhinit(&agent, church_membership_messages, household_membership_messages	);
	

	
    //No partitioned input may launch more threads than required - only write agent data within bounds. 
    if (index < d_xmachine_memory_HouseholdMembership_count){
    //continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_hhinit Coalesced memory write (ignore arrays)
	agents->household_id[index] = agent.household_id;
	agents->person_id[index] = agent.person_id;
	agents->household_size[index] = agent.household_size;
	agents->churchgoing[index] = agent.churchgoing;
	agents->churchfreq[index] = agent.churchfreq;
	}
}

/**
 *
 */
__global__ void GPUFLAME_chuupdate(xmachine_memory_Church_list* agents, xmachine_message_location_list* location_messages, xmachine_message_infection_list* infection_messages){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    
    //No partitioned input requires threads to be launched beyond the agent count to ensure full block sizes
    

	//SoA to AoS - xmachine_memory_chuupdate Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_Church agent;
    //No partitioned input may launch more threads than required - only load agent data within bounds. 
    if (index < d_xmachine_memory_Church_count){
    
	agent.id = agents->id[index];
	agent.size = agents->size[index];
	agent.lambda = agents->lambda[index];
	agent.active = agents->active[index];
	} else {
	
	agent.id = 0;
	agent.size = 0;
	agent.lambda = 0;
	agent.active = 0;
	}

	//FLAME function call
	int dead = !chuupdate(&agent, location_messages, infection_messages	);
	

	
    //No partitioned input may launch more threads than required - only write agent data within bounds. 
    if (index < d_xmachine_memory_Church_count){
    //continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_chuupdate Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
	agents->size[index] = agent.size;
	agents->lambda[index] = agent.lambda;
	agents->active[index] = agent.active;
	}
}

/**
 *
 */
__global__ void GPUFLAME_chuinit(xmachine_memory_ChurchMembership_list* agents, xmachine_message_church_membership_list* church_membership_messages){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    //For agents not using non partitioned message input check the agent bounds
    if (index >= d_xmachine_memory_ChurchMembership_count)
        return;
    

	//SoA to AoS - xmachine_memory_chuinit Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_ChurchMembership agent;
    
    // Thread bounds already checked, but the agent function will still execute. load default values?
	
	agent.church_id = agents->church_id[index];
	agent.household_id = agents->household_id[index];
	agent.churchdur = agents->churchdur[index];

	//FLAME function call
	int dead = !chuinit(&agent, church_membership_messages	);
	

	//continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_chuinit Coalesced memory write (ignore arrays)
	agents->church_id[index] = agent.church_id;
	agents->household_id[index] = agent.household_id;
	agents->churchdur[index] = agent.churchdur;
}

/**
 *
 */
__global__ void GPUFLAME_trupdate(xmachine_memory_Transport_list* agents, xmachine_message_location_list* location_messages, xmachine_message_infection_list* infection_messages){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    
    //No partitioned input requires threads to be launched beyond the agent count to ensure full block sizes
    

	//SoA to AoS - xmachine_memory_trupdate Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_Transport agent;
    //No partitioned input may launch more threads than required - only load agent data within bounds. 
    if (index < d_xmachine_memory_Transport_count){
    
	agent.id = agents->id[index];
	agent.lambda = agents->lambda[index];
	agent.active = agents->active[index];
	} else {
	
	agent.id = 0;
	agent.lambda = 0;
	agent.active = 0;
	}

	//FLAME function call
	int dead = !trupdate(&agent, location_messages, infection_messages	);
	

	
    //No partitioned input may launch more threads than required - only write agent data within bounds. 
    if (index < d_xmachine_memory_Transport_count){
    //continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_trupdate Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
	agents->lambda[index] = agent.lambda;
	agents->active[index] = agent.active;
	}
}

/**
 *
 */
__global__ void GPUFLAME_trinit(xmachine_memory_TransportMembership_list* agents, xmachine_message_transport_membership_list* transport_membership_messages){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    //For agents not using non partitioned message input check the agent bounds
    if (index >= d_xmachine_memory_TransportMembership_count)
        return;
    

	//SoA to AoS - xmachine_memory_trinit Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_TransportMembership agent;
    
    // Thread bounds already checked, but the agent function will still execute. load default values?
	
	agent.person_id = agents->person_id[index];
	agent.transport_id = agents->transport_id[index];
	agent.duration = agents->duration[index];

	//FLAME function call
	int dead = !trinit(&agent, transport_membership_messages	);
	

	//continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_trinit Coalesced memory write (ignore arrays)
	agents->person_id[index] = agent.person_id;
	agents->transport_id[index] = agent.transport_id;
	agents->duration[index] = agent.duration;
}

/**
 *
 */
__global__ void GPUFLAME_clupdate(xmachine_memory_Clinic_list* agents, xmachine_message_location_list* location_messages, xmachine_message_infection_list* infection_messages){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    
    //No partitioned input requires threads to be launched beyond the agent count to ensure full block sizes
    

	//SoA to AoS - xmachine_memory_clupdate Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_Clinic agent;
    //No partitioned input may launch more threads than required - only load agent data within bounds. 
    if (index < d_xmachine_memory_Clinic_count){
    
	agent.id = agents->id[index];
	agent.lambda = agents->lambda[index];
	} else {
	
	agent.id = 0;
	agent.lambda = 0;
	}

	//FLAME function call
	int dead = !clupdate(&agent, location_messages, infection_messages	);
	

	
    //No partitioned input may launch more threads than required - only write agent data within bounds. 
    if (index < d_xmachine_memory_Clinic_count){
    //continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_clupdate Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
	agents->lambda[index] = agent.lambda;
	}
}

/**
 *
 */
__global__ void GPUFLAME_wpupdate(xmachine_memory_Workplace_list* agents, xmachine_message_location_list* location_messages, xmachine_message_infection_list* infection_messages){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    
    //No partitioned input requires threads to be launched beyond the agent count to ensure full block sizes
    

	//SoA to AoS - xmachine_memory_wpupdate Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_Workplace agent;
    //No partitioned input may launch more threads than required - only load agent data within bounds. 
    if (index < d_xmachine_memory_Workplace_count){
    
	agent.id = agents->id[index];
	agent.lambda = agents->lambda[index];
	} else {
	
	agent.id = 0;
	agent.lambda = 0;
	}

	//FLAME function call
	int dead = !wpupdate(&agent, location_messages, infection_messages	);
	

	
    //No partitioned input may launch more threads than required - only write agent data within bounds. 
    if (index < d_xmachine_memory_Workplace_count){
    //continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_wpupdate Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
	agents->lambda[index] = agent.lambda;
	}
}

/**
 *
 */
__global__ void GPUFLAME_wpinit(xmachine_memory_WorkplaceMembership_list* agents, xmachine_message_workplace_membership_list* workplace_membership_messages){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    //For agents not using non partitioned message input check the agent bounds
    if (index >= d_xmachine_memory_WorkplaceMembership_count)
        return;
    

	//SoA to AoS - xmachine_memory_wpinit Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_WorkplaceMembership agent;
    
    // Thread bounds already checked, but the agent function will still execute. load default values?
	
	agent.person_id = agents->person_id[index];
	agent.workplace_id = agents->workplace_id[index];

	//FLAME function call
	int dead = !wpinit(&agent, workplace_membership_messages	);
	

	//continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_wpinit Coalesced memory write (ignore arrays)
	agents->person_id[index] = agent.person_id;
	agents->workplace_id[index] = agent.workplace_id;
}

/**
 *
 */
__global__ void GPUFLAME_bupdate(xmachine_memory_Bar_list* agents, xmachine_message_location_list* location_messages, xmachine_message_infection_list* infection_messages){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    
    //No partitioned input requires threads to be launched beyond the agent count to ensure full block sizes
    

	//SoA to AoS - xmachine_memory_bupdate Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_Bar agent;
    //No partitioned input may launch more threads than required - only load agent data within bounds. 
    if (index < d_xmachine_memory_Bar_count){
    
	agent.id = agents->id[index];
	agent.lambda = agents->lambda[index];
	} else {
	
	agent.id = 0;
	agent.lambda = 0;
	}

	//FLAME function call
	int dead = !bupdate(&agent, location_messages, infection_messages	);
	

	
    //No partitioned input may launch more threads than required - only write agent data within bounds. 
    if (index < d_xmachine_memory_Bar_count){
    //continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_bupdate Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
	agents->lambda[index] = agent.lambda;
	}
}

	
	
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Rand48 functions */

__device__ static glm::uvec2 RNG_rand48_iterate_single(glm::uvec2 Xn, glm::uvec2 A, glm::uvec2 C)
{
	unsigned int R0, R1;

	// low 24-bit multiplication
	const unsigned int lo00 = __umul24(Xn.x, A.x);
	const unsigned int hi00 = __umulhi(Xn.x, A.x);

	// 24bit distribution of 32bit multiplication results
	R0 = (lo00 & 0xFFFFFF);
	R1 = (lo00 >> 24) | (hi00 << 8);

	R0 += C.x; R1 += C.y;

	// transfer overflows
	R1 += (R0 >> 24);
	R0 &= 0xFFFFFF;

	// cross-terms, low/hi 24-bit multiplication
	R1 += __umul24(Xn.y, A.x);
	R1 += __umul24(Xn.x, A.y);

	R1 &= 0xFFFFFF;

	return glm::uvec2(R0, R1);
}

//Templated function
template <int AGENT_TYPE>
__device__ float rnd(RNG_rand48* rand48){

	int index;
	
	//calculate the agents index in global agent list
	if (AGENT_TYPE == DISCRETE_2D){
		int width = (blockDim.x * gridDim.x);
		glm::ivec2 global_position;
		global_position.x = (blockIdx.x * blockDim.x) + threadIdx.x;
		global_position.y = (blockIdx.y * blockDim.y) + threadIdx.y;
		index = global_position.x + (global_position.y * width);
	}else//AGENT_TYPE == CONTINOUS
		index = threadIdx.x + blockIdx.x*blockDim.x;

	glm::uvec2 state = rand48->seeds[index];
	glm::uvec2 A = rand48->A;
	glm::uvec2 C = rand48->C;

	int rand = ( state.x >> 17 ) | ( state.y << 7);

	// this actually iterates the RNG
	state = RNG_rand48_iterate_single(state, A, C);

	rand48->seeds[index] = state;

	return (float)rand/2147483647;
}

__device__ float rnd(RNG_rand48* rand48){
	return rnd<DISCRETE_2D>(rand48);
}

#endif //_FLAMEGPU_KERNELS_H_
