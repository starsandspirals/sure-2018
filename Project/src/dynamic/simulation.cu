#include "hip/hip_runtime.h"

/*
 * FLAME GPU v 1.5.X for CUDA 9
 * Copyright University of Sheffield.
 * Original Author: Dr Paul Richmond (user contributions tracked on https://github.com/FLAMEGPU/FLAMEGPU)
 * Contact: p.richmond@sheffield.ac.uk (http://www.paulrichmond.staff.shef.ac.uk)
 *
 * University of Sheffield retain all intellectual property and
 * proprietary rights in and to this software and related documentation.
 * Any use, reproduction, disclosure, or distribution of this software
 * and related documentation without an express license agreement from
 * University of Sheffield is strictly prohibited.
 *
 * For terms of licence agreement please attached licence or view licence
 * on www.flamegpu.com website.
 *
 */


  //Disable internal thrust warnings about conversions
  #ifdef _MSC_VER
  #pragma warning(push)
  #pragma warning (disable : 4267)
  #pragma warning (disable : 4244)
  #endif
  #ifdef __GNUC__
  #pragma GCC diagnostic push
  #pragma GCC diagnostic ignored "-Wunused-parameter"
  #endif

  // includes
  #include <hip/hip_runtime.h>
#include <>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <cmath>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/extrema.h>
#include <thrust/system/cuda/execution_policy.h>
#include <hipcub/hipcub.hpp>

// include FLAME kernels
#include "FLAMEGPU_kernals.cu"


#ifdef _MSC_VER
#pragma warning(pop)
#endif
#ifdef __GNUC__
#pragma GCC diagnostic pop
#endif

/* Error check function for safe CUDA API calling */
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/* Error check function for post CUDA Kernel calling */
#define gpuErrchkLaunch() { gpuLaunchAssert(__FILE__, __LINE__); }
inline void gpuLaunchAssert(const char *file, int line, bool abort=true)
{
	gpuAssert( hipPeekAtLastError(), file, line );
#ifdef _DEBUG
	gpuAssert( hipDeviceSynchronize(), file, line );
#endif
   
}

/* SM padding and offset variables */
int SM_START;
int PADDING;

unsigned int g_iterationNumber;

/* Agent Memory */

/* Agent Agent variables these lists are used in the agent function where as the other lists are used only outside the agent functions*/
xmachine_memory_Agent_list* d_Agents;      /**< Pointer to agent list (population) on the device*/
xmachine_memory_Agent_list* d_Agents_swap; /**< Pointer to agent list swap on the device (used when killing agents)*/
xmachine_memory_Agent_list* d_Agents_new;  /**< Pointer to new agent list on the device (used to hold new agents before they are appended to the population)*/
int h_xmachine_memory_Agent_count;   /**< Agent population size counter */ 
uint * d_xmachine_memory_Agent_keys;	  /**< Agent sort identifiers keys*/
uint * d_xmachine_memory_Agent_values;  /**< Agent sort identifiers value */

/* Agent state variables */
xmachine_memory_Agent_list* h_Agents_default;      /**< Pointer to agent list (population) on host*/
xmachine_memory_Agent_list* d_Agents_default;      /**< Pointer to agent list (population) on the device*/
int h_xmachine_memory_Agent_default_count;   /**< Agent population size counter */ 

/* Agent state variables */
xmachine_memory_Agent_list* h_Agents_s2;      /**< Pointer to agent list (population) on host*/
xmachine_memory_Agent_list* d_Agents_s2;      /**< Pointer to agent list (population) on the device*/
int h_xmachine_memory_Agent_s2_count;   /**< Agent population size counter */ 


/* Variables to track the state of host copies of state lists, for the purposes of host agent data access.
 * @future - if the host data is current it may be possible to avoid duplicating memcpy in xml output.
 */
unsigned int h_Agents_default_variable_id_data_iteration;
unsigned int h_Agents_default_variable_age_data_iteration;
unsigned int h_Agents_default_variable_example_array_data_iteration;
unsigned int h_Agents_default_variable_example_vector_data_iteration;
unsigned int h_Agents_default_variable_dead_data_iteration;
unsigned int h_Agents_s2_variable_id_data_iteration;
unsigned int h_Agents_s2_variable_age_data_iteration;
unsigned int h_Agents_s2_variable_example_array_data_iteration;
unsigned int h_Agents_s2_variable_example_vector_data_iteration;
unsigned int h_Agents_s2_variable_dead_data_iteration;


/* Message Memory */

  
/* CUDA Streams for function layers */
hipStream_t stream1;

/* Device memory and sizes for CUB values */

void * d_temp_scan_storage_Agent;
size_t temp_scan_storage_bytes_Agent;


/*Global condition counts*/

/* RNG rand48 */
RNG_rand48* h_rand48;    /**< Pointer to RNG_rand48 seed list on host*/
RNG_rand48* d_rand48;    /**< Pointer to RNG_rand48 seed list on device*/

/* Cuda Event Timers for Instrumentation */
#if defined(INSTRUMENT_ITERATIONS) && INSTRUMENT_ITERATIONS
	hipEvent_t instrument_iteration_start, instrument_iteration_stop;
	float instrument_iteration_milliseconds = 0.0f;
#endif
#if (defined(INSTRUMENT_AGENT_FUNCTIONS) && INSTRUMENT_AGENT_FUNCTIONS) || (defined(INSTRUMENT_INIT_FUNCTIONS) && INSTRUMENT_INIT_FUNCTIONS) || (defined(INSTRUMENT_STEP_FUNCTIONS) && INSTRUMENT_STEP_FUNCTIONS) || (defined(INSTRUMENT_EXIT_FUNCTIONS) && INSTRUMENT_EXIT_FUNCTIONS)
	hipEvent_t instrument_start, instrument_stop;
	float instrument_milliseconds = 0.0f;
#endif

/* CUDA Parallel Primatives variables */
int scan_last_sum;           /**< Indicates if the position (in message list) of last message*/
int scan_last_included;      /**< Indicates if last sum value is included in the total sum count*/

/* Agent function prototypes */

/** Agent_update
 * Agent function prototype for update function of Agent agent
 */
void Agent_update(hipStream_t &stream);

  
void setPaddingAndOffset()
{
    PROFILE_SCOPED_RANGE("setPaddingAndOffset");
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	int x64_sys = 0;

	// This function call returns 9999 for both major & minor fields, if no CUDA capable devices are present
	if (deviceProp.major == 9999 && deviceProp.minor == 9999){
		printf("Error: There is no device supporting CUDA.\n");
		exit(EXIT_FAILURE);
	}
    
    //check if double is used and supported
#ifdef _DOUBLE_SUPPORT_REQUIRED_
	printf("Simulation requires full precision double values\n");
	if ((deviceProp.major < 2)&&(deviceProp.minor < 3)){
		printf("Error: Hardware does not support full precision double values!\n");
		exit(EXIT_FAILURE);
	}
    
#endif

	//check 32 or 64bit
	x64_sys = (sizeof(void*)==8);
	if (x64_sys)
	{
		printf("64Bit System Detected\n");
	}
	else
	{
		printf("32Bit System Detected\n");
	}

	SM_START = 0;
	PADDING = 0;
  
	//copy padding and offset to GPU
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_SM_START), &SM_START, sizeof(int)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_PADDING), &PADDING, sizeof(int)));     
}

int is_sqr_pow2(int x){
	int r = (int)pow(4, ceil(log(x)/log(4)));
	return (r == x);
}

int lowest_sqr_pow2(int x){
	int l;
	
	//escape early if x is square power of 2
	if (is_sqr_pow2(x))
		return x;
	
	//lower bound		
	l = (int)pow(4, floor(log(x)/log(4)));
	
	return l;
}

/* Unary function required for hipOccupancyMaxPotentialBlockSizeVariableSMem to avoid warnings */
int no_sm(int b){
	return 0;
}

/* Unary function to return shared memory size for reorder message kernels */
int reorder_messages_sm_size(int blockSize)
{
	return sizeof(unsigned int)*(blockSize+1);
}


/** getIterationNumber
 *  Get the iteration number (host)
 *  @return a 1 indexed value for the iteration number, which is incremented at the start of each simulation step.
 *      I.e. it is 0 on up until the first call to singleIteration()
 */
extern unsigned int getIterationNumber(){
    return g_iterationNumber;
}

void initialise(char * inputfile){
    PROFILE_SCOPED_RANGE("initialise");

	//set the padding and offset values depending on architecture and OS
	setPaddingAndOffset();
  
    // Initialise some global variables
    g_iterationNumber = 0;

    // Initialise variables for tracking which iterations' data is accessible on the host.
    h_Agents_default_variable_id_data_iteration = 0;
    h_Agents_default_variable_age_data_iteration = 0;
    h_Agents_default_variable_example_array_data_iteration = 0;
    h_Agents_default_variable_example_vector_data_iteration = 0;
    h_Agents_default_variable_dead_data_iteration = 0;
    h_Agents_s2_variable_id_data_iteration = 0;
    h_Agents_s2_variable_age_data_iteration = 0;
    h_Agents_s2_variable_example_array_data_iteration = 0;
    h_Agents_s2_variable_example_vector_data_iteration = 0;
    h_Agents_s2_variable_dead_data_iteration = 0;
    



	printf("Allocating Host and Device memory\n");
    PROFILE_PUSH_RANGE("allocate host");
	/* Agent memory allocation (CPU) */
	int xmachine_Agent_SoA_size = sizeof(xmachine_memory_Agent_list);
	h_Agents_default = (xmachine_memory_Agent_list*)malloc(xmachine_Agent_SoA_size);
	h_Agents_s2 = (xmachine_memory_Agent_list*)malloc(xmachine_Agent_SoA_size);

	/* Message memory allocation (CPU) */

	//Exit if agent or message buffer sizes are to small for function outputs
    PROFILE_POP_RANGE(); //"allocate host"
	

	//read initial states
	readInitialStates(inputfile, h_Agents_default, &h_xmachine_memory_Agent_default_count);
	

    PROFILE_PUSH_RANGE("allocate device");
	
	/* Agent Agent memory allocation (GPU) */
	gpuErrchk( hipMalloc( (void**) &d_Agents, xmachine_Agent_SoA_size));
	gpuErrchk( hipMalloc( (void**) &d_Agents_swap, xmachine_Agent_SoA_size));
	gpuErrchk( hipMalloc( (void**) &d_Agents_new, xmachine_Agent_SoA_size));
    //continuous agent sort identifiers
  gpuErrchk( hipMalloc( (void**) &d_xmachine_memory_Agent_keys, xmachine_memory_Agent_MAX* sizeof(uint)));
	gpuErrchk( hipMalloc( (void**) &d_xmachine_memory_Agent_values, xmachine_memory_Agent_MAX* sizeof(uint)));
	/* default memory allocation (GPU) */
	gpuErrchk( hipMalloc( (void**) &d_Agents_default, xmachine_Agent_SoA_size));
	gpuErrchk( hipMemcpy( d_Agents_default, h_Agents_default, xmachine_Agent_SoA_size, hipMemcpyHostToDevice));
    
	/* s2 memory allocation (GPU) */
	gpuErrchk( hipMalloc( (void**) &d_Agents_s2, xmachine_Agent_SoA_size));
	gpuErrchk( hipMemcpy( d_Agents_s2, h_Agents_s2, xmachine_Agent_SoA_size, hipMemcpyHostToDevice));
    	
    PROFILE_POP_RANGE(); // "allocate device"

    /* Calculate and allocate CUB temporary memory for exclusive scans */
    
    d_temp_scan_storage_Agent = nullptr;
    temp_scan_storage_bytes_Agent = 0;
    hipcub::DeviceScan::ExclusiveSum(
        d_temp_scan_storage_Agent, 
        temp_scan_storage_bytes_Agent, 
        (int*) nullptr, 
        (int*) nullptr, 
        xmachine_memory_Agent_MAX
    );
    gpuErrchk(hipMalloc(&d_temp_scan_storage_Agent, temp_scan_storage_bytes_Agent));
    

	/*Set global condition counts*/

	/* RNG rand48 */
    PROFILE_PUSH_RANGE("Initialse RNG_rand48");
	int h_rand48_SoA_size = sizeof(RNG_rand48);
	h_rand48 = (RNG_rand48*)malloc(h_rand48_SoA_size);
	//allocate on GPU
	gpuErrchk( hipMalloc( (void**) &d_rand48, h_rand48_SoA_size));
	// calculate strided iteration constants
	static const unsigned long long a = 0x5DEECE66DLL, c = 0xB;
	int seed = 123;
	unsigned long long A, C;
	A = 1LL; C = 0LL;
	for (unsigned int i = 0; i < buffer_size_MAX; ++i) {
		C += A*c;
		A *= a;
	}
	h_rand48->A.x = A & 0xFFFFFFLL;
	h_rand48->A.y = (A >> 24) & 0xFFFFFFLL;
	h_rand48->C.x = C & 0xFFFFFFLL;
	h_rand48->C.y = (C >> 24) & 0xFFFFFFLL;
	// prepare first nThreads random numbers from seed
	unsigned long long x = (((unsigned long long)seed) << 16) | 0x330E;
	for (unsigned int i = 0; i < buffer_size_MAX; ++i) {
		x = a*x + c;
		h_rand48->seeds[i].x = x & 0xFFFFFFLL;
		h_rand48->seeds[i].y = (x >> 24) & 0xFFFFFFLL;
	}
	//copy to device
	gpuErrchk( hipMemcpy( d_rand48, h_rand48, h_rand48_SoA_size, hipMemcpyHostToDevice));

    PROFILE_POP_RANGE();

	/* Call all init functions */
	/* Prepare cuda event timers for instrumentation */
#if defined(INSTRUMENT_ITERATIONS) && INSTRUMENT_ITERATIONS
	hipEventCreate(&instrument_iteration_start);
	hipEventCreate(&instrument_iteration_stop);
#endif
#if (defined(INSTRUMENT_AGENT_FUNCTIONS) && INSTRUMENT_AGENT_FUNCTIONS) || (defined(INSTRUMENT_INIT_FUNCTIONS) && INSTRUMENT_INIT_FUNCTIONS) || (defined(INSTRUMENT_STEP_FUNCTIONS) && INSTRUMENT_STEP_FUNCTIONS) || (defined(INSTRUMENT_EXIT_FUNCTIONS) && INSTRUMENT_EXIT_FUNCTIONS)
	hipEventCreate(&instrument_start);
	hipEventCreate(&instrument_stop);
#endif

	
#if defined(INSTRUMENT_INIT_FUNCTIONS) && INSTRUMENT_INIT_FUNCTIONS
	hipEventRecord(instrument_start);
#endif
    initialiseHost();
    PROFILE_PUSH_RANGE("initialiseHost");
    PROFILE_POP_RANGE();
#if defined(INSTRUMENT_INIT_FUNCTIONS) && INSTRUMENT_INIT_FUNCTIONS
	hipEventRecord(instrument_stop);
	hipEventSynchronize(instrument_stop);
	hipEventElapsedTime(&instrument_milliseconds, instrument_start, instrument_stop);
	printf("Instrumentation: initialiseHost = %f (ms)\n", instrument_milliseconds);
#endif
	
#if defined(INSTRUMENT_INIT_FUNCTIONS) && INSTRUMENT_INIT_FUNCTIONS
	hipEventRecord(instrument_start);
#endif
    generateAgentInit();
    PROFILE_PUSH_RANGE("generateAgentInit");
    PROFILE_POP_RANGE();
#if defined(INSTRUMENT_INIT_FUNCTIONS) && INSTRUMENT_INIT_FUNCTIONS
	hipEventRecord(instrument_stop);
	hipEventSynchronize(instrument_stop);
	hipEventElapsedTime(&instrument_milliseconds, instrument_start, instrument_stop);
	printf("Instrumentation: generateAgentInit = %f (ms)\n", instrument_milliseconds);
#endif
	
  
  /* Init CUDA Streams for function layers */
  
  gpuErrchk(hipStreamCreate(&stream1));

#if defined(OUTPUT_POPULATION_PER_ITERATION) && OUTPUT_POPULATION_PER_ITERATION
	// Print the agent population size of all agents in all states
	
		printf("Init agent_Agent_default_count: %u\n",get_agent_Agent_default_count());
	
		printf("Init agent_Agent_s2_count: %u\n",get_agent_Agent_s2_count());
	
#endif
} 


void sort_Agents_default(void (*generate_key_value_pairs)(unsigned int* keys, unsigned int* values, xmachine_memory_Agent_list* agents))
{
	int blockSize;
	int minGridSize;
	int gridSize;

	//generate sort keys
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, generate_key_value_pairs, no_sm, h_xmachine_memory_Agent_default_count); 
	gridSize = (h_xmachine_memory_Agent_default_count + blockSize - 1) / blockSize;    // Round up according to array size 
	generate_key_value_pairs<<<gridSize, blockSize>>>(d_xmachine_memory_Agent_keys, d_xmachine_memory_Agent_values, d_Agents_default);
	gpuErrchkLaunch();

	//updated Thrust sort
	thrust::sort_by_key( thrust::device_pointer_cast(d_xmachine_memory_Agent_keys),  thrust::device_pointer_cast(d_xmachine_memory_Agent_keys) + h_xmachine_memory_Agent_default_count,  thrust::device_pointer_cast(d_xmachine_memory_Agent_values));
	gpuErrchkLaunch();

	//reorder agents
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, reorder_Agent_agents, no_sm, h_xmachine_memory_Agent_default_count); 
	gridSize = (h_xmachine_memory_Agent_default_count + blockSize - 1) / blockSize;    // Round up according to array size 
	reorder_Agent_agents<<<gridSize, blockSize>>>(d_xmachine_memory_Agent_values, d_Agents_default, d_Agents_swap);
	gpuErrchkLaunch();

	//swap
	xmachine_memory_Agent_list* d_Agents_temp = d_Agents_default;
	d_Agents_default = d_Agents_swap;
	d_Agents_swap = d_Agents_temp;	
}

void sort_Agents_s2(void (*generate_key_value_pairs)(unsigned int* keys, unsigned int* values, xmachine_memory_Agent_list* agents))
{
	int blockSize;
	int minGridSize;
	int gridSize;

	//generate sort keys
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, generate_key_value_pairs, no_sm, h_xmachine_memory_Agent_s2_count); 
	gridSize = (h_xmachine_memory_Agent_s2_count + blockSize - 1) / blockSize;    // Round up according to array size 
	generate_key_value_pairs<<<gridSize, blockSize>>>(d_xmachine_memory_Agent_keys, d_xmachine_memory_Agent_values, d_Agents_s2);
	gpuErrchkLaunch();

	//updated Thrust sort
	thrust::sort_by_key( thrust::device_pointer_cast(d_xmachine_memory_Agent_keys),  thrust::device_pointer_cast(d_xmachine_memory_Agent_keys) + h_xmachine_memory_Agent_s2_count,  thrust::device_pointer_cast(d_xmachine_memory_Agent_values));
	gpuErrchkLaunch();

	//reorder agents
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, reorder_Agent_agents, no_sm, h_xmachine_memory_Agent_s2_count); 
	gridSize = (h_xmachine_memory_Agent_s2_count + blockSize - 1) / blockSize;    // Round up according to array size 
	reorder_Agent_agents<<<gridSize, blockSize>>>(d_xmachine_memory_Agent_values, d_Agents_s2, d_Agents_swap);
	gpuErrchkLaunch();

	//swap
	xmachine_memory_Agent_list* d_Agents_temp = d_Agents_s2;
	d_Agents_s2 = d_Agents_swap;
	d_Agents_swap = d_Agents_temp;	
}


void cleanup(){
    PROFILE_SCOPED_RANGE("cleanup");

    /* Call all exit functions */
	
#if defined(INSTRUMENT_EXIT_FUNCTIONS) && INSTRUMENT_EXIT_FUNCTIONS
	hipEventRecord(instrument_start);
#endif

    exitFunction();
    PROFILE_PUSH_RANGE("exitFunction");
	PROFILE_POP_RANGE();

#if defined(INSTRUMENT_EXIT_FUNCTIONS) && INSTRUMENT_EXIT_FUNCTIONS
	hipEventRecord(instrument_stop);
	hipEventSynchronize(instrument_stop);
	hipEventElapsedTime(&instrument_milliseconds, instrument_start, instrument_stop);
	printf("Instrumentation: exitFunction = %f (ms)\n", instrument_milliseconds);
#endif
	

	/* Agent data free*/
	
	/* Agent Agent variables */
	gpuErrchk(hipFree(d_Agents));
	gpuErrchk(hipFree(d_Agents_swap));
	gpuErrchk(hipFree(d_Agents_new));
	
	free( h_Agents_default);
	gpuErrchk(hipFree(d_Agents_default));
	
	free( h_Agents_s2);
	gpuErrchk(hipFree(d_Agents_s2));
	

	/* Message data free */
	

    /* Free temporary CUB memory */
    
    gpuErrchk(hipFree(d_temp_scan_storage_Agent));
    d_temp_scan_storage_Agent = nullptr;
    temp_scan_storage_bytes_Agent = 0;
    
  
  /* CUDA Streams for function layers */
  
  gpuErrchk(hipStreamDestroy(stream1));

  /* CUDA Event Timers for Instrumentation */
#if defined(INSTRUMENT_ITERATIONS) && INSTRUMENT_ITERATIONS
	hipEventDestroy(instrument_iteration_start);
	hipEventDestroy(instrument_iteration_stop);
#endif
#if (defined(INSTRUMENT_AGENT_FUNCTIONS) && INSTRUMENT_AGENT_FUNCTIONS) || (defined(INSTRUMENT_INIT_FUNCTIONS) && INSTRUMENT_INIT_FUNCTIONS) || (defined(INSTRUMENT_STEP_FUNCTIONS) && INSTRUMENT_STEP_FUNCTIONS) || (defined(INSTRUMENT_EXIT_FUNCTIONS) && INSTRUMENT_EXIT_FUNCTIONS)
	hipEventDestroy(instrument_start);
	hipEventDestroy(instrument_stop);
#endif
}

void singleIteration(){
PROFILE_SCOPED_RANGE("singleIteration");

#if defined(INSTRUMENT_ITERATIONS) && INSTRUMENT_ITERATIONS
	hipEventRecord(instrument_iteration_start);
#endif

    // Increment the iteration number.
    g_iterationNumber++;

	/* set all non partitioned and spatial partitioned message counts to 0*/

	/* Call agent functions in order iterating through the layer functions */
	
	/* Layer 1*/
	
#if defined(INSTRUMENT_AGENT_FUNCTIONS) && INSTRUMENT_AGENT_FUNCTIONS
	hipEventRecord(instrument_start);
#endif
	
    PROFILE_PUSH_RANGE("Agent_update");
	Agent_update(stream1);
    PROFILE_POP_RANGE();
#if defined(INSTRUMENT_AGENT_FUNCTIONS) && INSTRUMENT_AGENT_FUNCTIONS
	hipEventRecord(instrument_stop);
	hipEventSynchronize(instrument_stop);
	hipEventElapsedTime(&instrument_milliseconds, instrument_start, instrument_stop);
	printf("Instrumentation: Agent_update = %f (ms)\n", instrument_milliseconds);
#endif
	hipDeviceSynchronize();
  
    
    /* Call all step functions */
	
#if defined(INSTRUMENT_STEP_FUNCTIONS) && INSTRUMENT_STEP_FUNCTIONS
	hipEventRecord(instrument_start);
#endif
    PROFILE_PUSH_RANGE("generateAgentStep");
	generateAgentStep();
	
    PROFILE_POP_RANGE();
#if defined(INSTRUMENT_STEP_FUNCTIONS) && INSTRUMENT_STEP_FUNCTIONS
	hipEventRecord(instrument_stop);
	hipEventSynchronize(instrument_stop);
	hipEventElapsedTime(&instrument_milliseconds, instrument_start, instrument_stop);
	printf("Instrumentation: generateAgentStep = %f (ms)\n", instrument_milliseconds);
#endif
#if defined(INSTRUMENT_STEP_FUNCTIONS) && INSTRUMENT_STEP_FUNCTIONS
	hipEventRecord(instrument_start);
#endif
    PROFILE_PUSH_RANGE("customOutputStepFunction");
	customOutputStepFunction();
	
    PROFILE_POP_RANGE();
#if defined(INSTRUMENT_STEP_FUNCTIONS) && INSTRUMENT_STEP_FUNCTIONS
	hipEventRecord(instrument_stop);
	hipEventSynchronize(instrument_stop);
	hipEventElapsedTime(&instrument_milliseconds, instrument_start, instrument_stop);
	printf("Instrumentation: customOutputStepFunction = %f (ms)\n", instrument_milliseconds);
#endif

#if defined(OUTPUT_POPULATION_PER_ITERATION) && OUTPUT_POPULATION_PER_ITERATION
	// Print the agent population size of all agents in all states
	
		printf("agent_Agent_default_count: %u\n",get_agent_Agent_default_count());
	
		printf("agent_Agent_s2_count: %u\n",get_agent_Agent_s2_count());
	
#endif

#if defined(INSTRUMENT_ITERATIONS) && INSTRUMENT_ITERATIONS
	hipEventRecord(instrument_iteration_stop);
	hipEventSynchronize(instrument_iteration_stop);
	hipEventElapsedTime(&instrument_iteration_milliseconds, instrument_iteration_start, instrument_iteration_stop);
	printf("Instrumentation: Iteration Time = %f (ms)\n", instrument_iteration_milliseconds);
#endif
}

/* Environment functions */

//host constant declaration
float h_env_PROB_DEATH;
unsigned int h_env_SCALE_FACTOR;
unsigned int h_env_MAX_AGE;


//constant setter
void set_PROB_DEATH(float* h_PROB_DEATH){
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(PROB_DEATH), h_PROB_DEATH, sizeof(float)));
    memcpy(&h_env_PROB_DEATH, h_PROB_DEATH,sizeof(float));
}

//constant getter
const float* get_PROB_DEATH(){
    return &h_env_PROB_DEATH;
}



//constant setter
void set_SCALE_FACTOR(unsigned int* h_SCALE_FACTOR){
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(SCALE_FACTOR), h_SCALE_FACTOR, sizeof(unsigned int)));
    memcpy(&h_env_SCALE_FACTOR, h_SCALE_FACTOR,sizeof(unsigned int));
}

//constant getter
const unsigned int* get_SCALE_FACTOR(){
    return &h_env_SCALE_FACTOR;
}



//constant setter
void set_MAX_AGE(unsigned int* h_MAX_AGE){
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(MAX_AGE), h_MAX_AGE, sizeof(unsigned int)));
    memcpy(&h_env_MAX_AGE, h_MAX_AGE,sizeof(unsigned int));
}

//constant getter
const unsigned int* get_MAX_AGE(){
    return &h_env_MAX_AGE;
}




/* Agent data access functions*/

    
int get_agent_Agent_MAX_count(){
    return xmachine_memory_Agent_MAX;
}


int get_agent_Agent_default_count(){
	//continuous agent
	return h_xmachine_memory_Agent_default_count;
	
}

xmachine_memory_Agent_list* get_device_Agent_default_agents(){
	return d_Agents_default;
}

xmachine_memory_Agent_list* get_host_Agent_default_agents(){
	return h_Agents_default;
}

int get_agent_Agent_s2_count(){
	//continuous agent
	return h_xmachine_memory_Agent_s2_count;
	
}

xmachine_memory_Agent_list* get_device_Agent_s2_agents(){
	return d_Agents_s2;
}

xmachine_memory_Agent_list* get_host_Agent_s2_agents(){
	return h_Agents_s2;
}



/* Host based access of agent variables*/

/** unsigned int get_Agent_default_variable_id(unsigned int index)
 * Gets the value of the id variable of an Agent agent in the default state on the host. 
 * If the data is not currently on the host, a memcpy of the data of all agents in that state list will be issued, via a global.
 * This has a potentially significant performance impact if used improperly.
 * @param index the index of the agent within the list.
 * @return value of agent variable id
 */
__host__ unsigned int get_Agent_default_variable_id(unsigned int index){
    unsigned int count = get_agent_Agent_default_count();
    unsigned int currentIteration = getIterationNumber();
    
    // If the index is within bounds - no need to check >= 0 due to unsigned.
    if(count > 0 && index < count ){
        // If necessary, copy agent data from the device to the host in the default stream
        if(h_Agents_default_variable_id_data_iteration != currentIteration){
            
            gpuErrchk(
                hipMemcpy(
                    h_Agents_default->id,
                    d_Agents_default->id,
                    count * sizeof(unsigned int),
                    hipMemcpyDeviceToHost
                )
            );
            // Update some global value indicating what data is currently present in that host array.
            h_Agents_default_variable_id_data_iteration = currentIteration;
        }

        // Return the value of the index-th element of the relevant host array.
        return h_Agents_default->id[index];

    } else {
        fprintf(stderr, "Warning: Attempting to access id for the %u th member of Agent_default. count is %u at iteration %u\n", index, count, currentIteration); //@todo
        // Otherwise we return a default value
        return 0;

    }
}

/** unsigned int get_Agent_default_variable_age(unsigned int index)
 * Gets the value of the age variable of an Agent agent in the default state on the host. 
 * If the data is not currently on the host, a memcpy of the data of all agents in that state list will be issued, via a global.
 * This has a potentially significant performance impact if used improperly.
 * @param index the index of the agent within the list.
 * @return value of agent variable age
 */
__host__ unsigned int get_Agent_default_variable_age(unsigned int index){
    unsigned int count = get_agent_Agent_default_count();
    unsigned int currentIteration = getIterationNumber();
    
    // If the index is within bounds - no need to check >= 0 due to unsigned.
    if(count > 0 && index < count ){
        // If necessary, copy agent data from the device to the host in the default stream
        if(h_Agents_default_variable_age_data_iteration != currentIteration){
            
            gpuErrchk(
                hipMemcpy(
                    h_Agents_default->age,
                    d_Agents_default->age,
                    count * sizeof(unsigned int),
                    hipMemcpyDeviceToHost
                )
            );
            // Update some global value indicating what data is currently present in that host array.
            h_Agents_default_variable_age_data_iteration = currentIteration;
        }

        // Return the value of the index-th element of the relevant host array.
        return h_Agents_default->age[index];

    } else {
        fprintf(stderr, "Warning: Attempting to access age for the %u th member of Agent_default. count is %u at iteration %u\n", index, count, currentIteration); //@todo
        // Otherwise we return a default value
        return 0;

    }
}

/** float get_Agent_default_variable_example_array(unsigned int index, unsigned int element)
 * Gets the element-th value of the example_array variable array of an Agent agent in the default state on the host. 
 * If the data is not currently on the host, a memcpy of the data of all agents in that state list will be issued, via a global.
 * This has a potentially significant performance impact if used improperly.
 * @param index the index of the agent within the list.
 * @param element the element index within the variable array
 * @return element-th value of agent variable example_array
 */
__host__ float get_Agent_default_variable_example_array(unsigned int index, unsigned int element){
    unsigned int count = get_agent_Agent_default_count();
    unsigned int numElements = 4;
    unsigned int currentIteration = getIterationNumber();
    
    // If the index is within bounds - no need to check >= 0 due to unsigned.
    if(count > 0 && index < count && element < numElements ){
        // If necessary, copy agent data from the device to the host in the default stream
        if(h_Agents_default_variable_example_array_data_iteration != currentIteration){
            
            for(unsigned int e = 0; e < numElements; e++){
                gpuErrchk(
                    hipMemcpy(
                        h_Agents_default->example_array + (e * xmachine_memory_Agent_MAX),
                        d_Agents_default->example_array + (e * xmachine_memory_Agent_MAX), 
                        count * sizeof(float), 
                        hipMemcpyDeviceToHost
                    )
                );
                // Update some global value indicating what data is currently present in that host array.
                h_Agents_default_variable_example_array_data_iteration = currentIteration;
            }
        }

        // Return the value of the index-th element of the relevant host array.
        return h_Agents_default->example_array[index + (element * xmachine_memory_Agent_MAX)];

    } else {
        fprintf(stderr, "Warning: Attempting to access the %u-th element of example_array for the %u th member of Agent_default. count is %u at iteration %u\n", element, index, count, currentIteration); //@todo
        // Otherwise we return a default value
        return 0;

    }
}

/** ivec4 get_Agent_default_variable_example_vector(unsigned int index)
 * Gets the value of the example_vector variable of an Agent agent in the default state on the host. 
 * If the data is not currently on the host, a memcpy of the data of all agents in that state list will be issued, via a global.
 * This has a potentially significant performance impact if used improperly.
 * @param index the index of the agent within the list.
 * @return value of agent variable example_vector
 */
__host__ ivec4 get_Agent_default_variable_example_vector(unsigned int index){
    unsigned int count = get_agent_Agent_default_count();
    unsigned int currentIteration = getIterationNumber();
    
    // If the index is within bounds - no need to check >= 0 due to unsigned.
    if(count > 0 && index < count ){
        // If necessary, copy agent data from the device to the host in the default stream
        if(h_Agents_default_variable_example_vector_data_iteration != currentIteration){
            
            gpuErrchk(
                hipMemcpy(
                    h_Agents_default->example_vector,
                    d_Agents_default->example_vector,
                    count * sizeof(ivec4),
                    hipMemcpyDeviceToHost
                )
            );
            // Update some global value indicating what data is currently present in that host array.
            h_Agents_default_variable_example_vector_data_iteration = currentIteration;
        }

        // Return the value of the index-th element of the relevant host array.
        return h_Agents_default->example_vector[index];

    } else {
        fprintf(stderr, "Warning: Attempting to access example_vector for the %u th member of Agent_default. count is %u at iteration %u\n", index, count, currentIteration); //@todo
        // Otherwise we return a default value
        return {0,0,0,0};

    }
}

/** unsigned int get_Agent_default_variable_dead(unsigned int index)
 * Gets the value of the dead variable of an Agent agent in the default state on the host. 
 * If the data is not currently on the host, a memcpy of the data of all agents in that state list will be issued, via a global.
 * This has a potentially significant performance impact if used improperly.
 * @param index the index of the agent within the list.
 * @return value of agent variable dead
 */
__host__ unsigned int get_Agent_default_variable_dead(unsigned int index){
    unsigned int count = get_agent_Agent_default_count();
    unsigned int currentIteration = getIterationNumber();
    
    // If the index is within bounds - no need to check >= 0 due to unsigned.
    if(count > 0 && index < count ){
        // If necessary, copy agent data from the device to the host in the default stream
        if(h_Agents_default_variable_dead_data_iteration != currentIteration){
            
            gpuErrchk(
                hipMemcpy(
                    h_Agents_default->dead,
                    d_Agents_default->dead,
                    count * sizeof(unsigned int),
                    hipMemcpyDeviceToHost
                )
            );
            // Update some global value indicating what data is currently present in that host array.
            h_Agents_default_variable_dead_data_iteration = currentIteration;
        }

        // Return the value of the index-th element of the relevant host array.
        return h_Agents_default->dead[index];

    } else {
        fprintf(stderr, "Warning: Attempting to access dead for the %u th member of Agent_default. count is %u at iteration %u\n", index, count, currentIteration); //@todo
        // Otherwise we return a default value
        return 0;

    }
}

/** unsigned int get_Agent_s2_variable_id(unsigned int index)
 * Gets the value of the id variable of an Agent agent in the s2 state on the host. 
 * If the data is not currently on the host, a memcpy of the data of all agents in that state list will be issued, via a global.
 * This has a potentially significant performance impact if used improperly.
 * @param index the index of the agent within the list.
 * @return value of agent variable id
 */
__host__ unsigned int get_Agent_s2_variable_id(unsigned int index){
    unsigned int count = get_agent_Agent_s2_count();
    unsigned int currentIteration = getIterationNumber();
    
    // If the index is within bounds - no need to check >= 0 due to unsigned.
    if(count > 0 && index < count ){
        // If necessary, copy agent data from the device to the host in the default stream
        if(h_Agents_s2_variable_id_data_iteration != currentIteration){
            
            gpuErrchk(
                hipMemcpy(
                    h_Agents_s2->id,
                    d_Agents_s2->id,
                    count * sizeof(unsigned int),
                    hipMemcpyDeviceToHost
                )
            );
            // Update some global value indicating what data is currently present in that host array.
            h_Agents_s2_variable_id_data_iteration = currentIteration;
        }

        // Return the value of the index-th element of the relevant host array.
        return h_Agents_s2->id[index];

    } else {
        fprintf(stderr, "Warning: Attempting to access id for the %u th member of Agent_s2. count is %u at iteration %u\n", index, count, currentIteration); //@todo
        // Otherwise we return a default value
        return 0;

    }
}

/** unsigned int get_Agent_s2_variable_age(unsigned int index)
 * Gets the value of the age variable of an Agent agent in the s2 state on the host. 
 * If the data is not currently on the host, a memcpy of the data of all agents in that state list will be issued, via a global.
 * This has a potentially significant performance impact if used improperly.
 * @param index the index of the agent within the list.
 * @return value of agent variable age
 */
__host__ unsigned int get_Agent_s2_variable_age(unsigned int index){
    unsigned int count = get_agent_Agent_s2_count();
    unsigned int currentIteration = getIterationNumber();
    
    // If the index is within bounds - no need to check >= 0 due to unsigned.
    if(count > 0 && index < count ){
        // If necessary, copy agent data from the device to the host in the default stream
        if(h_Agents_s2_variable_age_data_iteration != currentIteration){
            
            gpuErrchk(
                hipMemcpy(
                    h_Agents_s2->age,
                    d_Agents_s2->age,
                    count * sizeof(unsigned int),
                    hipMemcpyDeviceToHost
                )
            );
            // Update some global value indicating what data is currently present in that host array.
            h_Agents_s2_variable_age_data_iteration = currentIteration;
        }

        // Return the value of the index-th element of the relevant host array.
        return h_Agents_s2->age[index];

    } else {
        fprintf(stderr, "Warning: Attempting to access age for the %u th member of Agent_s2. count is %u at iteration %u\n", index, count, currentIteration); //@todo
        // Otherwise we return a default value
        return 0;

    }
}

/** float get_Agent_s2_variable_example_array(unsigned int index, unsigned int element)
 * Gets the element-th value of the example_array variable array of an Agent agent in the s2 state on the host. 
 * If the data is not currently on the host, a memcpy of the data of all agents in that state list will be issued, via a global.
 * This has a potentially significant performance impact if used improperly.
 * @param index the index of the agent within the list.
 * @param element the element index within the variable array
 * @return element-th value of agent variable example_array
 */
__host__ float get_Agent_s2_variable_example_array(unsigned int index, unsigned int element){
    unsigned int count = get_agent_Agent_s2_count();
    unsigned int numElements = 4;
    unsigned int currentIteration = getIterationNumber();
    
    // If the index is within bounds - no need to check >= 0 due to unsigned.
    if(count > 0 && index < count && element < numElements ){
        // If necessary, copy agent data from the device to the host in the default stream
        if(h_Agents_s2_variable_example_array_data_iteration != currentIteration){
            
            for(unsigned int e = 0; e < numElements; e++){
                gpuErrchk(
                    hipMemcpy(
                        h_Agents_s2->example_array + (e * xmachine_memory_Agent_MAX),
                        d_Agents_s2->example_array + (e * xmachine_memory_Agent_MAX), 
                        count * sizeof(float), 
                        hipMemcpyDeviceToHost
                    )
                );
                // Update some global value indicating what data is currently present in that host array.
                h_Agents_s2_variable_example_array_data_iteration = currentIteration;
            }
        }

        // Return the value of the index-th element of the relevant host array.
        return h_Agents_s2->example_array[index + (element * xmachine_memory_Agent_MAX)];

    } else {
        fprintf(stderr, "Warning: Attempting to access the %u-th element of example_array for the %u th member of Agent_s2. count is %u at iteration %u\n", element, index, count, currentIteration); //@todo
        // Otherwise we return a default value
        return 0;

    }
}

/** ivec4 get_Agent_s2_variable_example_vector(unsigned int index)
 * Gets the value of the example_vector variable of an Agent agent in the s2 state on the host. 
 * If the data is not currently on the host, a memcpy of the data of all agents in that state list will be issued, via a global.
 * This has a potentially significant performance impact if used improperly.
 * @param index the index of the agent within the list.
 * @return value of agent variable example_vector
 */
__host__ ivec4 get_Agent_s2_variable_example_vector(unsigned int index){
    unsigned int count = get_agent_Agent_s2_count();
    unsigned int currentIteration = getIterationNumber();
    
    // If the index is within bounds - no need to check >= 0 due to unsigned.
    if(count > 0 && index < count ){
        // If necessary, copy agent data from the device to the host in the default stream
        if(h_Agents_s2_variable_example_vector_data_iteration != currentIteration){
            
            gpuErrchk(
                hipMemcpy(
                    h_Agents_s2->example_vector,
                    d_Agents_s2->example_vector,
                    count * sizeof(ivec4),
                    hipMemcpyDeviceToHost
                )
            );
            // Update some global value indicating what data is currently present in that host array.
            h_Agents_s2_variable_example_vector_data_iteration = currentIteration;
        }

        // Return the value of the index-th element of the relevant host array.
        return h_Agents_s2->example_vector[index];

    } else {
        fprintf(stderr, "Warning: Attempting to access example_vector for the %u th member of Agent_s2. count is %u at iteration %u\n", index, count, currentIteration); //@todo
        // Otherwise we return a default value
        return {0,0,0,0};

    }
}

/** unsigned int get_Agent_s2_variable_dead(unsigned int index)
 * Gets the value of the dead variable of an Agent agent in the s2 state on the host. 
 * If the data is not currently on the host, a memcpy of the data of all agents in that state list will be issued, via a global.
 * This has a potentially significant performance impact if used improperly.
 * @param index the index of the agent within the list.
 * @return value of agent variable dead
 */
__host__ unsigned int get_Agent_s2_variable_dead(unsigned int index){
    unsigned int count = get_agent_Agent_s2_count();
    unsigned int currentIteration = getIterationNumber();
    
    // If the index is within bounds - no need to check >= 0 due to unsigned.
    if(count > 0 && index < count ){
        // If necessary, copy agent data from the device to the host in the default stream
        if(h_Agents_s2_variable_dead_data_iteration != currentIteration){
            
            gpuErrchk(
                hipMemcpy(
                    h_Agents_s2->dead,
                    d_Agents_s2->dead,
                    count * sizeof(unsigned int),
                    hipMemcpyDeviceToHost
                )
            );
            // Update some global value indicating what data is currently present in that host array.
            h_Agents_s2_variable_dead_data_iteration = currentIteration;
        }

        // Return the value of the index-th element of the relevant host array.
        return h_Agents_s2->dead[index];

    } else {
        fprintf(stderr, "Warning: Attempting to access dead for the %u th member of Agent_s2. count is %u at iteration %u\n", index, count, currentIteration); //@todo
        // Otherwise we return a default value
        return 0;

    }
}



/* Host based agent creation functions */
// These are only available for continuous agents.



/* copy_single_xmachine_memory_Agent_hostToDevice
 * Private function to copy a host agent struct into a device SoA agent list.
 * @param d_dst destination agent state list
 * @param h_agent agent struct
 */
void copy_single_xmachine_memory_Agent_hostToDevice(xmachine_memory_Agent_list * d_dst, xmachine_memory_Agent * h_agent){
 
		gpuErrchk(hipMemcpy(d_dst->id, &h_agent->id, sizeof(unsigned int), hipMemcpyHostToDevice));
 
		gpuErrchk(hipMemcpy(d_dst->age, &h_agent->age, sizeof(unsigned int), hipMemcpyHostToDevice));
 
	for(unsigned int i = 0; i < 4; i++){
		gpuErrchk(hipMemcpy(d_dst->example_array + (i * xmachine_memory_Agent_MAX), h_agent->example_array + i, sizeof(float), hipMemcpyHostToDevice));
    }
 
		gpuErrchk(hipMemcpy(d_dst->example_vector, &h_agent->example_vector, sizeof(ivec4), hipMemcpyHostToDevice));
 
		gpuErrchk(hipMemcpy(d_dst->dead, &h_agent->dead, sizeof(unsigned int), hipMemcpyHostToDevice));

}
/*
 * Private function to copy some elements from a host based struct of arrays to a device based struct of arrays for a single agent state.
 * Individual copies of `count` elements are performed for each agent variable or each component of agent array variables, to avoid wasted data transfer.
 * There will be a point at which a single hipMemcpy will outperform many smaller memcpys, however host based agent creation should typically only populate a fraction of the maximum buffer size, so this should be more efficient.
 * @todo - experimentally find the proportion at which transferring the whole SoA would be better and incorporate this. The same will apply to agent variable arrays.
 * 
 * @param d_dst device destination SoA
 * @oaram h_src host source SoA
 * @param count the number of agents to transfer data for
 */
void copy_partial_xmachine_memory_Agent_hostToDevice(xmachine_memory_Agent_list * d_dst, xmachine_memory_Agent_list * h_src, unsigned int count){
    // Only copy elements if there is data to move.
    if (count > 0){
	 
		gpuErrchk(hipMemcpy(d_dst->id, h_src->id, count * sizeof(unsigned int), hipMemcpyHostToDevice));
 
		gpuErrchk(hipMemcpy(d_dst->age, h_src->age, count * sizeof(unsigned int), hipMemcpyHostToDevice));
 
		for(unsigned int i = 0; i < 4; i++){
			gpuErrchk(hipMemcpy(d_dst->example_array + (i * xmachine_memory_Agent_MAX), h_src->example_array + (i * xmachine_memory_Agent_MAX), count * sizeof(float), hipMemcpyHostToDevice));
        }

 
		gpuErrchk(hipMemcpy(d_dst->example_vector, h_src->example_vector, count * sizeof(ivec4), hipMemcpyHostToDevice));
 
		gpuErrchk(hipMemcpy(d_dst->dead, h_src->dead, count * sizeof(unsigned int), hipMemcpyHostToDevice));

    }
}

xmachine_memory_Agent* h_allocate_agent_Agent(){
	xmachine_memory_Agent* agent = (xmachine_memory_Agent*)malloc(sizeof(xmachine_memory_Agent));
	// Memset the whole agent strcuture
    memset(agent, 0, sizeof(xmachine_memory_Agent));

    agent->age = 0;
	// Agent variable arrays must be allocated
    agent->example_array = (float*)malloc(4 * sizeof(float));
	// If we have a default value, set each element correctly.
	for(unsigned int index = 0; index < 4; index++){
		agent->example_array[index] = 1;
	}
	return agent;
}
void h_free_agent_Agent(xmachine_memory_Agent** agent){

    free((*agent)->example_array);
 
	free((*agent));
	(*agent) = NULL;
}
xmachine_memory_Agent** h_allocate_agent_Agent_array(unsigned int count){
	xmachine_memory_Agent ** agents = (xmachine_memory_Agent**)malloc(count * sizeof(xmachine_memory_Agent*));
	for (unsigned int i = 0; i < count; i++) {
		agents[i] = h_allocate_agent_Agent();
	}
	return agents;
}
void h_free_agent_Agent_array(xmachine_memory_Agent*** agents, unsigned int count){
	for (unsigned int i = 0; i < count; i++) {
		h_free_agent_Agent(&((*agents)[i]));
	}
	free((*agents));
	(*agents) = NULL;
}

void h_unpack_agents_Agent_AoS_to_SoA(xmachine_memory_Agent_list * dst, xmachine_memory_Agent** src, unsigned int count){
	if(count > 0){
		for(unsigned int i = 0; i < count; i++){
			 
			dst->id[i] = src[i]->id;
			 
			dst->age[i] = src[i]->age;
			 
			for(unsigned int j = 0; j < 4; j++){
				dst->example_array[(j * xmachine_memory_Agent_MAX) + i] = src[i]->example_array[j];
			}
			 
			dst->example_vector[i] = src[i]->example_vector;
			 
			dst->dead[i] = src[i]->dead;
			
		}
	}
}


void h_add_agent_Agent_default(xmachine_memory_Agent* agent){
	if (h_xmachine_memory_Agent_count + 1 > xmachine_memory_Agent_MAX){
		printf("Error: Buffer size of Agent agents in state default will be exceeded by h_add_agent_Agent_default\n");
		exit(EXIT_FAILURE);
	}	

	int blockSize;
	int minGridSize;
	int gridSize;
	unsigned int count = 1;
	
	// Copy data from host struct to device SoA for target state
	copy_single_xmachine_memory_Agent_hostToDevice(d_Agents_new, agent);

	// Use append kernel (@optimisation - This can be replaced with a pointer swap if the target state list is empty)
	hipOccupancyMaxPotentialBlockSizeVariableSMem(&minGridSize, &blockSize, append_Agent_Agents, no_sm, count);
	gridSize = (count + blockSize - 1) / blockSize;
	append_Agent_Agents <<<gridSize, blockSize, 0, stream1 >>>(d_Agents_default, d_Agents_new, h_xmachine_memory_Agent_default_count, count);
	gpuErrchkLaunch();
	// Update the number of agents in this state.
	h_xmachine_memory_Agent_default_count += count;
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_xmachine_memory_Agent_default_count), &h_xmachine_memory_Agent_default_count, sizeof(int)));
	hipDeviceSynchronize();

    // Reset host variable status flags for the relevant agent state list as the device state list has been modified.
    h_Agents_default_variable_id_data_iteration = 0;
    h_Agents_default_variable_age_data_iteration = 0;
    h_Agents_default_variable_example_array_data_iteration = 0;
    h_Agents_default_variable_example_vector_data_iteration = 0;
    h_Agents_default_variable_dead_data_iteration = 0;
    

}
void h_add_agents_Agent_default(xmachine_memory_Agent** agents, unsigned int count){
	if(count > 0){
		int blockSize;
		int minGridSize;
		int gridSize;

		if (h_xmachine_memory_Agent_count + count > xmachine_memory_Agent_MAX){
			printf("Error: Buffer size of Agent agents in state default will be exceeded by h_add_agents_Agent_default\n");
			exit(EXIT_FAILURE);
		}

		// Unpack data from AoS into the pre-existing SoA
		h_unpack_agents_Agent_AoS_to_SoA(h_Agents_default, agents, count);

		// Copy data from the host SoA to the device SoA for the target state
		copy_partial_xmachine_memory_Agent_hostToDevice(d_Agents_new, h_Agents_default, count);

		// Use append kernel (@optimisation - This can be replaced with a pointer swap if the target state list is empty)
		hipOccupancyMaxPotentialBlockSizeVariableSMem(&minGridSize, &blockSize, append_Agent_Agents, no_sm, count);
		gridSize = (count + blockSize - 1) / blockSize;
		append_Agent_Agents <<<gridSize, blockSize, 0, stream1 >>>(d_Agents_default, d_Agents_new, h_xmachine_memory_Agent_default_count, count);
		gpuErrchkLaunch();
		// Update the number of agents in this state.
		h_xmachine_memory_Agent_default_count += count;
		gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_xmachine_memory_Agent_default_count), &h_xmachine_memory_Agent_default_count, sizeof(int)));
		hipDeviceSynchronize();

        // Reset host variable status flags for the relevant agent state list as the device state list has been modified.
        h_Agents_default_variable_id_data_iteration = 0;
        h_Agents_default_variable_age_data_iteration = 0;
        h_Agents_default_variable_example_array_data_iteration = 0;
        h_Agents_default_variable_example_vector_data_iteration = 0;
        h_Agents_default_variable_dead_data_iteration = 0;
        

	}
}


void h_add_agent_Agent_s2(xmachine_memory_Agent* agent){
	if (h_xmachine_memory_Agent_count + 1 > xmachine_memory_Agent_MAX){
		printf("Error: Buffer size of Agent agents in state s2 will be exceeded by h_add_agent_Agent_s2\n");
		exit(EXIT_FAILURE);
	}	

	int blockSize;
	int minGridSize;
	int gridSize;
	unsigned int count = 1;
	
	// Copy data from host struct to device SoA for target state
	copy_single_xmachine_memory_Agent_hostToDevice(d_Agents_new, agent);

	// Use append kernel (@optimisation - This can be replaced with a pointer swap if the target state list is empty)
	hipOccupancyMaxPotentialBlockSizeVariableSMem(&minGridSize, &blockSize, append_Agent_Agents, no_sm, count);
	gridSize = (count + blockSize - 1) / blockSize;
	append_Agent_Agents <<<gridSize, blockSize, 0, stream1 >>>(d_Agents_s2, d_Agents_new, h_xmachine_memory_Agent_s2_count, count);
	gpuErrchkLaunch();
	// Update the number of agents in this state.
	h_xmachine_memory_Agent_s2_count += count;
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_xmachine_memory_Agent_s2_count), &h_xmachine_memory_Agent_s2_count, sizeof(int)));
	hipDeviceSynchronize();

    // Reset host variable status flags for the relevant agent state list as the device state list has been modified.
    h_Agents_s2_variable_id_data_iteration = 0;
    h_Agents_s2_variable_age_data_iteration = 0;
    h_Agents_s2_variable_example_array_data_iteration = 0;
    h_Agents_s2_variable_example_vector_data_iteration = 0;
    h_Agents_s2_variable_dead_data_iteration = 0;
    

}
void h_add_agents_Agent_s2(xmachine_memory_Agent** agents, unsigned int count){
	if(count > 0){
		int blockSize;
		int minGridSize;
		int gridSize;

		if (h_xmachine_memory_Agent_count + count > xmachine_memory_Agent_MAX){
			printf("Error: Buffer size of Agent agents in state s2 will be exceeded by h_add_agents_Agent_s2\n");
			exit(EXIT_FAILURE);
		}

		// Unpack data from AoS into the pre-existing SoA
		h_unpack_agents_Agent_AoS_to_SoA(h_Agents_s2, agents, count);

		// Copy data from the host SoA to the device SoA for the target state
		copy_partial_xmachine_memory_Agent_hostToDevice(d_Agents_new, h_Agents_s2, count);

		// Use append kernel (@optimisation - This can be replaced with a pointer swap if the target state list is empty)
		hipOccupancyMaxPotentialBlockSizeVariableSMem(&minGridSize, &blockSize, append_Agent_Agents, no_sm, count);
		gridSize = (count + blockSize - 1) / blockSize;
		append_Agent_Agents <<<gridSize, blockSize, 0, stream1 >>>(d_Agents_s2, d_Agents_new, h_xmachine_memory_Agent_s2_count, count);
		gpuErrchkLaunch();
		// Update the number of agents in this state.
		h_xmachine_memory_Agent_s2_count += count;
		gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_xmachine_memory_Agent_s2_count), &h_xmachine_memory_Agent_s2_count, sizeof(int)));
		hipDeviceSynchronize();

        // Reset host variable status flags for the relevant agent state list as the device state list has been modified.
        h_Agents_s2_variable_id_data_iteration = 0;
        h_Agents_s2_variable_age_data_iteration = 0;
        h_Agents_s2_variable_example_array_data_iteration = 0;
        h_Agents_s2_variable_example_vector_data_iteration = 0;
        h_Agents_s2_variable_dead_data_iteration = 0;
        

	}
}


/*  Analytics Functions */

unsigned int reduce_Agent_default_id_variable(){
    //reduce in default stream
    return thrust::reduce(thrust::device_pointer_cast(d_Agents_default->id),  thrust::device_pointer_cast(d_Agents_default->id) + h_xmachine_memory_Agent_default_count);
}

unsigned int count_Agent_default_id_variable(int count_value){
    //count in default stream
    return (int)thrust::count(thrust::device_pointer_cast(d_Agents_default->id),  thrust::device_pointer_cast(d_Agents_default->id) + h_xmachine_memory_Agent_default_count, count_value);
}
unsigned int min_Agent_default_id_variable(){
    //min in default stream
    thrust::device_ptr<unsigned int> thrust_ptr = thrust::device_pointer_cast(d_Agents_default->id);
    size_t result_offset = thrust::min_element(thrust_ptr, thrust_ptr + h_xmachine_memory_Agent_default_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}
unsigned int max_Agent_default_id_variable(){
    //max in default stream
    thrust::device_ptr<unsigned int> thrust_ptr = thrust::device_pointer_cast(d_Agents_default->id);
    size_t result_offset = thrust::max_element(thrust_ptr, thrust_ptr + h_xmachine_memory_Agent_default_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}
unsigned int reduce_Agent_default_age_variable(){
    //reduce in default stream
    return thrust::reduce(thrust::device_pointer_cast(d_Agents_default->age),  thrust::device_pointer_cast(d_Agents_default->age) + h_xmachine_memory_Agent_default_count);
}

unsigned int count_Agent_default_age_variable(int count_value){
    //count in default stream
    return (int)thrust::count(thrust::device_pointer_cast(d_Agents_default->age),  thrust::device_pointer_cast(d_Agents_default->age) + h_xmachine_memory_Agent_default_count, count_value);
}
unsigned int min_Agent_default_age_variable(){
    //min in default stream
    thrust::device_ptr<unsigned int> thrust_ptr = thrust::device_pointer_cast(d_Agents_default->age);
    size_t result_offset = thrust::min_element(thrust_ptr, thrust_ptr + h_xmachine_memory_Agent_default_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}
unsigned int max_Agent_default_age_variable(){
    //max in default stream
    thrust::device_ptr<unsigned int> thrust_ptr = thrust::device_pointer_cast(d_Agents_default->age);
    size_t result_offset = thrust::max_element(thrust_ptr, thrust_ptr + h_xmachine_memory_Agent_default_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}
ivec4 reduce_Agent_default_example_vector_variable(){
    //reduce in default stream
    return thrust::reduce(thrust::device_pointer_cast(d_Agents_default->example_vector),  thrust::device_pointer_cast(d_Agents_default->example_vector) + h_xmachine_memory_Agent_default_count);
}

unsigned int reduce_Agent_default_dead_variable(){
    //reduce in default stream
    return thrust::reduce(thrust::device_pointer_cast(d_Agents_default->dead),  thrust::device_pointer_cast(d_Agents_default->dead) + h_xmachine_memory_Agent_default_count);
}

unsigned int count_Agent_default_dead_variable(int count_value){
    //count in default stream
    return (int)thrust::count(thrust::device_pointer_cast(d_Agents_default->dead),  thrust::device_pointer_cast(d_Agents_default->dead) + h_xmachine_memory_Agent_default_count, count_value);
}
unsigned int min_Agent_default_dead_variable(){
    //min in default stream
    thrust::device_ptr<unsigned int> thrust_ptr = thrust::device_pointer_cast(d_Agents_default->dead);
    size_t result_offset = thrust::min_element(thrust_ptr, thrust_ptr + h_xmachine_memory_Agent_default_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}
unsigned int max_Agent_default_dead_variable(){
    //max in default stream
    thrust::device_ptr<unsigned int> thrust_ptr = thrust::device_pointer_cast(d_Agents_default->dead);
    size_t result_offset = thrust::max_element(thrust_ptr, thrust_ptr + h_xmachine_memory_Agent_default_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}
unsigned int reduce_Agent_s2_id_variable(){
    //reduce in default stream
    return thrust::reduce(thrust::device_pointer_cast(d_Agents_s2->id),  thrust::device_pointer_cast(d_Agents_s2->id) + h_xmachine_memory_Agent_s2_count);
}

unsigned int count_Agent_s2_id_variable(int count_value){
    //count in default stream
    return (int)thrust::count(thrust::device_pointer_cast(d_Agents_s2->id),  thrust::device_pointer_cast(d_Agents_s2->id) + h_xmachine_memory_Agent_s2_count, count_value);
}
unsigned int min_Agent_s2_id_variable(){
    //min in default stream
    thrust::device_ptr<unsigned int> thrust_ptr = thrust::device_pointer_cast(d_Agents_s2->id);
    size_t result_offset = thrust::min_element(thrust_ptr, thrust_ptr + h_xmachine_memory_Agent_s2_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}
unsigned int max_Agent_s2_id_variable(){
    //max in default stream
    thrust::device_ptr<unsigned int> thrust_ptr = thrust::device_pointer_cast(d_Agents_s2->id);
    size_t result_offset = thrust::max_element(thrust_ptr, thrust_ptr + h_xmachine_memory_Agent_s2_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}
unsigned int reduce_Agent_s2_age_variable(){
    //reduce in default stream
    return thrust::reduce(thrust::device_pointer_cast(d_Agents_s2->age),  thrust::device_pointer_cast(d_Agents_s2->age) + h_xmachine_memory_Agent_s2_count);
}

unsigned int count_Agent_s2_age_variable(int count_value){
    //count in default stream
    return (int)thrust::count(thrust::device_pointer_cast(d_Agents_s2->age),  thrust::device_pointer_cast(d_Agents_s2->age) + h_xmachine_memory_Agent_s2_count, count_value);
}
unsigned int min_Agent_s2_age_variable(){
    //min in default stream
    thrust::device_ptr<unsigned int> thrust_ptr = thrust::device_pointer_cast(d_Agents_s2->age);
    size_t result_offset = thrust::min_element(thrust_ptr, thrust_ptr + h_xmachine_memory_Agent_s2_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}
unsigned int max_Agent_s2_age_variable(){
    //max in default stream
    thrust::device_ptr<unsigned int> thrust_ptr = thrust::device_pointer_cast(d_Agents_s2->age);
    size_t result_offset = thrust::max_element(thrust_ptr, thrust_ptr + h_xmachine_memory_Agent_s2_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}
ivec4 reduce_Agent_s2_example_vector_variable(){
    //reduce in default stream
    return thrust::reduce(thrust::device_pointer_cast(d_Agents_s2->example_vector),  thrust::device_pointer_cast(d_Agents_s2->example_vector) + h_xmachine_memory_Agent_s2_count);
}

unsigned int reduce_Agent_s2_dead_variable(){
    //reduce in default stream
    return thrust::reduce(thrust::device_pointer_cast(d_Agents_s2->dead),  thrust::device_pointer_cast(d_Agents_s2->dead) + h_xmachine_memory_Agent_s2_count);
}

unsigned int count_Agent_s2_dead_variable(int count_value){
    //count in default stream
    return (int)thrust::count(thrust::device_pointer_cast(d_Agents_s2->dead),  thrust::device_pointer_cast(d_Agents_s2->dead) + h_xmachine_memory_Agent_s2_count, count_value);
}
unsigned int min_Agent_s2_dead_variable(){
    //min in default stream
    thrust::device_ptr<unsigned int> thrust_ptr = thrust::device_pointer_cast(d_Agents_s2->dead);
    size_t result_offset = thrust::min_element(thrust_ptr, thrust_ptr + h_xmachine_memory_Agent_s2_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}
unsigned int max_Agent_s2_dead_variable(){
    //max in default stream
    thrust::device_ptr<unsigned int> thrust_ptr = thrust::device_pointer_cast(d_Agents_s2->dead);
    size_t result_offset = thrust::max_element(thrust_ptr, thrust_ptr + h_xmachine_memory_Agent_s2_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}



/* Agent functions */


	
/* Shared memory size calculator for agent function */
int Agent_update_sm_size(int blockSize){
	int sm_size;
	sm_size = SM_START;
  
	return sm_size;
}

/** Agent_update
 * Agent function prototype for update function of Agent agent
 */
void Agent_update(hipStream_t &stream){

    int sm_size;
    int blockSize;
    int minGridSize;
    int gridSize;
    int state_list_size;
	dim3 g; //grid for agent func
	dim3 b; //block for agent func

	
	//CHECK THE CURRENT STATE LIST COUNT IS NOT EQUAL TO 0
	
	if (h_xmachine_memory_Agent_default_count == 0)
	{
		return;
	}
	
	
	//SET SM size to 0 and save state list size for occupancy calculations
	sm_size = SM_START;
	state_list_size = h_xmachine_memory_Agent_default_count;

	

	//******************************** AGENT FUNCTION CONDITION *********************
	//THERE IS NOT A FUNCTION CONDITION
	//currentState maps to working list
	xmachine_memory_Agent_list* Agents_default_temp = d_Agents;
	d_Agents = d_Agents_default;
	d_Agents_default = Agents_default_temp;
	//set working count to current state count
	h_xmachine_memory_Agent_count = h_xmachine_memory_Agent_default_count;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_Agent_count), &h_xmachine_memory_Agent_count, sizeof(int)));	
	//set current state count to 0
	h_xmachine_memory_Agent_default_count = 0;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_Agent_default_count), &h_xmachine_memory_Agent_default_count, sizeof(int)));	
	
 

	//******************************** AGENT FUNCTION *******************************

	
	
	//calculate the grid block size for main agent function
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, GPUFLAME_update, Agent_update_sm_size, state_list_size);
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	b.x = blockSize;
	g.x = gridSize;
	
	sm_size = Agent_update_sm_size(blockSize);
	
	
	
	//IF CONTINUOUS AGENT CAN REALLOCATE (process dead agents) THEN RESET AGENT SWAPS	
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, reset_Agent_scan_input, no_sm, state_list_size); 
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	reset_Agent_scan_input<<<gridSize, blockSize, 0, stream>>>(d_Agents);
	gpuErrchkLaunch();
	
	
	//MAIN XMACHINE FUNCTION CALL (update)
	//Reallocate   : true
	//Input        : 
	//Output       : 
	//Agent Output : 
	GPUFLAME_update<<<g, b, sm_size, stream>>>(d_Agents, d_rand48);
	gpuErrchkLaunch();
	
	
	//FOR CONTINUOUS AGENTS WITH REALLOCATION REMOVE POSSIBLE DEAD AGENTS	
    hipcub::DeviceScan::ExclusiveSum(
        d_temp_scan_storage_Agent, 
        temp_scan_storage_bytes_Agent, 
        d_Agents->_scan_input,
        d_Agents->_position,
        h_xmachine_memory_Agent_count, 
        stream
    );

	//Scatter into swap
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, scatter_Agent_Agents, no_sm, state_list_size); 
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	scatter_Agent_Agents<<<gridSize, blockSize, 0, stream>>>(d_Agents_swap, d_Agents, 0, h_xmachine_memory_Agent_count);
	gpuErrchkLaunch();
	//use a temp pointer to make swap default
	xmachine_memory_Agent_list* update_Agents_temp = d_Agents;
	d_Agents = d_Agents_swap;
	d_Agents_swap = update_Agents_temp;
	//reset agent count
	gpuErrchk( hipMemcpy( &scan_last_sum, &d_Agents_swap->_position[h_xmachine_memory_Agent_count-1], sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk( hipMemcpy( &scan_last_included, &d_Agents_swap->_scan_input[h_xmachine_memory_Agent_count-1], sizeof(int), hipMemcpyDeviceToHost));
	if (scan_last_included == 1)
		h_xmachine_memory_Agent_count = scan_last_sum+1;
	else
		h_xmachine_memory_Agent_count = scan_last_sum;
	//Copy count to device
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_Agent_count), &h_xmachine_memory_Agent_count, sizeof(int)));	
	
	
	//************************ MOVE AGENTS TO NEXT STATE ****************************
    
	//check the working agents wont exceed the buffer size in the new state list
	if (h_xmachine_memory_Agent_default_count+h_xmachine_memory_Agent_count > xmachine_memory_Agent_MAX){
		printf("Error: Buffer size of update agents in state default will be exceeded moving working agents to next state in function update\n");
      exit(EXIT_FAILURE);
      }
      
  //append agents to next state list
  hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, append_Agent_Agents, no_sm, state_list_size);
  gridSize = (state_list_size + blockSize - 1) / blockSize;
  append_Agent_Agents<<<gridSize, blockSize, 0, stream>>>(d_Agents_default, d_Agents, h_xmachine_memory_Agent_default_count, h_xmachine_memory_Agent_count);
  gpuErrchkLaunch();
        
	//update new state agent size
	h_xmachine_memory_Agent_default_count += h_xmachine_memory_Agent_count;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_Agent_default_count), &h_xmachine_memory_Agent_default_count, sizeof(int)));	
	
	
}


 
extern void reset_Agent_default_count()
{
    h_xmachine_memory_Agent_default_count = 0;
}
 
extern void reset_Agent_s2_count()
{
    h_xmachine_memory_Agent_s2_count = 0;
}
